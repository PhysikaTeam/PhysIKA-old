#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "NeighborQuery.h"
#include "Core/Utility.h"
#include "Framework/Framework/Node.h"
#include "Framework/Topology/NeighborList.h"
#include "Framework/Topology/FieldNeighbor.h"
#include "Framework/Framework/SceneGraph.h"
#include "Core/Utility/Scan.h"



namespace PhysIKA
{
	__constant__ int offset1[27][3] = { 0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	IMPLEMENT_CLASS_1(NeighborQuery, TDataType)

	template<typename TDataType>
	NeighborQuery<TDataType>::NeighborQuery()
		: ComputeModule()
		, m_maxNum(0)
	{

		Vector3f sceneLow = SceneGraph::getInstance().getLowerBound();
		Vector3f sceneUp = SceneGraph::getInstance().getUpperBound();

		m_lowBound = Coord(sceneLow[0], sceneLow[1], sceneLow[2]);
		m_highBound = Coord(sceneUp[0], sceneUp[1], sceneUp[2]);
		this->inRadius()->setValue(Real(0.011));

		m_hash.setSpace(this->inRadius()->getValue(), m_lowBound, m_highBound);

//		attachField(&m_radius, "Radius", "Radius of the searching area", false);
//		attachField(&m_position, "position", "Storing the particle positions!", false);
//		attachField(&m_neighborhood, "ParticleNeighbor", "Storing particle neighbors!", false);
	}


	template<typename TDataType>
	NeighborQuery<TDataType>::NeighborQuery(DeviceArray<Coord>& position)
		: ComputeModule()
	{
		Vector3f sceneLow = SceneGraph::getInstance().getLowerBound();
		Vector3f sceneUp = SceneGraph::getInstance().getUpperBound();

		m_lowBound = Coord(sceneLow[0], sceneLow[1], sceneLow[2]);
		m_highBound = Coord(sceneUp[0], sceneUp[1], sceneUp[2]);
		this->inRadius()->setValue(Real(0.011));

		this->inPosition()->setElementCount(position.size());
		Function1Pt::copy(this->inPosition()->getValue(), position);

//		attachField(&m_radius, "Radius", "Radius of the searching area", false);
//		attachField(&m_position, "position", "Storing the particle positions!", false);
//		attachField(&m_neighborhood, "ParticleNeighbor", "Storing particle neighbors!", false);
	}

	template<typename TDataType>
	NeighborQuery<TDataType>::~NeighborQuery()
	{
		m_hash.release();
	}

	template<typename TDataType>
	NeighborQuery<TDataType>::NeighborQuery(Real s, Coord lo, Coord hi)
		: ComputeModule()
		, m_maxNum(0)
	{
		this->inRadius()->setValue(Real(s));

		m_lowBound = lo;
		m_highBound = hi;

//		attachField(&m_radius, "Radius", "Radius of the searching area", false);
//		attachField(&m_position, "position", "Storing the particle positions!", false);
//		attachField(&m_neighborhood, "ParticleNeighbor", "Storing particle neighbors!", false);
	}

	template<typename TDataType>
	bool NeighborQuery<TDataType>::initializeImpl()
	{
		if (!this->inPosition()->isEmpty() && this->outNeighborhood()->isEmpty())
		{
			this->outNeighborhood()->setElementCount(this->inPosition()->getElementCount(), m_maxNum);
		}

		if (this->inPosition()->isEmpty() || this->inRadius()->isEmpty())
		{
			std::cout << "Exception: " << std::string("NeighborQuery's fields are not fully initialized!") << "\n";
			return false;
		}

		int pNum = this->inPosition()->getElementCount();

		HostArray<Coord> hostPos;
		hostPos.resize(pNum);

// 		Function1Pt::copy(hostPos, m_position.getValue());
// 
// 		m_lowBound = Vector3f(10000000, 10000000, 10000000);
// 		m_highBound = Vector3f(-10000000, -10000000, -10000000);
// 
// 		for (int i = 0; i < pNum; i++)
// 		{
// 			m_lowBound[0] = min(hostPos[i][0], m_lowBound[0]);
// 			m_lowBound[1] = min(hostPos[i][1], m_lowBound[1]);
// 			m_lowBound[2] = min(hostPos[i][2], m_lowBound[2]);
// 
// 			m_highBound[0] = max(hostPos[i][0], m_highBound[0]);
// 			m_highBound[1] = max(hostPos[i][1], m_highBound[1]);
// 			m_highBound[2] = max(hostPos[i][2], m_highBound[2]);
// 		}

	
		m_hash.setSpace(this->inRadius()->getValue(), m_lowBound, m_highBound);

//		m_reduce = Reduction<int>::Create(m_position.getElementCount());
		triangle_first = true;
		compute();

		return true;
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::compute()
	{
		if(this->inTriangleIndex()->isEmpty())
		{ 
			if (!this->inPosition()->isEmpty())
			{
				int p_num = this->inPosition()->getElementCount();
				if (this->outNeighborhood()->getElementCount() != p_num)
				{
					this->outNeighborhood()->setElementCount(p_num);
				}

				m_hash.clear();
				m_hash.construct(this->inPosition()->getValue());

				if (!this->outNeighborhood()->getValue().isLimited())
				{
					queryNeighborDynamic(this->outNeighborhood()->getValue(), this->inPosition()->getValue(), this->inRadius()->getValue());
				}
				else
				{
					queryNeighborFixed(this->outNeighborhood()->getValue(), this->inPosition()->getValue(), this->inRadius()->getValue());
				}
			}
		}
		else
		{
			if (!this->inPosition()->isEmpty())
			{
				int p_num = this->inPosition()->getElementCount();
				if (this->outNeighborhood()->getElementCount() != p_num)
				{
					this->outNeighborhood()->setElementCount(p_num);
				}

				if (triangle_first)
					m_hash.clear();
				//printf("hash clear\n");

				if (triangle_first)
					m_hash.construct(this->inPosition()->getValue(), this->inTriangleIndex()->getValue(), this->inTrianglePosition()->getValue());

				//printf("hash constract\n");
				triangle_first = false;
				queryNeighborTriDynamic(this->outNeighborhood()->getValue(), this->inPosition()->getValue(), this->inTrianglePosition()->getValue(), this->inTriangleIndex()->getValue(), this->inRadius()->getValue());
			}
		}
	}


	template<typename TDataType>
	void NeighborQuery<TDataType>::setBoundingBox(Coord lowerBound, Coord upperBound)
	{
		m_lowBound = lowerBound;
		m_highBound = upperBound;
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::queryParticleNeighbors(NeighborList<int>& nbr, DeviceArray<Coord>& pos, Real radius)
	{
		HostArray<Coord> hostPos;
		hostPos.resize(pos.size());

		Function1Pt::copy(hostPos, pos);

// 		m_lowBound = Vector3f(10000000, 10000000, 10000000);
// 		m_highBound = Vector3f(-10000000, -10000000, -10000000);
// 
// 		for (int i = 0; i < pos.size(); i++)
// 		{
// 			m_lowBound[0] = min(hostPos[i][0], m_lowBound[0]);
// 			m_lowBound[1] = min(hostPos[i][1], m_lowBound[1]);
// 			m_lowBound[2] = min(hostPos[i][2], m_lowBound[2]);
// 
// 			m_highBound[0] = max(hostPos[i][0], m_highBound[0]);
// 			m_highBound[1] = max(hostPos[i][1], m_highBound[1]);
// 			m_highBound[2] = max(hostPos[i][2], m_highBound[2]);
// 		}

		m_hash.setSpace(radius, m_lowBound, m_highBound);
		m_hash.construct(this->inPosition()->getValue());

		if (!nbr.isLimited())
		{
			queryNeighborDynamic(nbr, pos, radius);
		}
		else
		{
			queryNeighborFixed(nbr, pos, radius);
		}
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_CalNeighborSize(
		DeviceArray<int> count,
		DeviceArray<Coord> position_new,
		DeviceArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId > position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset1[c][0], gId3.y + offset1[c][1], gId3.z + offset1[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						counter++;
					}
				}
			}
		}

		count[pId] = counter;
	}
	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_CalNeighborSizeTri(
		DeviceArray<int> count,
		DeviceArray<Coord> position_new,
		DeviceArray<Coord> position,
		DeviceArray<TopologyModule::Triangle> m_triangle_index,
		DeviceArray<Coord> positionTri,
		GridHash<TDataType> hash,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int nbr_tri[300];
		bool flag[100];
		//for (int i = 0; i < 100; i++)flag[i] = false;

		int counter = 0;
		for (int c = 0; c < 1; c++)
		{
			int cId = hash.getIndex(gId3.x + offset1[c][0], gId3.y + offset1[c][1], gId3.z + offset1[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);// min(hash.getCounter(cId), hash.npMax);
			//	if(counter > 0)
				//printf("%d\n", counter);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij;
					if (nbId >= 0)
						d_ij = h + EPSILON;//(pos_ijk - position[nbId]).norm();
					else
					{
						nbId *= -1;
						nbId -= 1;
						Point3D p3d = Point3D(pos_ijk);
						Triangle3D t3d = Triangle3D(positionTri[m_triangle_index[nbId][0]], positionTri[m_triangle_index[nbId][1]], positionTri[m_triangle_index[nbId][2]]);
						d_ij = p3d.distance(t3d);
						
					}
					if (d_ij < h)
					{
						bool legal = true;
						if (legal)
						{
							flag[nbId % 100] = 1;
							nbr_tri[counter] = nbId;
							counter++;
						}
					}
				}
			}
		}

		count[pId] = counter;
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_GetNeighborElements(
		NeighborList<int> nbr,
		DeviceArray<Coord> position_new,
		DeviceArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId > position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int j = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset1[c][0], gId3.y + offset1[c][1], gId3.z + offset1[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);// min(hash.getCounter(cId), hash.npMax);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						nbr.setElement(pId, j, nbId);
						j++;
					}
				}
			}
		}
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_GetNeighborElementsTri(
		NeighborList<int> nbr,
		DeviceArray<Coord> position_new,
		DeviceArray<Coord> position,
		DeviceArray<TopologyModule::Triangle> m_triangle_index,
		DeviceArray<Coord> positionTri,
		GridHash<TDataType> hash,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);
		
		int j = 0;
		bool tmp = false;
		bool flag[100];
		//for (int i = 0; i < 100; i++)flag[i] = false;

		for (int c = 0; c < 1; c++)
		{
			int cId = hash.getIndex(gId3.x + offset1[c][0], gId3.y + offset1[c][1], gId3.z + offset1[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					if (nbId >= 0)
					{
					}
					else
					{
						int nb_id = nbId;
						Real d_ij;
						nb_id *= -1;
						nbId *= -1;
						nbId -= 1;
						nb_id -= 1;
						Point3D p3d = Point3D(pos_ijk);
						Triangle3D t3d = Triangle3D(positionTri[m_triangle_index[nbId][0]], positionTri[m_triangle_index[nbId][1]], positionTri[m_triangle_index[nbId][2]]);
						d_ij = p3d.distance(t3d);
						if ((d_ij) < h)
						{
							if (abs(d_ij) < h)
							{
								bool legal = true;
								if (legal)
								{
									nbr.setElement(pId, j, (-nbId - 1));
									j++;
								}
							}


						}

					}
				}
			}
		}
		
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::queryNeighborSize(DeviceArray<int>& num, DeviceArray<Coord>& pos, Real h)
	{
		uint pDims = cudaGridSize(num.size(), BLOCK_SIZE);
		K_CalNeighborSize << <pDims, BLOCK_SIZE >> > (num, pos, this->inPosition()->getValue(), m_hash, h);
		cuSynchronize();
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::queryNeighborSizeTri(DeviceArray<int>& num, DeviceArray<Coord>& pos, DeviceArray<Triangle>& Tris, DeviceArray<Coord>& posT, Real h)
	{
		uint pDims = cudaGridSize(num.size(), BLOCK_SIZE);
		K_CalNeighborSizeTri << <pDims, BLOCK_SIZE >> > (num, pos, this->inPosition()->getValue(), Tris, posT, m_hash, h);
		cuSynchronize();
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::queryNeighborDynamic(NeighborList<int>& nbrList, DeviceArray<Coord>& pos, Real h)
	{
		if (pos.size() <= 0)
		{
			return;
		}

		DeviceArray<int>& nbrNum = nbrList.getIndex();
		if (nbrNum.size() != pos.size())
			nbrList.resize(pos.size());

		queryNeighborSize(nbrNum, pos, h);

		int sum = m_reduce.accumulate(nbrNum.getDataPtr(), nbrNum.size());

		m_scan.exclusive(nbrNum, true);
		cuSynchronize();


		if (sum > 0)
		{
			DeviceArray<int>& elements = nbrList.getElements();
			elements.resize(sum);

			uint pDims = cudaGridSize(pos.size(), BLOCK_SIZE);
			K_GetNeighborElements << <pDims, BLOCK_SIZE >> > (nbrList, pos, this->inPosition()->getValue(), m_hash, h);
			cuSynchronize();
		}
	}
	template<typename TDataType>
	void NeighborQuery<TDataType>::queryNeighborTriDynamic(NeighborList<int>& nbrList, DeviceArray<Coord>& pos, DeviceArray<Coord>& posT, DeviceArray<Triangle>& Tris, Real h)
	{
		if (pos.size() <= 0)
		{
			return;
		}

		DeviceArray<int>& nbrNum = nbrList.getIndex();
		if (nbrNum.size() != pos.size())
			nbrList.resize(pos.size());

		queryNeighborSizeTri(nbrNum, pos, Tris, posT, h);

		int sum = m_reduce.accumulate(nbrNum.getDataPtr(), nbrNum.size());

		m_scan.exclusive(nbrNum, true);
		cuSynchronize();


		if (sum > 0)
		{
			DeviceArray<int>& elements = nbrList.getElements();
			elements.resize(sum);
			elements.reset();

			uint pDims = cudaGridSize(pos.size(), BLOCK_SIZE);
			K_GetNeighborElementsTri << <pDims, BLOCK_SIZE >> > (nbrList, pos, this->inPosition()->getValue(), Tris, posT, m_hash, h);
			cuSynchronize();
		}


	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_ComputeNeighborFixed(
		NeighborList<int> neighbors, 
		DeviceArray<Coord> position_new,
		DeviceArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h,
		int* heapIDs,
		Real* heapDistance)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId > position_new.size()) return;

		int nbrLimit = neighbors.getNeighborLimit();

		int* ids(heapIDs + pId * nbrLimit);// = new int[nbrLimit];
		Real* distance(heapDistance + pId * nbrLimit);// = new Real[nbrLimit];

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset1[c][0], gId3.y + offset1[c][1], gId3.z + offset1[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);// min(hash.getCounter(cId), hash.npMax);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					float d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						if (counter < nbrLimit)
						{
							ids[counter] = nbId;
							distance[counter] = d_ij;
							counter++;
						}
						else
						{
							int maxId = 0;
							float maxDist = distance[0];
							for (int ne = 1; ne < nbrLimit; ne++)
							{
								if (maxDist < distance[ne])
								{
									maxDist = distance[ne];
									maxId = ne;
								}
							}
							if (d_ij < distance[maxId])
							{
								distance[maxId] = d_ij;
								ids[maxId] = nbId;
							}
						}
					}
				}
			}
		}

		neighbors.setNeighborSize(pId, counter);

		int bId;
		for (bId = 0; bId < counter; bId++)
		{
			neighbors.setElement(pId, bId, ids[bId]);
		}
	}

	template<typename TDataType>
	void NeighborQuery<TDataType>::queryNeighborFixed(NeighborList<int>& nbrList, DeviceArray<Coord>& pos, Real h)
	{
		int num = pos.size();
		int* ids;
		Real* distance;
		cuSafeCall(hipMalloc((void**)&ids, num * sizeof(int) * nbrList.getNeighborLimit()));
		cuSafeCall(hipMalloc((void**)&distance, num * sizeof(int) * nbrList.getNeighborLimit()));

		uint pDims = cudaGridSize(num, BLOCK_SIZE);
		K_ComputeNeighborFixed << <pDims, BLOCK_SIZE >> > (
			nbrList, 
			pos, 
			this->inPosition()->getValue(), 
			m_hash, 
			h, 
			ids, 
			distance);
		cuSynchronize();

		cuSafeCall(hipFree(ids));
		cuSafeCall(hipFree(distance));
	}
}