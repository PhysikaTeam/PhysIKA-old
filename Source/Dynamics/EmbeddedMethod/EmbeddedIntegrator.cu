#include "hip/hip_runtime.h"
/**
 * @author     : ZHAO CHONGYAO (cyzhao@zju.edu.cn)
 * @date       : 2021-05-30
 * @description: A implicit integrator source for physika library
 * @version    : 2.2.1
 */
#include <iostream>
#include <hip/hip_runtime.h>

#include "Framework/Framework/FieldArray.h"
#include "Framework/Framework/FieldVar.h"
#include "Framework/Framework/Node.h"
#include "Framework/Framework/SceneGraph.h"
#include "FiniteElementMethod/Problem/IntegratedProblem/FEMProblemIntegratedEmbeddedElasFemProblem.h"
#include "FiniteElementMethod/Problem/IntegratedProblem/FEMProblemIntegratedFastMsProblem.h"
#include "FiniteElementMethod/Common/FEMCommonDataStream.h"
#include "FiniteElementMethod/Solver/FEMSolverLists.h"
#include "EmbeddedIntegrator.h"

using namespace std;
namespace PhysIKA {
template <typename TDataType>
EmbeddedIntegrator<TDataType>::EmbeddedIntegrator()
    : NumericalIntegrator()
{
    /*	attachField(&inPosition, "position", "Storing the particle positions!", false);
		attachField(&inVelocity, "velocity", "Storing the particle velocities!", false);
		attachField(&inForceDensity, "force", "Particle forces", false);*/
}

template <typename TDataType>
void EmbeddedIntegrator<TDataType>::begin()
{
    Function1Pt::copy(m_prePosition, this->inPosition()->getValue());
    Function1Pt::copy(m_preVelocity, this->inVelocity()->getValue());
    this->inForceDensity()->getReference()->reset();

    //========see velo=====//
    // static size_t  cnt = 0;

    // HostArray<Coord> m_velo_host(this->inVelocity()->getElementCount());
    // Function1Pt::copy(m_velo_host, this->inVelocity()->getValue());
    // std::cout << m_velo_host[0][1] << std::endl;
    // ++cnt;
    // if(cnt == 2)
    //   exit(EXIT_FAILURE);

    //========see velo=====//

    const size_t num = this->inPosition()->getElementCount();

    m_position_host.resize(num);
    Function1Pt::copy(m_position_host, this->inPosition()->getValue());
    pos_.resize(num * 3);
#pragma omp parallel for
    for (size_t i = 0; i < num; ++i)
        for (size_t j = 0; j < 3; ++j)
            pos_[i * 3 + j] = m_position_host[i][j];

    m_velocity_host.resize(num);
    Function1Pt::copy(m_velocity_host, this->inVelocity()->getValue());
    vel_.resize(num * 3);
#pragma omp parallel for
    for (size_t i = 0; i < num; ++i)
        for (size_t j = 0; j < 3; ++j)
            vel_[i * 3 + j] = m_velocity_host[i][j];

    epb_fac_->update_problem(&pos_[0]);
}

template <typename TDataType>
void EmbeddedIntegrator<TDataType>::end()
{
}

template <typename TDataType>
bool EmbeddedIntegrator<TDataType>::initializeImpl()
{
    if (!isAllFieldsReady())
    {
        std::cout << "Exception: " << std::string("DensitySummation's fields are not fully initialized!") << "\n";
        return false;
    }

    int num = this->inPosition()->getElementCount();

    m_prePosition.resize(num);
    m_preVelocity.resize(num);

    return true;
}

template <typename Real, typename Coord>
__global__ void K_UpdateVelocity(
    DeviceArray<Coord> vel,
    DeviceArray<Coord> forceDensity,
    Coord              gravity,
    Real               dt)
{
    int pId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (pId >= forceDensity.size())
        return;

    vel[pId] += dt * (forceDensity[pId] + gravity);
}

template <typename Real, typename Coord>
__global__ void K_UpdateVelocity(
    DeviceArray<Coord> vel,
    DeviceArray<Coord> pos,
    DeviceArray<Coord> pre_pos,
    Real               dt)
{
    int pId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (pId >= pos.size())
        return;

    vel[pId] = (pos[pId] - pre_pos[pId]) / dt;
}

template <typename Real, typename Coord>
__global__ void K_UpdateVelocity(
    DeviceArray<Coord> vel,
    DeviceArray<Coord> force,
    DeviceArray<Real>  mass,
    Real               dt)
{
    int pId = threadIdx.x + (blockIdx.x * blockDim.x);
    if (pId >= force.size())
        return;

    vel[pId] += dt * force[pId] / mass[pId];
}

template <typename TDataType>
bool EmbeddedIntegrator<TDataType>::updateVelocity()
{
    Real  dt    = getParent()->getDt();
    cuint pDims = cudaGridSize(this->inPosition()->getReference()->size(), BLOCK_SIZE);
    K_UpdateVelocity<<<pDims, BLOCK_SIZE>>>(
        this->inVelocity()->getValue(),
        this->inPosition()->getValue(),
        m_prePosition,
        dt);

    return true;
}

template <typename TDataType>
bool EmbeddedIntegrator<TDataType>::updatePosition()
{
    auto pb = epb_fac_->build_problem();

    if (solver_type_ == "explicit")
    {
        for (int i = 0; i < 100; ++i)
        {
            pb      = epb_fac_->build_problem();
            solver_ = newton_with_pcg_and_embedded<Real, 3>(pb, pt_, dat_str_, pos_.size(), epb_fac_->get_embedded_interpolate(), semi_implicit_);
            solver_->solve(&pos_[0]);
            epb_fac_->update_problem(&pos_[0]);
        }
    }
    else if (solver_type_ == "fast_ms")
    {
        solver_ = newton_with_fast_ms_and_embedded<Real, 3>(pb, pt_, dat_str_, pos_.size(), epb_fac_->get_embedded_interpolate(), semi_implicit_, fast_ms_solver_info_);
    }
    else
        solver_ = newton_with_pcg_and_embedded<Real, 3>(pb, pt_, dat_str_, pos_.size(), epb_fac_->get_embedded_interpolate());

    solver_->solve(&pos_[0]);

    const size_t num = this->inPosition()->getElementCount();
#pragma omp parallel for
    for (size_t i = 0; i < num; ++i)
        for (size_t j = 0; j < 3; ++j)
            m_position_host[i][j] = pos_[i * 3 + j];
    Function1Pt::copy(this->inPosition()->getValue(), m_position_host);

    return true;
}

template <typename TDataType>
bool EmbeddedIntegrator<TDataType>::integrate()
{
    updatePosition();
    updateVelocity();

    return true;
}

template <typename TDataType>
void EmbeddedIntegrator<TDataType>::bind_problem(const std::shared_ptr<embedded_problem_builder<Real, 3>>& epb_fac, const boost::property_tree::ptree& pt)
{
    solver_type_ = pt.get<string>("solver_type", "implicit");
    if (solver_type_ == "explicit")
    {
        embedded_interp_ = epb_fac->get_embedded_interpolate();
        semi_implicit_   = epb_fac->get_semi_implicit();

        Eigen::Matrix<Real, -1, -1>            nods        = epb_fac->get_nods();
        Eigen::Matrix<Real, -1, -1>            nods_coarse = nods * embedded_interp_->get_fine_to_coarse_coefficient();
        Eigen::Map<Eigen::Matrix<Real, -1, 1>> init_nods_coarse(nods_coarse.data(), nods_coarse.size());
        semi_implicit_->update_status(init_nods_coarse);
    }
    else if (solver_type_ == "fast_ms")
    {
        shared_ptr<fast_ms_builder<Real>> fast_ms_epb_fac = dynamic_pointer_cast<fast_ms_builder<Real>>(epb_fac);
        fast_ms_solver_info_                              = fast_ms_epb_fac->get_fast_ms_solver_info();
    }

    auto pb  = epb_fac->build_problem();
    dat_str_ = make_shared<dat_str_core<Real, 3>>(pb->Nx() / 3, pt.get<bool>("hes_is_const", false));
    exit_if(compute_hes_pattern(pb->energy_, dat_str_), "compute hes pattern fail");
    epb_fac_ = epb_fac;
    pt_      = pt;
}

}  // namespace PhysIKA
