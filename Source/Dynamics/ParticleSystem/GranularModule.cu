#include "hip/hip_runtime.h"
#include "GranularModule.h"
#include "SummationDensity.h"

namespace PhysIKA
{
	template<typename TDataType>
	GranularModule<TDataType>::GranularModule()
		: ElastoplasticityModule<TDataType>()
	{
	}


	template<typename TDataType>
	bool GranularModule<TDataType>::initializeImpl()
	{
		m_densitySum = std::make_shared<SummationDensity<TDataType>>();

		this->inHorizon()->connect(m_densitySum->varSmoothingLength());
		this->inPosition()->connect(m_densitySum->inPosition());
		this->inNeighborhood()->connect(m_densitySum->inNeighborIndex());

		m_densitySum->initialize();

		return ElastoplasticityModule<TDataType>::initializeImpl();
	}


	__device__ Real Hardening(Real rho, Real restRho)
	{
		if (rho >= restRho)
		{
			float ratio = rho / restRho;
			//ratio = ratio > 1.1f ? 1.1f : ratio;
			return pow(Real(M_E), Real(ratio - 1.0f));
		}
		else
		{
			return Real(0);
		};
	}

	template <typename Real>
	__global__ void PM_ComputeStiffness(
		DeviceArray<Real> stiffiness,
		DeviceArray<Real> density)
	{
		int i = threadIdx.x + (blockIdx.x * blockDim.x);
		if (i >= stiffiness.size()) return;

		stiffiness[i] = Hardening(density[i], Real(1000));
	}

	template<typename TDataType>
	void GranularModule<TDataType>::computeMaterialStiffness()
	{
		int num = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(num, BLOCK_SIZE);

		m_densitySum->compute();

		PM_ComputeStiffness << <pDims, BLOCK_SIZE >> > (
			this->m_bulkCoefs,
			m_densitySum->outDensity()->getValue());
		cuSynchronize();
	}
}