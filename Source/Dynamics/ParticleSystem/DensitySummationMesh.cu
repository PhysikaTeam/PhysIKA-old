#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "DensitySummationMesh.h"
#include "Framework/Framework/MechanicalState.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "Kernel.h"
#include "Framework/Topology/Primitive3D.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(DensitySummationMesh, TDataType)


	__device__ inline float kernWeightMeshPBD(const float r, const float h)
	{
		const Real q = r / h;
		if (q > 1.0f) return 0.0f;
		else {
			const Real d = 1.0 - q;
			const Real hh = h * h;
			return 15.0f / ((Real)M_PI * hh * h) *
				(
				   1.0f / 3.0f * (hh * h - r * r * r)
				 - 3.0f / 4.0f / h * (hh * hh - r * r * r * r)
				 + 3.0f / 5.0f / hh * (hh * hh * h - r * r * r * r * r)
				 - 1.0f / 6.0f / hh / h * (hh * hh * hh - r * r * r * r * r * r)
					);
		}
		/*
		const Real q = r / h;
			if (q > 1.0f) return 0.0f;
			else {
				const Real d = 1.0 - q;
				const Real hh = h*h;
				return 15.0f / ((Real)M_PI * hh * h) * d * d * d * this->m_scale;
			}
		*/
	}

	template<typename Real, typename Coord>
	__global__ void K_ComputeDensityMesh(
		DeviceArray<Real> rhoArr,
		DeviceArray<Coord> posArr,
		DeviceArray<TopologyModule::Triangle> Tri,
		DeviceArray<Coord> positionTri,
		NeighborList<int> neighbors,
		NeighborList<int> neighborsTri,
		Real smoothingLength,
		Real mass,
		Real sampling_distance,
		int use_mesh,
		int use_ghost,
		int Start
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;
		//if (pId >= Start)  return;

		SpikyKernel<Real> kern;
		Real r;
		Real rho_i = Real(0);
		Real rho_tmp;
		Coord pos_i = posArr[pId];

		int nbSizeTri = neighborsTri.getNeighborSize(pId);
		int nbSize = neighbors.getNeighborSize(pId);

		if (use_mesh && pId < Start)
			for (int ne = 0; ne < nbSizeTri; ne++)
			{
				int j = neighborsTri.getElement(pId, ne);
				if (j >= 0) continue;
				j *= -1;
				j--;
				
				Triangle3D t3d(positionTri[Tri[j][0]], positionTri[Tri[j][1]], positionTri[Tri[j][2]]);
				Plane3D PL(positionTri[Tri[j][0]], t3d.normal());
				Point3D p3d(pos_i);
				//Point3D nearest_pt = p3d.project(t3d);
				Point3D nearest_pt = p3d.project(PL);
				Real r = (nearest_pt.origin - pos_i).norm();
				//r = max((r - sampling_distance / 2.0), 0.0);

				Real AreaSum = p3d.areaTriangle(t3d, smoothingLength);
				Real MinDistance = abs(p3d.distance(t3d));
				Coord Min_Pt = (p3d.project(t3d)).origin - pos_i;
				Coord Min_Pos = p3d.project(t3d).origin;
				if (ne < nbSizeTri - 1 && neighborsTri.getElement(pId, ne + 1) < 0)
				{
					int jn;
					do
					{
						jn = neighborsTri.getElement(pId, ne + 1);
						if (jn > 0) break;
						jn *= -1; jn--;

						Triangle3D t3d_n(positionTri[Tri[jn][0]], positionTri[Tri[jn][1]], positionTri[Tri[jn][2]]);
						if ((t3d.normal().cross(t3d_n.normal())).norm() > EPSILON) break;

						AreaSum += p3d.areaTriangle(t3d_n, smoothingLength);

						if (abs(p3d.distance(t3d_n)) < abs(MinDistance))
						{
							MinDistance = (p3d.distance(t3d_n));
							Min_Pt = (p3d.project(t3d_n)).origin - pos_i;
							Min_Pos = (p3d.project(t3d_n)).origin;
						}
						//printf("%d %d\n", j, jn);
						ne++;
					} while (ne < nbSizeTri - 1);
				}
				Min_Pt /= (-Min_Pt.norm());

				float d = p3d.distance(PL);
				d = abs(d);
				if (smoothingLength - d > EPSILON&& smoothingLength * smoothingLength - d * d > EPSILON&& d > EPSILON)
				{

					Real a_ij =
						kernWeightMeshPBD(r, smoothingLength)
						* 2.0 * (M_PI) * (1 - d / smoothingLength)
						* AreaSum// p3d.areaTriangle(t3d, smoothingLength)
						/ ((M_PI) * (smoothingLength * smoothingLength - d * d))
						* t3d.normal().dot(Min_Pt)/t3d.normal().norm() /// (p3d.project(t3d).origin - p3d.origin).norm()
						/ 
						(sampling_distance * sampling_distance * sampling_distance) * kern.m_scale;
					rho_i += 1.0 * mass * a_ij;

				//	printf("%.3lf %.3lf %.3lf\n", r, a_ij, kern.Weight(r, smoothingLength));
					
				}
			}
		if (rho_i < 0) rho_i *= -1;
		rho_tmp = rho_i;
		

		bool tmp = false;
		
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors.getElement(pId, ne);

			if (j >= Start && (!use_ghost)) continue;
			//if (j >= Start) continue;
			if (j >= Start) tmp = true;

			if (j >= Start && pId % 10000 == 0 && pId < Start)
			{
			//	printf("%d :%.3lf %.3lf %.3lf\n",pId , posArr[j][0], posArr[j][1], posArr[j][2]);
			}

			r = (pos_i - posArr[j]).norm();
			rho_i += mass*kern.Weight(r, smoothingLength);
			if(j < Start && pId < Start) rho_tmp += mass * kern.Weight(r, smoothingLength);
		}
		//if (tmp && pId < Start)
			//printf("%.3lf %.3lf %d\n", rho_i, rho_tmp, nbSize);
		rhoArr[pId] = rho_i;

	}

	template<typename TDataType>
	DensitySummationMesh<TDataType>::DensitySummationMesh()
		: ComputeModule()
		, m_factor(Real(1))
	{
		m_mass.setValue(Real(1));
		m_restDensity.setValue(Real(1000));
		m_smoothingLength.setValue(Real(0.011));

		attachField(&m_mass, "mass", "particle mass", false);
		attachField(&m_restDensity, "rest_density", "Reference density", false);
		attachField(&m_smoothingLength, "smoothing_length", "The smoothing length in SPH!", false);

		attachField(&m_position, "position", "Storing the particle positions!", false);
		attachField(&m_density, "density", "Storing the particle densities!", false);
		attachField(&m_neighborhood, "neighborhood", "Storing neighboring particles' ids!", false);
	}

	template<typename TDataType>
	void DensitySummationMesh<TDataType>::compute()
	{
		printf("%d\n", m_position.getElementCount());
		if (!m_position.isEmpty() && m_density.getElementCount() != m_position.getElementCount())
		{
			printf("%d\n", m_position.getElementCount());
			m_density.setElementCount(m_position.getElementCount());
			
		}
		compute(
			m_density.getValue(),
			m_position.getValue(),
			Tri.getValue(),
			TriPoint.getValue(),
			m_neighborhood.getValue(),
			m_neighborhoodTri.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue(),
			sampling_distance.getValue(),
			use_mesh.getValue(),
			use_ghost.getValue(),
			Start.getValue());
	}


	template<typename TDataType>
	void DensitySummationMesh<TDataType>::compute(DeviceArray<Real>& rho)
	{

		compute(
			rho,
			m_position.getValue(),
			Tri.getValue(),
			TriPoint.getValue(),
			m_neighborhood.getValue(),
			m_neighborhoodTri.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue(),
			sampling_distance.getValue(),
			use_mesh.getValue(),
			use_ghost.getValue(),
			Start.getValue());
	}

	template<typename TDataType>
	void DensitySummationMesh<TDataType>::compute(
		DeviceArray<Real>& rho, 
		DeviceArray<Coord>& pos,
		DeviceArray<TopologyModule::Triangle>& Tri,
		DeviceArray<Coord>& positionTri,
		NeighborList<int>& neighbors, 
		NeighborList<int>& neighborsTri,
		Real smoothingLength,
		Real mass,
		Real sampling_distance,
		int use_mesh,
		int use_ghost,
		int Start)
	{
		cuint pDims = cudaGridSize(rho.size(), BLOCK_SIZE);
		K_ComputeDensityMesh <Real, Coord> << <pDims, BLOCK_SIZE >> > (
			rho, 
			pos,
			Tri, 
			positionTri, 
			neighbors, 
			neighborsTri, 
			smoothingLength, 
			m_factor*mass, 
			sampling_distance, 
			use_mesh,
			use_ghost,
			Start
			);
	}

	template<typename TDataType>
	bool DensitySummationMesh<TDataType>::initializeImpl()
	{
		if (m_position.isEmpty())
		{
			
			Real d = sampling_distance.getValue();
			Real H = m_smoothingLength.getValue();
			Real rho_0 = m_restDensity.getValue();
			Real m = m_mass.getValue();

			SpikyKernel<Real> kern;

			Real rho_e(0);
			int half_res = H / d + 1;
			for (int i = -half_res; i <= half_res; i++)
				for (int j = -half_res; j <= half_res; j++)
					for (int k = -half_res; k <= half_res; k++)
					{
						Real x = i * d;
						Real y = j * d;
						Real z = k * d;
						Real r = sqrt(x * x + y * y + z * z);
						rho_e += m * kern.Weight(r, H);
					}

			m_factor = rho_0 / rho_e;
			//printf("@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@ %.18lf from density\n", m_factor);
			return true;
		}
		if (!m_position.isEmpty() && m_density.isEmpty())
		{
			m_density.setElementCount(m_position.getElementCount());
		}

		if (!isAllFieldsReady())
		{
			std::cout << "Exception: " << std::string("DensitySummationMesh's fields are not fully initialized!") << "\n";
			return false;
		}

		printf("%d\n", m_position.getValue().size());

		compute(
			m_density.getValue(),
			m_position.getValue(),
			Tri.getValue(),
			TriPoint.getValue(),
			m_neighborhood.getValue(),
			m_neighborhoodTri.getValue(),
			m_smoothingLength.getValue(),
			m_mass.getValue(),
			sampling_distance.getValue(),
			use_mesh.getValue(),
			0,
			Start.getValue());

		auto rho = m_density.getReference();

		Reduction<Real>* pReduce = Reduction<Real>::Create(rho->size());

		Real maxRho = pReduce->maximum(rho->getDataPtr(), rho->size());

		m_factor = m_restDensity.getValue() / maxRho;
		
		delete pReduce;

		return true;
	}
}