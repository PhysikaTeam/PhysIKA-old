#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ElasticityModule.h"
#include "Framework/Framework/Node.h"
#include "Core/Algorithm/MatrixFunc.h"
#include "Core/Utility.h"
#include "Kernel.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(ElasticityModule, TDataType)

	template<typename Real>
	__device__ Real D_Weight(Real r, Real h)
	{
		SmoothKernel<Real> kernSmooth;
		Real q = r / h;
		return q*q*kernSmooth.Weight(r, h);
	}


	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void EM_PrecomputeShape(
		DeviceArray<Matrix> invK,
		NeighborList<NPair> restShapes,
		Real smoothingLength)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= invK.size()) return;

		CorrectedKernel<float> g_weightKernel;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);

		Real total_weight = 0.0f;
		Matrix mat_i = Matrix(0);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			Real r = (rest_i - rest_j).norm();

			if (r > EPSILON)
			{
				Real weight = g_weightKernel.Weight(r, smoothingLength);
				Coord q = (rest_j - rest_i) / smoothingLength*sqrt(weight);

				mat_i(0, 0) += q[0] * q[0]; mat_i(0, 1) += q[0] * q[1]; mat_i(0, 2) += q[0] * q[2];
				mat_i(1, 0) += q[1] * q[0]; mat_i(1, 1) += q[1] * q[1]; mat_i(1, 2) += q[1] * q[2];
				mat_i(2, 0) += q[2] * q[0]; mat_i(2, 1) += q[2] * q[1]; mat_i(2, 2) += q[2] * q[2];

				total_weight += weight;
			}
		}

		if (total_weight > EPSILON)
		{
			mat_i *= (1.0f / total_weight);
		}

		Matrix R(0), U(0), D(0), V(0);

// 		if (pId == 0)
// 		{
// 			printf("EM_PrecomputeShape**************************************");
// 
// 			printf("K: \n %f %f %f \n %f %f %f \n %f %f %f \n\n\n",
// 				mat_i(0, 0), mat_i(0, 1), mat_i(0, 2),
// 				mat_i(1, 0), mat_i(1, 1), mat_i(1, 2),
// 				mat_i(2, 0), mat_i(2, 1), mat_i(2, 2));
// 		}

		polarDecomposition(mat_i, R, U, D, V);

		Real threshold = 0.0001f*smoothingLength;
		D(0, 0) = D(0, 0) > threshold ? 1.0 / D(0, 0) : 1.0;
		D(1, 1) = D(1, 1) > threshold ? 1.0 / D(1, 1) : 1.0;
		D(2, 2) = D(2, 2) > threshold ? 1.0 / D(2, 2) : 1.0;

		mat_i = V*D*U.transpose();

// 		polarDecomposition(mat_i, R, U, D);
// 
// 		Real threshold = 0.0001f*smoothingLength;
// 		D(0, 0) = D(0, 0) > threshold ? 1.0 / D(0, 0) : 1.0;
// 		D(1, 1) = D(1, 1) > threshold ? 1.0 / D(1, 1) : 1.0;
// 		D(2, 2) = D(2, 2) > threshold ? 1.0 / D(2, 2) : 1.0;
// 
// 		mat_i = R.transpose()*U*D*U.transpose();

// 		printf("Mat: \n %f %f %f \n %f %f %f \n %f %f %f \n	R: \n %f %f %f \n %f %f %f \n %f %f %f \n D: \n %f %f %f \n %f %f %f \n %f %f %f \n U :\n %f %f %f \n %f %f %f \n %f %f %f \n Determinant: %f \n\n",
// 			mat_i(0, 0), mat_i(0, 1), mat_i(0, 2),
// 			mat_i(1, 0), mat_i(1, 1), mat_i(1, 2),
// 			mat_i(2, 0), mat_i(2, 1), mat_i(2, 2),
// 			R(0, 0), R(0, 1), R(0, 2),
// 			R(1, 0), R(1, 1), R(1, 2),
// 			R(2, 0), R(2, 1), R(2, 2),
// 			D(0, 0), D(0, 1), D(0, 2),
// 			D(1, 0), D(1, 1), D(1, 2),
// 			D(2, 0), D(2, 1), D(2, 2),
// 			U(0, 0), U(0, 1), U(0, 2),
// 			U(1, 0), U(1, 1), U(1, 2),
// 			U(2, 0), U(2, 1), U(2, 2),
// 			R.determinant());
// 		printf("Mat: \n %f %f %f \n %f %f %f \n %f %f %f \n	U :\n %f %f %f \n %f %f %f \n %f %f %f \n Determinant: %f \n\n",
// 			mat_i(0, 0), mat_i(0, 1), mat_i(0, 2),
// 			mat_i(1, 0), mat_i(1, 1), mat_i(1, 2),
// 			mat_i(2, 0), mat_i(2, 1), mat_i(2, 2),
// 			U(0, 0), U(0, 1), U(0, 2),
// 			U(1, 0), U(1, 1), U(1, 2),
// 			U(2, 0), U(2, 1), U(2, 2),
// 			R.determinant());

		invK[pId] = mat_i;
	}

	__device__ float EM_GetStiffness(int r)
	{
		return 10.0f;
	}

	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void EM_EnforceElasticity(
		DeviceArray<Coord> delta_position,
		DeviceArray<Real> weights,
		DeviceArray<Real> bulkCoefs,
		DeviceArray<Matrix> invK,
		DeviceArray<Coord> position,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real mu,
		Real lambda)
	{

		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		CorrectedKernel<Real> g_weightKernel;

		NPair np_i = restShapes.getElement(pId, 0);
		Coord rest_i = np_i.pos;
		int size_i = restShapes.getNeighborSize(pId);

		Coord cur_pos_i = position[pId];

		Coord accPos = Coord(0);
		Real accA = Real(0);
		Real bulk_i = bulkCoefs[pId];


		Real total_weight = 0.0f;
		Matrix deform_i = Matrix(0.0f);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.index;

			Real r = (rest_j - rest_i).norm();

			if (r > EPSILON)
			{
				Real weight = g_weightKernel.Weight(r, horizon);

				Coord p = (position[j] - position[pId]) / horizon;
				Coord q = (rest_j - rest_i) / horizon*weight;

				deform_i(0, 0) += p[0] * q[0]; deform_i(0, 1) += p[0] * q[1]; deform_i(0, 2) += p[0] * q[2];
				deform_i(1, 0) += p[1] * q[0]; deform_i(1, 1) += p[1] * q[1]; deform_i(1, 2) += p[1] * q[2];
				deform_i(2, 0) += p[2] * q[0]; deform_i(2, 1) += p[2] * q[1]; deform_i(2, 2) += p[2] * q[2];
				total_weight += weight;
			}
		}


		if (total_weight > EPSILON)
		{
			deform_i *= (1.0f / total_weight);
			deform_i = deform_i * invK[pId];
		}
		else
		{
			total_weight = 1.0f;
		}

		//Check whether the reference shape is inverted, if yes, simply set K^{-1} to be an identity matrix
		//Note other solutions are possible.
		if ((deform_i.determinant()) < -0.001f)
		{
			deform_i = Matrix::identityMatrix();
		}


		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_j = np_j.pos;
			int j = np_j.index;

			Coord cur_pos_j = position[j];
			Real r = (rest_j - rest_i).norm();

			if (r > 0.01f*horizon)
			{
				Real weight = g_weightKernel.WeightRR(r, horizon);

				Coord rest_dir_ij = deform_i*(rest_i - rest_j);
				Coord cur_dir_ij = cur_pos_i - cur_pos_j;

				cur_dir_ij = cur_dir_ij.norm() > EPSILON ? cur_dir_ij.normalize() : Coord(0);
				rest_dir_ij = rest_dir_ij.norm() > EPSILON ? rest_dir_ij.normalize() : Coord(0, 0, 0);

				Real mu_ij = mu*bulk_i* g_weightKernel.WeightRR(r, horizon);
				Coord mu_pos_ij = position[j] + r*rest_dir_ij;
				Coord mu_pos_ji = position[pId] - r*rest_dir_ij;

				Real lambda_ij = lambda*bulk_i*g_weightKernel.WeightRR(r, horizon);
				Coord lambda_pos_ij = position[j] + r*cur_dir_ij;
				Coord lambda_pos_ji = position[pId] - r*cur_dir_ij;

				Coord delta_pos_ij = mu_ij*mu_pos_ij + lambda_ij*lambda_pos_ij;
				Real delta_weight_ij = mu_ij + lambda_ij;

				Coord delta_pos_ji = mu_ij*mu_pos_ji + lambda_ij*lambda_pos_ji;

				accA += delta_weight_ij;
				accPos += delta_pos_ij;


				atomicAdd(&weights[j], delta_weight_ij);
				atomicAdd(&delta_position[j][0], delta_pos_ji[0]);
				atomicAdd(&delta_position[j][1], delta_pos_ji[1]);
				atomicAdd(&delta_position[j][2], delta_pos_ji[2]);
			}
		}

		atomicAdd(&weights[pId], accA);
		atomicAdd(&delta_position[pId][0], accPos[0]);
		atomicAdd(&delta_position[pId][1], accPos[1]);
		atomicAdd(&delta_position[pId][2], accPos[2]);
	}


	template <typename Real, typename Coord, typename NPair>
	__global__ void K_UpdatePosition(
		DeviceArray<Coord> position,
		DeviceArray<Coord> delta_position,
		NeighborList<NPair> restShapes,
		Real horizon)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		CorrectedKernel<float> g_weightKernel;
		Coord delta_pos_i = delta_position[pId];

		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;

		Coord new_delta_pos_i = Coord(0);
		int size_i = restShapes.getNeighborSize(pId);
		for (int ne = 0; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			Coord rest_pos_j = np_j.pos;
			int j = np_j.index;
			Real r = (rest_pos_j - rest_pos_i).norm();

			Coord delta_pos_j = delta_position[j];

			new_delta_pos_i += 0.1*(delta_pos_i)*g_weightKernel.Weight(r, horizon);
		}



//		position[pId] += delta_pos_i;
		position[pId] += delta_position[pId];

	}

	template <typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DeviceArray<Coord> position,
		DeviceArray<Coord> old_position,
		DeviceArray<Coord> delta_position,
		DeviceArray<Real> delta_weights)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		position[pId] = (old_position[pId] + delta_position[pId]) / (1.0+delta_weights[pId]);
	}


	template <typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> velArr,
		DeviceArray<Coord> prePos,
		DeviceArray<Coord> curPos,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velArr.size()) return;

		velArr[pId] += (curPos[pId] - prePos[pId]) / dt;
	}

	template<typename TDataType>
	ElasticityModule<TDataType>::ElasticityModule()
		: ConstraintModule()
	{
//		this->attachField(&m_horizon, "horizon", "Supporting radius!", false);
//		this->attachField(&m_distance, "distance", "The sampling distance!", false);
		this->attachField(&m_mu, "mu", "Material stiffness!", false);
		this->attachField(&m_lambda, "lambda", "Material stiffness!", false);
		this->attachField(&m_iterNum, "Iterations", "Iteration Number", false);

//		this->attachField(&m_position, "position", "Storing the particle positions!", false);
//		this->attachField(&m_velocity, "velocity", "Storing the particle velocities!", false);
//		this->attachField(&m_neighborhood, "neighborhood", "Storing neighboring particles' ids!", false);

//		this->attachField(&testing, "testing", "For testing", false);
//		this->attachField(&TetOut, "TetOut", "For testing", false);

		this->inHorizon()->setValue(0.0125);
 		m_mu.setValue(0.05);
 		m_lambda.setValue(0.1);
		m_iterNum.setValue(10);
	}


	template<typename TDataType>
	ElasticityModule<TDataType>::~ElasticityModule()
	{
		m_weights.release();
		m_displacement.release();
		m_invK.release();
		m_F.release();
		m_position_old.release();
	}

	template<typename TDataType>
	void ElasticityModule<TDataType>::enforceElasticity()
	{
		int num = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(num, BLOCK_SIZE);

		m_displacement.reset();
		m_weights.reset();

		EM_EnforceElasticity << <pDims, BLOCK_SIZE >> > (
			m_displacement,
			m_weights,
			m_bulkCoefs,
			m_invK,
			this->inPosition()->getValue(),
			m_restShape.getValue(),
			this->inHorizon()->getValue(),
			m_mu.getValue(),
			m_lambda.getValue());
		cuSynchronize();

		K_UpdatePosition << <pDims, BLOCK_SIZE >> > (
			this->inPosition()->getValue(),
			m_position_old,
			m_displacement,
			m_weights);
		cuSynchronize();
	}

	template<typename Real>
	__global__ void EM_InitBulkStiffness(DeviceArray<Real> stiffness)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= stiffness.size()) return;

		stiffness[pId] = Real(1);
	}

	template<typename TDataType>
	void ElasticityModule<TDataType>::computeMaterialStiffness()
	{
		int num = this->inPosition()->getElementCount();

		uint pDims = cudaGridSize(num, BLOCK_SIZE);
		EM_InitBulkStiffness << <pDims, BLOCK_SIZE >> > (m_bulkCoefs);
	}


	template<typename TDataType>
	void ElasticityModule<TDataType>::computeInverseK()
	{
		int num = m_restShape.getElementCount();
		uint pDims = cudaGridSize(num, BLOCK_SIZE);

		EM_PrecomputeShape <Real, Coord, Matrix, NPair> << <pDims, BLOCK_SIZE >> > (
			m_invK,
			m_restShape.getValue(),
			this->inHorizon()->getValue());
		cuSynchronize();
	}


	template<typename TDataType>
	void ElasticityModule<TDataType>::solveElasticity()
	{
		//Save new positions
		Function1Pt::copy(m_position_old, this->inPosition()->getValue());

		this->computeInverseK();

		int itor = 0;
		while (itor < m_iterNum.getValue())
		{
			this->enforceElasticity();

			itor++;
		}

		this->updateVelocity();
	}

	template<typename TDataType>
	void ElasticityModule<TDataType>::updateVelocity()
	{
		int num = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(num, BLOCK_SIZE);

		Real dt = this->getParent()->getDt();

		K_UpdateVelocity << <pDims, BLOCK_SIZE >> > (
			this->inVelocity()->getValue(),
			m_position_old,
			this->inPosition()->getValue(),
			dt);
		cuSynchronize();
	}


	template<typename TDataType>
	bool ElasticityModule<TDataType>::constrain()
	{
		this->solveElasticity();

		return true;
	}


	template <typename Coord, typename NPair>
	__global__ void K_UpdateRestShape(
		NeighborList<NPair> shape,
		NeighborList<int> nbr,
		DeviceArray<Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		NPair np;
		int nbSize = nbr.getNeighborSize(pId);
		
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = nbr.getElement(pId, ne);
			np.index = j;
			np.pos = pos[j];
 			if (pId != j)
 			{
// 				if (pId == 4 && j == 5)
// 				{
// 					np.pos += Coord(0.0001, 0, 0);
// 				}
// 
// 				if (pId == 5 && j == 4)
// 				{
// 					np.pos += Coord(-0.0001, 0, 0);
// 				}

 				shape.setElement(pId, ne, np);
			}
			else
			{
				if (ne == 0)
				{
					shape.setElement(pId, ne, np);
				}
				else
				{
					auto ele = shape.getElement(pId, 0);
					shape.setElement(pId, 0, np);
					shape.setElement(pId, ne, ele);
				}
			}
		}
	}

	template<typename TDataType>
	void ElasticityModule<TDataType>::resetRestShape()
	{
		m_restShape.setElementCount(this->inNeighborhood()->getValue().size());
		m_restShape.getValue().getIndex().resize(this->inNeighborhood()->getValue().getIndex().size());

		if (this->inNeighborhood()->getValue().isLimited())
		{
			m_restShape.getValue().setNeighborLimit(this->inNeighborhood()->getValue().getNeighborLimit());
		}
		else
		{
			m_restShape.getValue().getElements().resize(this->inNeighborhood()->getValue().getElements().size());
		}

		Function1Pt::copy(m_restShape.getValue().getIndex(), this->inNeighborhood()->getValue().getIndex());

		uint pDims = cudaGridSize(this->inPosition()->getValue().size(), BLOCK_SIZE);

		K_UpdateRestShape<< <pDims, BLOCK_SIZE >> > (m_restShape.getValue(), this->inNeighborhood()->getValue(), this->inPosition()->getValue());
		cuSynchronize();
	}

	template<typename TDataType>
	bool ElasticityModule<TDataType>::initializeImpl()
	{
		if (this->inHorizon()->isEmpty() || this->inPosition()->isEmpty() || this->inVelocity()->isEmpty() || this->inNeighborhood()->isEmpty())
		{
			std::cout << "Exception: " << std::string("ElasticityModule's fields are not fully initialized!") << "\n";
			return false;
		}

		int num = this->inPosition()->getElementCount();
		
		m_invK.resize(num);
		m_weights.resize(num);
		m_displacement.resize(num);

		m_F.resize(num);
		
		m_position_old.resize(num);
		m_bulkCoefs.resize(num);

		resetRestShape();

		this->computeMaterialStiffness();

		Function1Pt::copy(m_position_old, this->inPosition()->getValue());

		return true;
	}

}