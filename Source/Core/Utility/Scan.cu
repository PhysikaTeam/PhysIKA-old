#include "hip/hip_runtime.h"
#include "Scan.h"
#include <hip/hip_runtime.h>
#include "Function1Pt.h"
#include "cuda_utilities.h"

namespace PhysIKA
{
#define SCAN_SHARED_MEMORY_BANKS 32
#define SCAN_LOG_MEM_BANKS 5
#define SCAN_CONFLICT_FREE_OFFSET(n) ((n) >> SCAN_LOG_MEM_BANKS)

	int SCAN_THREADS_PER_BLOCK = 512;
	int SCAN_ELEMENTS_PER_BLOCK = SCAN_THREADS_PER_BLOCK * 2;

	Scan::Scan()
	{
	}


	Scan::~Scan()
	{
		m_buffer.release();

		for (int i = 0; i < SCAN_LEVEL; i++)
		{
			m_sums[i].release();
			m_incr[i].release();
		}
	}

	void Scan::exclusive(DeviceArray<int>& output, DeviceArray<int>& input, bool bcao)
	{
		assert(input.size() == output.size());

		if (input.size() > SCAN_ELEMENTS_PER_BLOCK) {
			scanLargeDeviceArray(output.getDataPtr(), input.getDataPtr(), input.size(), bcao, 0);
		}
		else {
			scanSmallDeviceArray(output.getDataPtr(), input.getDataPtr(), input.size(), bcao);
		}
	}

	void Scan::exclusive(DeviceArray<int>& data, bool bcao /*= true*/)
	{
		if (m_buffer.size() != data.size())
		{
			m_buffer.resize(data.size());
		}
		
		Function1Pt::copy(m_buffer, data);
		this->exclusive(data, m_buffer, bcao);
	}

	void Scan::exclusive(int* data, int length, bool bcao /*= true*/)
	{
		if (m_buffer.size() != length)
		{
			m_buffer.resize(length);
		}

		hipMemcpy(m_buffer.getDataPtr(), data, length*sizeof(int), hipMemcpyDeviceToDevice);

		this->exclusive(data, m_buffer.getDataPtr(), length, bcao);
	}

	void Scan::exclusive(int* output, int* input, int length, bool bcao /*= true*/)
	{
		if (length > SCAN_ELEMENTS_PER_BLOCK) {
			scanLargeDeviceArray(output, input, length, bcao, 0);
		}
		else {
			scanSmallDeviceArray(output, input, length, bcao);
		}
	}

	__global__ void k_prescan_arbitrary(int *output, int *input, int n, int powerOfTwo)
	{
		extern __shared__ int temp[];// allocated on invocation
		int threadID = threadIdx.x;

		int ai = threadID;
		int bi = threadID + (n / 2);
		int bankOffsetA = SCAN_CONFLICT_FREE_OFFSET(ai);
		int bankOffsetB = SCAN_CONFLICT_FREE_OFFSET(bi);


		if (threadID < n) {
			temp[ai + bankOffsetA] = input[ai];
			temp[bi + bankOffsetB] = input[bi];
		}
		else {
			temp[ai + bankOffsetA] = 0;
			temp[bi + bankOffsetB] = 0;
		}


		int offset = 1;
		for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
		{
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				ai += SCAN_CONFLICT_FREE_OFFSET(ai);
				bi += SCAN_CONFLICT_FREE_OFFSET(bi);

				temp[bi] += temp[ai];
			}
			offset *= 2;
		}

		if (threadID == 0) {
			temp[powerOfTwo - 1 + SCAN_CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
		}

		for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
		{
			offset >>= 1;
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				ai += SCAN_CONFLICT_FREE_OFFSET(ai);
				bi += SCAN_CONFLICT_FREE_OFFSET(bi);

				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		if (threadID < n) {
			output[ai] = temp[ai + bankOffsetA];
			output[bi] = temp[bi + bankOffsetB];
		}
	}

	__global__ void k_prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
		extern __shared__ int temp[];// allocated on invocation
		int threadID = threadIdx.x;

		if (threadID < n) {
			temp[2 * threadID] = input[2 * threadID]; // load input into shared memory
			temp[2 * threadID + 1] = input[2 * threadID + 1];
		}
		else {
			temp[2 * threadID] = 0;
			temp[2 * threadID + 1] = 0;
		}


		int offset = 1;
		for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
		{
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				temp[bi] += temp[ai];
			}
			offset *= 2;
		}

		if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // clear the last element

		for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
		{
			offset >>= 1;
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		if (threadID < n) {
			output[2 * threadID] = temp[2 * threadID]; // write results to device memory
			output[2 * threadID + 1] = temp[2 * threadID + 1];
		}
	}

	__global__ void k_add(int *output, int length, int *n) {
		int blockID = blockIdx.x;
		int threadID = threadIdx.x;
		int blockOffset = blockID * length;

		output[blockOffset + threadID] += n[blockID];
	}

	__global__ void k_add(int *output, int length, int *n1, int *n2) {
		int blockID = blockIdx.x;
		int threadID = threadIdx.x;
		int blockOffset = blockID * length;

		output[blockOffset + threadID] += n1[blockID] + n2[blockID];
	}

	void Scan::scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao, int level)
	{
		int remainder = length % (SCAN_ELEMENTS_PER_BLOCK);
		if (remainder == 0) {
			scanLargeEvenDeviceArray(d_out, d_in, length, bcao, level);
		}
		else {
			// perform a large scan on a compatible multiple of elements
			int lengthMultiple = length - remainder;
			scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao, level);

			// scan the remaining elements and add the (inclusive) last element of the large scan to this
			int *startOfOutputArray = &(d_out[lengthMultiple]);
			scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

			k_add << <1, remainder >> > (startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
			cuSynchronize();
		}
	}

	void Scan::scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao)
	{
		int powerOfTwo = nextPowerOfTwo(length);

		if (bcao) {
			k_prescan_arbitrary << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >> > (d_out, d_in, length, powerOfTwo);
			cuSynchronize();
		}
		else {
			k_prescan_arbitrary_unoptimized << <1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int) >> > (d_out, d_in, length, powerOfTwo);
			cuSynchronize();
		}
	}

	__global__ void k_prescan_large(int *output, int *input, int n, int *sums) {
		extern __shared__ int temp[];

		int blockID = blockIdx.x;
		int threadID = threadIdx.x;
		int blockOffset = blockID * n;

		int ai = threadID;
		int bi = threadID + (n / 2);
		int bankOffsetA = SCAN_CONFLICT_FREE_OFFSET(ai);
		int bankOffsetB = SCAN_CONFLICT_FREE_OFFSET(bi);
		temp[ai + bankOffsetA] = input[blockOffset + ai];
		temp[bi + bankOffsetB] = input[blockOffset + bi];

		int offset = 1;
		for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
		{
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				ai += SCAN_CONFLICT_FREE_OFFSET(ai);
				bi += SCAN_CONFLICT_FREE_OFFSET(bi);

				temp[bi] += temp[ai];
			}
			offset *= 2;
		}
		__syncthreads();


		if (threadID == 0) {
			sums[blockID] = temp[n - 1 + SCAN_CONFLICT_FREE_OFFSET(n - 1)];
			temp[n - 1 + SCAN_CONFLICT_FREE_OFFSET(n - 1)] = 0;
		}

		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
		{
			offset >>= 1;
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				ai += SCAN_CONFLICT_FREE_OFFSET(ai);
				bi += SCAN_CONFLICT_FREE_OFFSET(bi);

				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		output[blockOffset + ai] = temp[ai + bankOffsetA];
		output[blockOffset + bi] = temp[bi + bankOffsetB];
	}

	__global__ void k_prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
		int blockID = blockIdx.x;
		int threadID = threadIdx.x;
		int blockOffset = blockID * n;

		extern __shared__ int temp[];
		temp[2 * threadID] = input[blockOffset + (2 * threadID)];
		temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

		int offset = 1;
		for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
		{
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				temp[bi] += temp[ai];
			}
			offset *= 2;
		}
		__syncthreads();


		if (threadID == 0) {
			sums[blockID] = temp[n - 1];
			temp[n - 1] = 0;
		}

		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
		{
			offset >>= 1;
			__syncthreads();
			if (threadID < d)
			{
				int ai = offset * (2 * threadID + 1) - 1;
				int bi = offset * (2 * threadID + 2) - 1;
				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		output[blockOffset + (2 * threadID)] = temp[2 * threadID];
		output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
	}

	void Scan::scanLargeEvenDeviceArray(int *output, int *input, int length, bool bcao, int level)
	{
		const int blocks = length / SCAN_ELEMENTS_PER_BLOCK;
		const int sharedMemArraySize = SCAN_ELEMENTS_PER_BLOCK * sizeof(int);

		//The following code is used to avoid malloc GPU memory for each call
		if (level < SCAN_LEVEL)
		{
			const int blocks = length / SCAN_ELEMENTS_PER_BLOCK;
			const int sharedMemArraySize = SCAN_ELEMENTS_PER_BLOCK * sizeof(int);

			if (m_sums[level].size() != blocks)
			{
				m_sums[level].resize(blocks);
				m_incr[level].resize(blocks);
			}

			if (bcao) {
				k_prescan_large << <blocks, SCAN_THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (output, input, SCAN_ELEMENTS_PER_BLOCK, m_sums[level].getDataPtr());
				cuSynchronize();
			}
			else {
				k_prescan_large_unoptimized << <blocks, SCAN_THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (output, input, SCAN_ELEMENTS_PER_BLOCK, m_sums[level].getDataPtr());
				cuSynchronize();
			}

			const int sumsArrThreadsNeeded = (blocks + 1) / 2;
			if (sumsArrThreadsNeeded > SCAN_THREADS_PER_BLOCK) {
				// perform a large scan on the sums arr
				scanLargeDeviceArray(m_incr[level].getDataPtr(), m_sums[level].getDataPtr(), blocks, bcao, level+1);
			}
			else {
				// only need one block to scan sums arr so can use small scan
				scanSmallDeviceArray(m_incr[level].getDataPtr(), m_sums[level].getDataPtr(), blocks, bcao);
			}

			k_add << <blocks, SCAN_ELEMENTS_PER_BLOCK >> > (output, SCAN_ELEMENTS_PER_BLOCK, m_incr[level].getDataPtr());
			cuSynchronize();
		}
		else
		{
			int *d_sums, *d_incr;
			hipMalloc((void **)&d_sums, blocks * sizeof(int));
			hipMalloc((void **)&d_incr, blocks * sizeof(int));

			if (bcao) {
				k_prescan_large << <blocks, SCAN_THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (output, input, SCAN_ELEMENTS_PER_BLOCK, d_sums);
				cuSynchronize();
			}
			else {
				k_prescan_large_unoptimized << <blocks, SCAN_THREADS_PER_BLOCK, 2 * sharedMemArraySize >> > (output, input, SCAN_ELEMENTS_PER_BLOCK, d_sums);
				cuSynchronize();
			}

			const int sumsArrThreadsNeeded = (blocks + 1) / 2;
			if (sumsArrThreadsNeeded > SCAN_THREADS_PER_BLOCK) {
				// perform a large scan on the sums arr
				scanLargeDeviceArray(d_incr, d_sums, blocks, bcao, level + 1);
			}
			else {
				// only need one block to scan sums arr so can use small scan
				scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
			}

			k_add << <blocks, SCAN_ELEMENTS_PER_BLOCK >> > (output, SCAN_ELEMENTS_PER_BLOCK, d_incr);
			cuSynchronize();

			hipFree(d_sums);
			hipFree(d_incr);
		}
	}

	bool Scan::isPowerOfTwo(int x)
	{
		return x && !(x & (x - 1));
	}

	int Scan::nextPowerOfTwo(int x)
	{
		int power = 1;
		while (power < x) {
			power *= 2;
		}
		return power;
	}

}

