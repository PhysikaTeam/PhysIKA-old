#include "hip/hip_runtime.h"
#include "ForEach.h"
#include <hip/hip_runtime.h>


namespace PhysIKA
{
	template<typename Operation> __global__ void kernel_foreach(Operation o)
	{
		size_t i = blockIdx.x * blockDim.x + threadIdx.x;
		o(i);
	}

	void ForEach(size_t size, ...)
	{
	}
}

