#include "hip/hip_runtime.h"
#include "Reduction.h"
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime.h>
#include "cuda_utilities.h"
#include "sharedmem.h"
#include "Functional.h"

namespace PhysIKA {

#define REDUCTION_BLOCK 128

	template<typename T>
	Reduction<T>::Reduction()
		: m_num(0)
		, m_aux(NULL)
	{

	}


	template<typename T>
	Reduction<T>::Reduction(unsigned num)
		: m_num(num)
		, m_aux(NULL)
	{
		allocAuxiliaryArray(m_num);
	}

	template<typename T>
	Reduction<T>::~Reduction()
	{
		hipFree(m_aux);
	}

	template<typename T>
	Reduction<T>* Reduction<T>::Create(int n)
	{
		return new Reduction<T>(n);
	}


	template<typename T>
	int Reduction<T>::getAuxiliaryArraySize(int n)
	{
		return (n / REDUCTION_BLOCK + 1) + (n / (REDUCTION_BLOCK*REDUCTION_BLOCK) + REDUCTION_BLOCK);
	}

	/*!
	*	\brief	Reduction using maximum of float values in shared memory for a warp.
	*/
	template <typename T, 
			  unsigned blockSize,
			  typename Function>
	__device__ 	void KerReduceWarp(volatile T* pData, unsigned tid, Function func)
	{
		if (blockSize >= 64)pData[tid] = func(pData[tid], pData[tid + 32]);
		if (blockSize >= 32)pData[tid] = func(pData[tid], pData[tid + 16]);
		if (blockSize >= 16)pData[tid] = func(pData[tid], pData[tid + 8]);
		if (blockSize >= 8)pData[tid] = func(pData[tid], pData[tid + 4]);
		if (blockSize >= 4)pData[tid] = func(pData[tid], pData[tid + 2]);
		if (blockSize >= 2)pData[tid] = func(pData[tid], pData[tid + 1]);
	}

	/*!
	*	\brief	Accumulates the sum of n values of array pData[], 
	*	storing the result in the beginning of res[].
	*	(Many positions of res[] are used as blocks, storing the final result in res[0]).
	*/
	template <typename T, 
			  unsigned blockSize,
			  typename Function>
	__global__ void KerReduce(const T *pData, unsigned n, T *pAux, Function func, T val)
	{
		//extern __shared__ T sharedMem[];

		SharedMemory<T> smem;
		T* sharedMem = smem.getPointer();

		unsigned tid = threadIdx.x;
		unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
		sharedMem[tid] = (id < n ? pData[id] : val);
		__syncthreads();
		if (blockSize >= 512) { if (tid < 256)sharedMem[tid] = func(sharedMem[tid], sharedMem[tid + 256]);  __syncthreads(); }
		if (blockSize >= 256) { if (tid < 128)sharedMem[tid] = func(sharedMem[tid], sharedMem[tid + 128]);  __syncthreads(); }
		if (blockSize >= 128) { if (tid < 64) sharedMem[tid] = func(sharedMem[tid], sharedMem[tid + 64]);   __syncthreads(); }
		if (tid < 32)KerReduceWarp<T, blockSize>(sharedMem, tid, func);
		if (tid == 0)pAux[blockIdx.x] = sharedMem[0];
	}

	template<typename T, typename Function>
	T Reduce(T* pData, unsigned num, T* pAux, Function func, T v0)
	{
		unsigned n = num;
		unsigned sharedMemSize = REDUCTION_BLOCK * sizeof(T);
		unsigned blockNum = cudaGridSize(num, REDUCTION_BLOCK);
		T* subData = pData;
		T* aux1 = pAux;
		T* aux2 = pAux + blockNum;
		T* subAux = aux1;
		while (n > 1) {
			KerReduce<T, REDUCTION_BLOCK, Function> << <blockNum, REDUCTION_BLOCK, sharedMemSize >> > (subData, n, subAux, func, v0);
			n = blockNum; 
			blockNum = cudaGridSize(n, REDUCTION_BLOCK);
			if (n > 1) {
				subData = subAux; subAux = (subData == aux1 ? aux2 : aux1);
			}
		}

		T val;
		if (num > 1)
			hipMemcpy(&val, subAux, sizeof(T), hipMemcpyDeviceToHost);
		else 
			hipMemcpy(&val, pData, sizeof(T), hipMemcpyDeviceToHost);

		return val;
	}

	template<typename T>
	T PhysIKA::Reduction<T>::accumulate(T* val, int num)
	{
		if (num != m_num)
			allocAuxiliaryArray(num);

		return Reduce(val, num, m_aux, PlusFunc<T>(), (T)0);
	}

	template<typename T>
	T PhysIKA::Reduction<T>::maximum(T* val, int num)
	{
		if (num != m_num)
			allocAuxiliaryArray(num);

		return Reduce(val, num, m_aux, MaximumFunc<T>(), (T)-FLT_MAX);
	}

	template<typename T>
	T PhysIKA::Reduction<T>::minimum(T* val, int num)
	{
		if (num != m_num)
			allocAuxiliaryArray(num);

		return Reduce(val, num, m_aux, MinimumFunc<T>(), (T)FLT_MAX);
	}

	template<typename T>
	T PhysIKA::Reduction<T>::average(T* val, int num)
	{
		if (num != m_num)
			allocAuxiliaryArray(num);

		return Reduce(val, num, m_aux, PlusFunc<T>(), (T)0) / num;
	}

	template<typename T>
	void Reduction<T>::allocAuxiliaryArray(int num)
	{
		if (m_aux == nullptr)
		{
			hipFree(m_aux);
		}

		m_num = num;

		m_auxNum = getAuxiliaryArraySize(num);
		hipMalloc((void**)&m_aux, m_auxNum * sizeof(T));
	}

}