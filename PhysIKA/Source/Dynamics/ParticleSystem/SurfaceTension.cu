#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Core/Utility.h"
#include "SurfaceTension.h"
#include "Framework/Framework/MechanicalState.h"
#include "Framework/Topology/NeighborList.h"
#include "Kernel.h"

namespace PhysIKA
{
	template<typename Real, typename Coord>
	__global__ void ST_ComputeSurfaceEnergy
	(
		DeviceArray<Real> energyArr,
		DeviceArray<Coord> posArr,
		NeighborList<int> neighbors,
		Real smoothingLength
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Real total_weight = Real(0);
		Coord dir_i(0);

		SmoothKernel<Real> kern;

		Coord pos_i = posArr[pId];
		int nbSize = neighbors.getNeighborSize(pId);
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors.getElement(pId, ne);
			Real r = (pos_i - posArr[j]).norm();

			if (r > EPSILON)
			{
				Real weight = -kern.Gradient(r, smoothingLength);
				total_weight += weight;
				dir_i += (posArr[j] - pos_i)*(weight / r);
			}
		}

		total_weight = total_weight < EPSILON ? 1.0f : total_weight;
		Real absDir = dir_i.norm() / total_weight;

		energyArr[pId] = absDir*absDir;
	}

	template<typename Real, typename Coord>
	__global__ void ST_ComputeSurfaceTension
	(
		DeviceArray<Coord> velArr, 
		DeviceArray<Real> energyArr, 
		DeviceArray<Coord> posArr, 
		NeighborList<int> neighbors,
		Real smoothingLength,
		Real mass,
		Real restDensity,
		float dt
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Real Vref = mass / restDensity;

		float alpha = (float) 945.0f / (32.0f * (float)M_PI * smoothingLength * smoothingLength * smoothingLength);
		float ceof = 16000.0f * alpha;

		SmoothKernel<Real> kern;

		Coord F_i(0);
		Coord dv_pi(0);
		Coord pos_i = posArr[pId];
		int nbSize = neighbors.getNeighborSize(pId);
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors.getElement(pId, ne);
			float r = (pos_i - posArr[j]).norm();

			if (r > EPSILON)
			{
				Coord temp = Vref*Vref*kern.Gradient(r, smoothingLength)*(posArr[j] - pos_i) * (1.0f / r);
				Coord dv_ij = dt * ceof*1.0f*(energyArr[pId])*temp / mass;
				F_i += dv_ij;

// 				atomicAdd(&velArr[j].x, dv_ij.x);
// 				atomicAdd(&velArr[j].y, dv_ij.y);
// 				atomicAdd(&velArr[j].z, dv_ij.z);
			}
		}
		velArr[pId] -= F_i;

// 		atomicAdd(&velArr[pId].x, -F_i.x);
// 		atomicAdd(&velArr[pId].y, -F_i.y);
// 		atomicAdd(&velArr[pId].z, -F_i.z);
	}

	template<typename TDataType>
	SurfaceTension<TDataType>::SurfaceTension()
		: ForceModule()
		, m_posID(MechanicalState::position())
		, m_velID(MechanicalState::velocity())
		, m_neighborhoodID(MechanicalState::particle_neighbors())
		, m_intensity(Real(1))
		, m_soothingLength(Real(0.0125))
	{

	}

	template<typename TDataType>
	bool SurfaceTension<TDataType>::execute()
	{
// 		m_energy = DeviceBuffer<Real>::create(num);
// 
// 		DeviceArray<Coord>* posArr = m_parent->GetNewPositionBuffer()->getDataPtr();
// 		DeviceArray<Coord>* velArr = m_parent->GetNewVelocityBuffer()->getDataPtr();
// 		DeviceArray<Attribute>* attArr = m_parent->GetAttributeBuffer()->getDataPtr();
// 		float dt = m_parent->getDt();
// 
// 		DeviceArray<SPHNeighborList>* neighborArr = m_parent->GetNeighborBuffer()->getDataPtr();
// 
// 		DeviceArray<Real>* energy = m_energy->getDataPtr();
// 
// 		Real mass = m_parent->GetParticleMass();
// 		Real smoothingLength = m_parent->GetSmoothingLength();
// 		Real restDensity = m_parent->GetRestDensity();
// 
// 		uint pDims = cudaGridSize(posArr->size(), BLOCK_SIZE);
// 		ST_ComputeSurfaceEnergy <Real, Coord> << < pDims, BLOCK_SIZE >> > (*energy, *posArr, *neighborArr, smoothingLength);
// 		ST_ComputeSurfaceTension <Real, Coord> << < pDims, BLOCK_SIZE >> > (*velArr, *energy, *posArr, *attArr, *neighborArr, smoothingLength, mass, restDensity, dt);

		return true;
	}

	template<typename TDataType>
	bool SurfaceTension<TDataType>::applyForce()
	{
		return true;
	}

}