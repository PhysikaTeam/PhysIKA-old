#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ParticleIntegrator.h"
#include "Framework/Framework/FieldArray.h"
#include "Framework/Framework/FieldVar.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "Framework/Framework/SceneGraph.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS_1(ParticleIntegrator, TDataType)

	template<typename TDataType>
	ParticleIntegrator<TDataType>::ParticleIntegrator()
		: NumericalIntegrator()
	{
	}

	template<typename TDataType>
	void ParticleIntegrator<TDataType>::begin()
	{
		if (!this->inPosition()->isEmpty())
		{
			int num = this->inPosition()->getElementCount();
			
			m_prePosition.resize(num);
			m_preVelocity.resize(num);

			Function1Pt::copy(m_prePosition, this->inPosition()->getValue());
			Function1Pt::copy(m_preVelocity, this->inVelocity()->getValue());

			this->inForceDensity()->getReference()->reset();
		}
	}

	template<typename TDataType>
	void ParticleIntegrator<TDataType>::end()
	{

	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::initializeImpl()
	{
// 		if (!isAllFieldsReady())
// 		{
// 			std::cout << "Exception: " << std::string("DensitySummation's fields are not fully initialized!") << "\n";
// 			return false;
// 		}
// 
// 		int num = this->inPosition()->getElementCount();
// 
// 		m_prePosition.resize(num);
// 		m_preVelocity.resize(num);

		return true;
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> vel,
		DeviceArray<Coord> forceDensity,
		Coord gravity,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= forceDensity.size()) return;

		vel[pId] += dt * (forceDensity[pId] + gravity);
	}


	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DeviceArray<Coord> vel,
		DeviceArray<Coord> force,
		DeviceArray<Real> mass,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= force.size()) return;

		vel[pId] += dt * force[pId] / mass[pId];
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::updateVelocity()
	{
		Real dt = getParent()->getDt();
		Coord gravity = SceneGraph::getInstance().getGravity();

		int total_num = this->inPosition()->getElementCount();
		cuExecute(total_num, K_UpdateVelocity,
			this->inVelocity()->getValue(),
			this->inForceDensity()->getValue(),
			gravity,
			dt);

		return true;
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DeviceArray<Coord> pos,
		DeviceArray<Coord> vel,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		pos[pId] += dt * vel[pId];
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::updatePosition()
	{
		Real dt = getParent()->getDt();

		int total_num = this->inPosition()->getReference()->size();
		cuExecute(total_num, K_UpdatePosition,
			this->inPosition()->getValue(),
			this->inVelocity()->getValue(),
			dt);

		return true;
	}

	template<typename TDataType>
	bool ParticleIntegrator<TDataType>::integrate()
	{
		if (!this->inPosition()->isEmpty())
		{
			updateVelocity();
			updatePosition();
		}

		return true;
	}
}