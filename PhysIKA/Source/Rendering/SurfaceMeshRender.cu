#include "hip/hip_runtime.h"
#pragma once
#include <GL/glew.h>
#include "SurfaceMeshRender.h"
#include "Framework/Topology/TriangleSet.h"
#include "Core/Vector.h"
#include "Framework/Framework/Node.h"
#include "Core/Utility.h"
#include "OpenGLContext.h"

namespace PhysIKA
{
	IMPLEMENT_CLASS(SurfaceMeshRender)

		SurfaceMeshRender::SurfaceMeshRender()
		: VisualModule()
		, m_color(Vector3f(0.2f, 0.3, 0.0f))
	{
	}

	SurfaceMeshRender::~SurfaceMeshRender()
	{
		vertices.release();
		normals.release();
		colors.release();
	}

	bool SurfaceMeshRender::initializeImpl()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return false;
		}

		auto triSet = TypeInfo::CastPointerDown<TriangleSet<DataType3f>>(parent->getTopologyModule());
		if (triSet == nullptr)
		{
			Log::sendMessage(Log::Error, "TriangleModule: The topology module is not supported!");
			return false;
		}

//		point_render_util = std::make_shared<PointRenderUtil>();

		auto triangles = triSet->getTriangles();

		m_triangleRender = std::make_shared<TriangleRender>();
		m_triangleRender->resize(triangles->size());

		vertices.resize(3 * triangles->size());
		normals.resize(3 * triangles->size());
		colors.resize(3 * triangles->size());
	}

	__global__ void SetupTriangles(
		DeviceArray<float3> originVerts,
		DeviceArray<float3> vertices,
		DeviceArray<float3> normals,
		DeviceArray<float3> colors,
		DeviceArray<TopologyModule::Triangle> triangles,
		float3 color
		)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= triangles.size()) return;

		TopologyModule::Triangle tri = triangles[pId];
		float3 v1 = originVerts[tri[0]];
		float3 v2 = originVerts[tri[1]];
		float3 v3 = originVerts[tri[2]];

		vertices[3 * pId + 0] = v1;
		vertices[3 * pId + 1] = v2;
		vertices[3 * pId + 2] = v3;

		float3 triN = cross(v2-v1, v3-v1);
		triN = normalize(triN);

		normals[3 * pId + 0] = triN;
		normals[3 * pId + 1] = triN;
		normals[3 * pId + 2] = triN;

		colors[3 * pId + 0] = color;
		colors[3 * pId + 1] = color;
		colors[3 * pId + 2] = color;
	}

	void SurfaceMeshRender::updateRenderingContext()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return;
		}

		auto triSet = TypeInfo::CastPointerDown<TriangleSet<DataType3f>>(parent->getTopologyModule());
		if (triSet == nullptr)
		{
			Log::sendMessage(Log::Error, "TriangleModule: The topology module is not supported!");
			return;
		}

		auto verts = triSet->getPoints();
		auto triangles = triSet->getTriangles();

		uint pDims = cudaGridSize(triangles->size(), BLOCK_SIZE);

		DeviceArray<float3>* fverts = (DeviceArray<float3>*)&verts;
		SetupTriangles << <pDims, BLOCK_SIZE >> >(*fverts, vertices, normals, colors, *triangles, make_float3(m_color[0], m_color[1], m_color[2]));


		m_triangleRender->setVertexArray(vertices);
		m_triangleRender->setColorArray(colors);
		m_triangleRender->setNormalArray(normals);
	}

	void SurfaceMeshRender::display()
	{
		glMatrixMode(GL_MODELVIEW_MATRIX);
		glPushMatrix();

		glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
		glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
		glScalef(m_scale[0], m_scale[1], m_scale[2]);

		m_triangleRender->display();

		glPopMatrix();
	}

	void SurfaceMeshRender::setColor(Vector3f color)
	{
		m_color = color;
	}

}