#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/type_traits.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"

namespace cv { namespace gpu { namespace device
{
    void writeScalar(const uchar*);
    void writeScalar(const schar*);
    void writeScalar(const ushort*);
    void writeScalar(const short int*);
    void writeScalar(const int*);
    void writeScalar(const float*);
    void writeScalar(const double*);
    void copyToWithMask_gpu(PtrStepSzb src, PtrStepSzb dst, size_t elemSize1, int cn, PtrStepSzb mask, bool colorMask, hipStream_t stream);
    void convert_gpu(PtrStepSzb, int, PtrStepSzb, int, double, double, hipStream_t);
}}}

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct shift_and_sizeof;
    template <> struct shift_and_sizeof<signed char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<unsigned char> { enum { shift = 0 }; };
    template <> struct shift_and_sizeof<short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<unsigned short> { enum { shift = 1 }; };
    template <> struct shift_and_sizeof<int> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<float> { enum { shift = 2 }; };
    template <> struct shift_and_sizeof<double> { enum { shift = 3 }; };

    ///////////////////////////////////////////////////////////////////////////
    ////////////////////////////////// CopyTo /////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    template <typename T> void copyToWithMask(PtrStepSzb src, PtrStepSzb dst, int cn, PtrStepSzb mask, bool colorMask, hipStream_t stream)
    {
        if (colorMask)
            cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<T>)dst, identity<T>(), SingleMask(mask), stream);
        else
            cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<T>)dst, identity<T>(), SingleMaskChannels(mask, cn), stream);
    }

    void copyToWithMask_gpu(PtrStepSzb src, PtrStepSzb dst, size_t elemSize1, int cn, PtrStepSzb mask, bool colorMask, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, PtrStepSzb dst, int cn, PtrStepSzb mask, bool colorMask, hipStream_t stream);

        static func_t tab[] =
        {
            0,
            copyToWithMask<unsigned char>,
            copyToWithMask<unsigned short>,
            0,
            copyToWithMask<int>,
            0,
            0,
            0,
            copyToWithMask<double>
        };

        tab[elemSize1](src, dst, cn, mask, colorMask, stream);
    }

    ///////////////////////////////////////////////////////////////////////////
    ////////////////////////////////// SetTo //////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    template<typename T>
    __global__ void set_to_without_mask(PtrStepSz<T> mat, typename TypeVec<T, 4>::vec_type val, int channels)
    {
        const int y = blockIdx.x * blockDim.y + threadIdx.y;

        if (y < mat.rows)
        {
            const T vals[] = {
                val.x, val.y, val.z, val.w
            };

            T* row = mat.ptr(y);

            for (int x = threadIdx.x; x < mat.cols * channels; x += 32)
            {
                row[x] = vals[x % channels];
            }
        }
    }

    template<typename T>
    __global__ void set_to_with_mask(PtrStepSz<T> mat, const PtrStepb mask, typename TypeVec<T, 4>::vec_type val, int channels)
    {
        const int y = blockIdx.x * blockDim.y + threadIdx.y;

        if (y < mat.rows)
        {
            const T vals[] = {
                val.x, val.y, val.z, val.w
            };

            T* row = mat.ptr(y);
            const uchar* mask_row = mask.ptr(y);

            for (int x = threadIdx.x; x < mat.cols * channels; x += 32)
            {
                if (mask_row[x / channels])
                {
                    row[x] = vals[x % channels];
                }
            }
        }
    }

    template <typename T>
    void set_to_gpu(PtrStepSzb mat, const T* scalar, PtrStepSzb mask, int channels, hipStream_t stream)
    {
        typedef typename TypeVec<T, 4>::vec_type vec_type;

        dim3 block(32, 8);
        dim3 grid(divUp(mat.rows, block.y));

        set_to_with_mask<T><<<grid, block, 0, stream>>>(PtrStepSz<T>(mat), mask, VecTraits<vec_type>::make(scalar), channels);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(PtrStepSzb mat, const uchar*  scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(PtrStepSzb mat, const schar*  scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(PtrStepSzb mat, const ushort* scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<short >(PtrStepSzb mat, const short*  scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(PtrStepSzb mat, const int*    scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<float >(PtrStepSzb mat, const float*  scalar, PtrStepSzb mask, int channels, hipStream_t stream);
    template void set_to_gpu<double>(PtrStepSzb mat, const double* scalar, PtrStepSzb mask, int channels, hipStream_t stream);

    template <typename T>
    void set_to_gpu(PtrStepSzb mat, const T* scalar, int channels, hipStream_t stream)
    {
        typedef typename TypeVec<T, 4>::vec_type vec_type;

        dim3 block(32, 8);
        dim3 grid(divUp(mat.rows, block.y));

        set_to_without_mask<T><<<grid, block, 0, stream>>>(PtrStepSz<T>(mat), VecTraits<vec_type>::make(scalar), channels);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall ( hipDeviceSynchronize() );
    }

    template void set_to_gpu<uchar >(PtrStepSzb mat, const uchar*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<schar >(PtrStepSzb mat, const schar*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<ushort>(PtrStepSzb mat, const ushort* scalar, int channels, hipStream_t stream);
    template void set_to_gpu<short >(PtrStepSzb mat, const short*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<int   >(PtrStepSzb mat, const int*    scalar, int channels, hipStream_t stream);
    template void set_to_gpu<float >(PtrStepSzb mat, const float*  scalar, int channels, hipStream_t stream);
    template void set_to_gpu<double>(PtrStepSzb mat, const double* scalar, int channels, hipStream_t stream);

    ///////////////////////////////////////////////////////////////////////////
    //////////////////////////////// ConvertTo ////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    template <typename T, typename D, typename S> struct Convertor : unary_function<T, D>
    {
        Convertor(S alpha_, S beta_) : alpha(alpha_), beta(beta_) {}

        __device__ __forceinline__ D operator()(typename TypeTraits<T>::ParameterType src) const
        {
            return saturate_cast<D>(alpha * src + beta);
        }

        S alpha, beta;
    };

    namespace detail
    {
        template <size_t src_size, size_t dst_size, typename F> struct ConvertTraitsDispatcher : DefaultTransformFunctorTraits<F>
        {
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 1, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 8 };
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<1, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 4 };
        };

        template <typename F> struct ConvertTraitsDispatcher<2, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<2, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_shift = 2 };
        };

        template <typename F> struct ConvertTraitsDispatcher<4, 2, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 4 };
        };
        template <typename F> struct ConvertTraitsDispatcher<4, 4, F> : DefaultTransformFunctorTraits<F>
        {
            enum { smart_block_dim_y = 8 };
            enum { smart_shift = 2 };
        };

        template <typename F> struct ConvertTraits : ConvertTraitsDispatcher<sizeof(typename F::argument_type), sizeof(typename F::result_type), F>
        {
        };
    }

    template <typename T, typename D, typename S> struct TransformFunctorTraits< Convertor<T, D, S> > : detail::ConvertTraits< Convertor<T, D, S> >
    {
    };

    template<typename T, typename D, typename S>
    void cvt_(PtrStepSzb src, PtrStepSzb dst, double alpha, double beta, hipStream_t stream)
    {
        cudaSafeCall( cudaSetDoubleForDevice(&alpha) );
        cudaSafeCall( cudaSetDoubleForDevice(&beta) );
        Convertor<T, D, S> op(static_cast<S>(alpha), static_cast<S>(beta));
        cv::gpu::device::transform((PtrStepSz<T>)src, (PtrStepSz<D>)dst, op, WithOutMask(), stream);
    }

#if defined  __clang__
# pragma clang diagnostic push
# pragma clang diagnostic ignored "-Wmissing-declarations"
#endif

    void convert_gpu(PtrStepSzb src, int sdepth, PtrStepSzb dst, int ddepth, double alpha, double beta, hipStream_t stream)
    {
        typedef void (*caller_t)(PtrStepSzb src, PtrStepSzb dst, double alpha, double beta, hipStream_t stream);

        static const caller_t tab[7][7] =
        {
            {
                cvt_<uchar, uchar, float>,
                cvt_<uchar, schar, float>,
                cvt_<uchar, ushort, float>,
                cvt_<uchar, short, float>,
                cvt_<uchar, int, float>,
                cvt_<uchar, float, float>,
                cvt_<uchar, double, double>
            },
            {
                cvt_<schar, uchar, float>,
                cvt_<schar, schar, float>,
                cvt_<schar, ushort, float>,
                cvt_<schar, short, float>,
                cvt_<schar, int, float>,
                cvt_<schar, float, float>,
                cvt_<schar, double, double>
            },
            {
                cvt_<ushort, uchar, float>,
                cvt_<ushort, schar, float>,
                cvt_<ushort, ushort, float>,
                cvt_<ushort, short, float>,
                cvt_<ushort, int, float>,
                cvt_<ushort, float, float>,
                cvt_<ushort, double, double>
            },
            {
                cvt_<short, uchar, float>,
                cvt_<short, schar, float>,
                cvt_<short, ushort, float>,
                cvt_<short, short, float>,
                cvt_<short, int, float>,
                cvt_<short, float, float>,
                cvt_<short, double, double>
            },
            {
                cvt_<int, uchar, float>,
                cvt_<int, schar, float>,
                cvt_<int, ushort, float>,
                cvt_<int, short, float>,
                cvt_<int, int, double>,
                cvt_<int, float, double>,
                cvt_<int, double, double>
            },
            {
                cvt_<float, uchar, float>,
                cvt_<float, schar, float>,
                cvt_<float, ushort, float>,
                cvt_<float, short, float>,
                cvt_<float, int, float>,
                cvt_<float, float, float>,
                cvt_<float, double, double>
            },
            {
                cvt_<double, uchar, double>,
                cvt_<double, schar, double>,
                cvt_<double, ushort, double>,
                cvt_<double, short, double>,
                cvt_<double, int, double>,
                cvt_<double, float, double>,
                cvt_<double, double, double>
            }
        };

        caller_t func = tab[sdepth][ddepth];
        func(src, dst, alpha, beta, stream);
    }

#if defined __clang__
# pragma clang diagnostic pop
#endif
}}} // namespace cv { namespace gpu { namespace device
