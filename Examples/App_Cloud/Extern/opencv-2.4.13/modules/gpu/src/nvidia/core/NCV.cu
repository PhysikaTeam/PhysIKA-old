#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <algorithm>
#include <iostream>
#include <string>
#include <vector>
#include "NCV.hpp"


//==============================================================================
//
// Error handling helpers
//
//==============================================================================


static void stdDebugOutput(const std::string &msg)
{
    std::cout << msg;
}


static NCVDebugOutputHandler *debugOutputHandler = stdDebugOutput;


void ncvDebugOutput(const std::string &msg)
{
    debugOutputHandler(msg);
}


void ncvSetDebugOutputHandler(NCVDebugOutputHandler *func)
{
    debugOutputHandler = func;
}

#if !defined CUDA_DISABLER


//==============================================================================
//
// Memory wrappers and helpers
//
//==============================================================================


Ncv32u alignUp(Ncv32u what, Ncv32u alignment)
{
    Ncv32u alignMask = alignment-1;
    Ncv32u inverseAlignMask = ~alignMask;
    Ncv32u res = (what + alignMask) & inverseAlignMask;
    return res;
}


void NCVMemPtr::clear()
{
    ptr = NULL;
    memtype = NCVMemoryTypeNone;
}


void NCVMemSegment::clear()
{
    begin.clear();
    size = 0;
}


NCVStatus memSegCopyHelper(void *dst, NCVMemoryType dstType, const void *src, NCVMemoryType srcType, size_t sz, hipStream_t cuStream)
{
    NCVStatus ncvStat;
    switch (dstType)
    {
    case NCVMemoryTypeHostPageable:
    case NCVMemoryTypeHostPinned:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            memcpy(dst, src, sz);
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    case NCVMemoryTypeDevice:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyHostToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyDeviceToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    default:
        ncvStat = NCV_MEM_RESIDENCE_ERROR;
    }

    return ncvStat;
}


NCVStatus memSegCopyHelper2D(void *dst, Ncv32u dstPitch, NCVMemoryType dstType,
                             const void *src, Ncv32u srcPitch, NCVMemoryType srcType,
                             Ncv32u widthbytes, Ncv32u height, hipStream_t cuStream)
{
    NCVStatus ncvStat;
    switch (dstType)
    {
    case NCVMemoryTypeHostPageable:
    case NCVMemoryTypeHostPinned:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            for (Ncv32u i=0; i<height; i++)
            {
                memcpy((char*)dst + i * dstPitch, (char*)src + i * srcPitch, widthbytes);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToHost), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    case NCVMemoryTypeDevice:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyHostToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    default:
        ncvStat = NCV_MEM_RESIDENCE_ERROR;
    }

    return ncvStat;
}


//===================================================================
//
// NCVMemStackAllocator class members implementation
//
//===================================================================


NCVMemStackAllocator::NCVMemStackAllocator(Ncv32u alignment_)
    :
    currentSize(0),
    _maxSize(0),
    allocBegin(NULL),
    begin(NULL),
    end(NULL),
    _memType(NCVMemoryTypeNone),
    _alignment(alignment_),
    bReusesMemory(false)
{
    NcvBool bProperAlignment = (alignment_ & (alignment_ - 1)) == 0;
    ncvAssertPrintCheck(bProperAlignment, "NCVMemStackAllocator ctor:: alignment not power of 2");
}


NCVMemStackAllocator::NCVMemStackAllocator(NCVMemoryType memT, size_t capacity, Ncv32u alignment_, void *reusePtr)
    :
    currentSize(0),
    _maxSize(0),
    allocBegin(NULL),
    _memType(memT),
    _alignment(alignment_)
{
    NcvBool bProperAlignment = (alignment_ & (alignment_ - 1)) == 0;
    ncvAssertPrintCheck(bProperAlignment, "NCVMemStackAllocator ctor:: _alignment not power of 2");
    ncvAssertPrintCheck(memT != NCVMemoryTypeNone, "NCVMemStackAllocator ctor:: Incorrect allocator type");

    allocBegin = NULL;

    if (reusePtr == NULL && capacity != 0)
    {
        bReusesMemory = false;
        switch (memT)
        {
        case NCVMemoryTypeDevice:
            ncvAssertCUDAReturn(hipMalloc(&allocBegin, capacity), );
            break;
        case NCVMemoryTypeHostPinned:
            ncvAssertCUDAReturn(hipHostMalloc(&allocBegin, capacity), );
            break;
        case NCVMemoryTypeHostPageable:
            allocBegin = (Ncv8u *)malloc(capacity);
            break;
        default:;
        }
    }
    else
    {
        bReusesMemory = true;
        allocBegin = (Ncv8u *)reusePtr;
    }

    if (capacity == 0)
    {
        allocBegin = (Ncv8u *)(0x1);
    }

    if (!isCounting())
    {
        begin = allocBegin;
        end = begin + capacity;
    }
}


NCVMemStackAllocator::~NCVMemStackAllocator()
{
    if (allocBegin != NULL)
    {
        ncvAssertPrintCheck(currentSize == 0, "NCVMemStackAllocator dtor:: not all objects were deallocated properly, forcing destruction");

        if (!bReusesMemory && (allocBegin != (Ncv8u *)(0x1)))
        {
            switch (_memType)
            {
            case NCVMemoryTypeDevice:
                ncvAssertCUDAReturn(hipFree(allocBegin), );
                break;
            case NCVMemoryTypeHostPinned:
                ncvAssertCUDAReturn(hipHostFree(allocBegin), );
                break;
            case NCVMemoryTypeHostPageable:
                free(allocBegin);
                break;
            default:;
            }
        }

        allocBegin = NULL;
    }
}


NCVStatus NCVMemStackAllocator::alloc(NCVMemSegment &seg, size_t size)
{
    seg.clear();
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);

    size = alignUp(static_cast<Ncv32u>(size), this->_alignment);
    this->currentSize += size;
    this->_maxSize = std::max(this->_maxSize, this->currentSize);

    if (!isCounting())
    {
        size_t availSize = end - begin;
        ncvAssertReturn(size <= availSize, NCV_ALLOCATOR_INSUFFICIENT_CAPACITY);
    }

    seg.begin.ptr = begin;
    seg.begin.memtype = this->_memType;
    seg.size = size;
    begin += size;

    return NCV_SUCCESS;
}


NCVStatus NCVMemStackAllocator::dealloc(NCVMemSegment &seg)
{
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);
    ncvAssertReturn(seg.begin.memtype == this->_memType, NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr != NULL || isCounting(), NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr == begin - seg.size, NCV_ALLOCATOR_DEALLOC_ORDER);

    currentSize -= seg.size;
    begin -= seg.size;

    seg.clear();

    ncvAssertReturn(allocBegin <= begin, NCV_ALLOCATOR_BAD_DEALLOC);

    return NCV_SUCCESS;
}


NcvBool NCVMemStackAllocator::isInitialized(void) const
{
    return ((this->_alignment & (this->_alignment-1)) == 0) && isCounting() || this->allocBegin != NULL;
}


NcvBool NCVMemStackAllocator::isCounting(void) const
{
    return this->_memType == NCVMemoryTypeNone;
}


NCVMemoryType NCVMemStackAllocator::memType(void) const
{
    return this->_memType;
}


Ncv32u NCVMemStackAllocator::alignment(void) const
{
    return this->_alignment;
}


size_t NCVMemStackAllocator::maxSize(void) const
{
    return this->_maxSize;
}


//===================================================================
//
// NCVMemNativeAllocator class members implementation
//
//===================================================================


NCVMemNativeAllocator::NCVMemNativeAllocator(NCVMemoryType memT, Ncv32u alignment_)
    :
    currentSize(0),
    _maxSize(0),
    _memType(memT),
    _alignment(alignment_)
{
    ncvAssertPrintReturn(memT != NCVMemoryTypeNone, "NCVMemNativeAllocator ctor:: counting not permitted for this allocator type", );
}


NCVMemNativeAllocator::~NCVMemNativeAllocator()
{
    ncvAssertPrintCheck(currentSize == 0, "NCVMemNativeAllocator dtor:: detected memory leak");
}


NCVStatus NCVMemNativeAllocator::alloc(NCVMemSegment &seg, size_t size)
{
    seg.clear();
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);

    switch (this->_memType)
    {
    case NCVMemoryTypeDevice:
        ncvAssertCUDAReturn(hipMalloc(&seg.begin.ptr, size), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPinned:
        ncvAssertCUDAReturn(hipHostMalloc(&seg.begin.ptr, size), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPageable:
        seg.begin.ptr = (Ncv8u *)malloc(size);
        break;
    default:;
    }

    this->currentSize += alignUp(static_cast<Ncv32u>(size), this->_alignment);
    this->_maxSize = std::max(this->_maxSize, this->currentSize);

    seg.begin.memtype = this->_memType;
    seg.size = size;

    return NCV_SUCCESS;
}


NCVStatus NCVMemNativeAllocator::dealloc(NCVMemSegment &seg)
{
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);
    ncvAssertReturn(seg.begin.memtype == this->_memType, NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr != NULL, NCV_ALLOCATOR_BAD_DEALLOC);

    ncvAssertReturn(currentSize >= alignUp(static_cast<Ncv32u>(seg.size), this->_alignment), NCV_ALLOCATOR_BAD_DEALLOC);
    currentSize -= alignUp(static_cast<Ncv32u>(seg.size), this->_alignment);

    switch (this->_memType)
    {
    case NCVMemoryTypeDevice:
        ncvAssertCUDAReturn(hipFree(seg.begin.ptr), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPinned:
        ncvAssertCUDAReturn(hipHostFree(seg.begin.ptr), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPageable:
        free(seg.begin.ptr);
        break;
    default:;
    }

    seg.clear();

    return NCV_SUCCESS;
}


NcvBool NCVMemNativeAllocator::isInitialized(void) const
{
    return (this->_alignment != 0);
}


NcvBool NCVMemNativeAllocator::isCounting(void) const
{
    return false;
}


NCVMemoryType NCVMemNativeAllocator::memType(void) const
{
    return this->_memType;
}


Ncv32u NCVMemNativeAllocator::alignment(void) const
{
    return this->_alignment;
}


size_t NCVMemNativeAllocator::maxSize(void) const
{
    return this->_maxSize;
}


//===================================================================
//
// Time and timer routines
//
//===================================================================


typedef struct _NcvTimeMoment NcvTimeMoment;

#if defined(_WIN32) || defined(_WIN64)

    #include <Windows.h>

    typedef struct _NcvTimeMoment
    {
        LONGLONG moment, freq;
    } NcvTimeMoment;


    static void _ncvQueryMoment(NcvTimeMoment *t)
    {
        QueryPerformanceFrequency((LARGE_INTEGER *)&(t->freq));
        QueryPerformanceCounter((LARGE_INTEGER *)&(t->moment));
    }


    double _ncvMomentToMicroseconds(NcvTimeMoment *t)
    {
        return 1000000.0 * t->moment / t->freq;
    }


    double _ncvMomentsDiffToMicroseconds(NcvTimeMoment *t1, NcvTimeMoment *t2)
    {
        return 1000000.0 * 2 * ((t2->moment) - (t1->moment)) / (t1->freq + t2->freq);
    }


    double _ncvMomentsDiffToMilliseconds(NcvTimeMoment *t1, NcvTimeMoment *t2)
    {
        return 1000.0 * 2 * ((t2->moment) - (t1->moment)) / (t1->freq + t2->freq);
    }

#elif defined(__GNUC__)

    #include <sys/time.h>

    typedef struct _NcvTimeMoment
    {
        struct timeval tv;
        struct timezone tz;
    } NcvTimeMoment;


    void _ncvQueryMoment(NcvTimeMoment *t)
    {
        gettimeofday(& t->tv, & t->tz);
    }


    double _ncvMomentToMicroseconds(NcvTimeMoment *t)
    {
        return 1000000.0 * t->tv.tv_sec + (double)t->tv.tv_usec;
    }


    double _ncvMomentsDiffToMicroseconds(NcvTimeMoment *t1, NcvTimeMoment *t2)
    {
        return (((double)t2->tv.tv_sec - (double)t1->tv.tv_sec) * 1000000 + (double)t2->tv.tv_usec - (double)t1->tv.tv_usec);
    }

    double _ncvMomentsDiffToMilliseconds(NcvTimeMoment *t1, NcvTimeMoment *t2)
    {
        return ((double)t2->tv.tv_sec - (double)t1->tv.tv_sec) * 1000;
    }

#endif //#if defined(_WIN32) || defined(_WIN64)


struct _NcvTimer
{
    NcvTimeMoment t1, t2;
};


NcvTimer ncvStartTimer(void)
{
    struct _NcvTimer *t;
    t = (struct _NcvTimer *)malloc(sizeof(struct _NcvTimer));
    _ncvQueryMoment(&t->t1);
    return t;
}


double ncvEndQueryTimerUs(NcvTimer t)
{
    double res;
    _ncvQueryMoment(&t->t2);
    res = _ncvMomentsDiffToMicroseconds(&t->t1, &t->t2);
    free(t);
    return res;
}


double ncvEndQueryTimerMs(NcvTimer t)
{
    double res;
    _ncvQueryMoment(&t->t2);
    res = _ncvMomentsDiffToMilliseconds(&t->t1, &t->t2);
    free(t);
    return res;
}


//===================================================================
//
// Operations with rectangles
//
//===================================================================


//from OpenCV
void groupRectangles(std::vector<NcvRect32u> &hypotheses, int groupThreshold, double eps, std::vector<Ncv32u> *weights);


NCVStatus ncvGroupRectangles_host(NCVVector<NcvRect32u> &hypotheses,
                                  Ncv32u &numHypotheses,
                                  Ncv32u minNeighbors,
                                  Ncv32f intersectEps,
                                  NCVVector<Ncv32u> *hypothesesWeights)
{
    ncvAssertReturn(hypotheses.memType() == NCVMemoryTypeHostPageable ||
                    hypotheses.memType() == NCVMemoryTypeHostPinned, NCV_MEM_RESIDENCE_ERROR);
    if (hypothesesWeights != NULL)
    {
        ncvAssertReturn(hypothesesWeights->memType() == NCVMemoryTypeHostPageable ||
                        hypothesesWeights->memType() == NCVMemoryTypeHostPinned, NCV_MEM_RESIDENCE_ERROR);
    }

    if (numHypotheses == 0)
    {
        return NCV_SUCCESS;
    }

    std::vector<NcvRect32u> rects(numHypotheses);
    memcpy(&rects[0], hypotheses.ptr(), numHypotheses * sizeof(NcvRect32u));

    std::vector<Ncv32u> weights;
    if (hypothesesWeights != NULL)
    {
        groupRectangles(rects, minNeighbors, intersectEps, &weights);
    }
    else
    {
        groupRectangles(rects, minNeighbors, intersectEps, NULL);
    }

    numHypotheses = (Ncv32u)rects.size();
    if (numHypotheses > 0)
    {
        memcpy(hypotheses.ptr(), &rects[0], numHypotheses * sizeof(NcvRect32u));
    }

    if (hypothesesWeights != NULL)
    {
        memcpy(hypothesesWeights->ptr(), &weights[0], numHypotheses * sizeof(Ncv32u));
    }

    return NCV_SUCCESS;
}


template <class T>
static NCVStatus drawRectsWrapperHost(T *h_dst,
                                      Ncv32u dstStride,
                                      Ncv32u dstWidth,
                                      Ncv32u dstHeight,
                                      NcvRect32u *h_rects,
                                      Ncv32u numRects,
                                      T color)
{
    ncvAssertReturn(h_dst != NULL && h_rects != NULL, NCV_NULL_PTR);
    ncvAssertReturn(dstWidth > 0 && dstHeight > 0, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(dstStride >= dstWidth, NCV_INVALID_STEP);
    ncvAssertReturn(numRects != 0, NCV_SUCCESS);
    ncvAssertReturn(numRects <= dstWidth * dstHeight, NCV_DIMENSIONS_INVALID);

    for (Ncv32u i=0; i<numRects; i++)
    {
        NcvRect32u rect = h_rects[i];

        if (rect.x < dstWidth)
        {
            for (Ncv32u each=rect.y; each<rect.y+rect.height && each<dstHeight; each++)
            {
                h_dst[each*dstStride+rect.x] = color;
            }
        }
        if (rect.x+rect.width-1 < dstWidth)
        {
            for (Ncv32u each=rect.y; each<rect.y+rect.height && each<dstHeight; each++)
            {
                h_dst[each*dstStride+rect.x+rect.width-1] = color;
            }
        }
        if (rect.y < dstHeight)
        {
            for (Ncv32u j=rect.x; j<rect.x+rect.width && j<dstWidth; j++)
            {
                h_dst[rect.y*dstStride+j] = color;
            }
        }
        if (rect.y + rect.height - 1 < dstHeight)
        {
            for (Ncv32u j=rect.x; j<rect.x+rect.width && j<dstWidth; j++)
            {
                h_dst[(rect.y+rect.height-1)*dstStride+j] = color;
            }
        }
    }

    return NCV_SUCCESS;
}


NCVStatus ncvDrawRects_8u_host(Ncv8u *h_dst,
                               Ncv32u dstStride,
                               Ncv32u dstWidth,
                               Ncv32u dstHeight,
                               NcvRect32u *h_rects,
                               Ncv32u numRects,
                               Ncv8u color)
{
    return drawRectsWrapperHost(h_dst, dstStride, dstWidth, dstHeight, h_rects, numRects, color);
}


NCVStatus ncvDrawRects_32u_host(Ncv32u *h_dst,
                                Ncv32u dstStride,
                                Ncv32u dstWidth,
                                Ncv32u dstHeight,
                                NcvRect32u *h_rects,
                                Ncv32u numRects,
                                Ncv32u color)
{
    return drawRectsWrapperHost(h_dst, dstStride, dstWidth, dstHeight, h_rects, numRects, color);
}


const Ncv32u NUMTHREADS_DRAWRECTS = 32;
const Ncv32u NUMTHREADS_DRAWRECTS_LOG2 = 5;


template <class T>
__global__ void drawRects(T *d_dst,
                          Ncv32u dstStride,
                          Ncv32u dstWidth,
                          Ncv32u dstHeight,
                          NcvRect32u *d_rects,
                          Ncv32u numRects,
                          T color)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    if (blockId > numRects * 4)
    {
        return;
    }

    NcvRect32u curRect = d_rects[blockId >> 2];
    NcvBool bVertical = blockId & 0x1;
    NcvBool bTopLeft = blockId & 0x2;

    Ncv32u pt0x, pt0y;
    if (bVertical)
    {
        Ncv32u numChunks = (curRect.height + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = bTopLeft ? curRect.x : curRect.x + curRect.width - 1;
        pt0y = curRect.y;

        if (pt0x < dstWidth)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptY = pt0y + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptY < pt0y + curRect.height && ptY < dstHeight)
                {
                    d_dst[ptY * dstStride + pt0x] = color;
                }
            }
        }
    }
    else
    {
        Ncv32u numChunks = (curRect.width + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = curRect.x;
        pt0y = bTopLeft ? curRect.y : curRect.y + curRect.height - 1;

        if (pt0y < dstHeight)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptX = pt0x + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptX < pt0x + curRect.width && ptX < dstWidth)
                {
                    d_dst[pt0y * dstStride + ptX] = color;
                }
            }
        }
    }
}


template <class T>
static NCVStatus drawRectsWrapperDevice(T *d_dst,
                                        Ncv32u dstStride,
                                        Ncv32u dstWidth,
                                        Ncv32u dstHeight,
                                        NcvRect32u *d_rects,
                                        Ncv32u numRects,
                                        T color,
                                        hipStream_t cuStream)
{
    (void)cuStream;
    ncvAssertReturn(d_dst != NULL && d_rects != NULL, NCV_NULL_PTR);
    ncvAssertReturn(dstWidth > 0 && dstHeight > 0, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(dstStride >= dstWidth, NCV_INVALID_STEP);
    ncvAssertReturn(numRects <= dstWidth * dstHeight, NCV_DIMENSIONS_INVALID);

    if (numRects == 0)
    {
        return NCV_SUCCESS;
    }

    dim3 grid(numRects * 4);
    dim3 block(NUMTHREADS_DRAWRECTS);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }

    drawRects<T><<<grid, block>>>(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color);

    ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

    return NCV_SUCCESS;
}


NCVStatus ncvDrawRects_8u_device(Ncv8u *d_dst,
                                 Ncv32u dstStride,
                                 Ncv32u dstWidth,
                                 Ncv32u dstHeight,
                                 NcvRect32u *d_rects,
                                 Ncv32u numRects,
                                 Ncv8u color,
                                 hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}


NCVStatus ncvDrawRects_32u_device(Ncv32u *d_dst,
                                  Ncv32u dstStride,
                                  Ncv32u dstWidth,
                                  Ncv32u dstHeight,
                                  NcvRect32u *d_rects,
                                  Ncv32u numRects,
                                  Ncv32u color,
                                  hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}

#endif /* CUDA_DISABLER */
