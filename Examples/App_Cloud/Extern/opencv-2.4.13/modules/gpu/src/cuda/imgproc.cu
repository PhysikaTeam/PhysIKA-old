#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        /////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

        texture<uchar4, 2> tex_meanshift;

        __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out,
                                        size_t out_step, int cols, int rows,
                                        int sp, int sr, int maxIter, float eps)
        {
            int isr2 = sr*sr;
            uchar4 c = tex2D(tex_meanshift, x0, y0 );

            // iterate meanshift procedure
            for( int iter = 0; iter < maxIter; iter++ )
            {
                int count = 0;
                int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
                float icount;

                //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
                int minx = x0-sp;
                int miny = y0-sp;
                int maxx = x0+sp;
                int maxy = y0+sp;

                for( int y = miny; y <= maxy; y++)
                {
                    int rowCount = 0;
                    for( int x = minx; x <= maxx; x++ )
                    {
                        uchar4 t = tex2D( tex_meanshift, x, y );

                        int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                        if( norm2 <= isr2 )
                        {
                            s0 += t.x; s1 += t.y; s2 += t.z;
                            sx += x; rowCount++;
                        }
                    }
                    count += rowCount;
                    sy += y*rowCount;
                }

                if( count == 0 )
                    break;

                icount = 1.f/count;
                int x1 = __float2int_rz(sx*icount);
                int y1 = __float2int_rz(sy*icount);
                s0 = __float2int_rz(s0*icount);
                s1 = __float2int_rz(s1*icount);
                s2 = __float2int_rz(s2*icount);

                int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

                bool stopFlag = (x0 == x1 && y0 == y1) || (::abs(x1-x0) + ::abs(y1-y0) + norm2 <= eps);

                x0 = x1; y0 = y1;
                c.x = s0; c.y = s1; c.z = s2;

                if( stopFlag )
                    break;
            }

            int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
            *(uchar4*)(out + base) = c;

            return make_short2((short)x0, (short)y0);
        }

        __global__ void meanshift_kernel(unsigned char* out, size_t out_step, int cols, int rows, int sp, int sr, int maxIter, float eps )
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
                do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
        }

        __global__ void meanshiftproc_kernel(unsigned char* outr, size_t outrstep,
                                             unsigned char* outsp, size_t outspstep,
                                             int cols, int rows,
                                             int sp, int sr, int maxIter, float eps)
        {
            int x0 = blockIdx.x * blockDim.x + threadIdx.x;
            int y0 = blockIdx.y * blockDim.y + threadIdx.y;

            if( x0 < cols && y0 < rows )
            {
                int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
                *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
            }
        }

        void meanShiftFiltering_gpu(const PtrStepSzb& src, PtrStepSzb dst, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshift_kernel<<< grid, threads, 0, stream >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            //cudaSafeCall( hipUnbindTexture( tex_meanshift ) );
        }

        void meanShiftProc_gpu(const PtrStepSzb& src, PtrStepSzb dstr, PtrStepSzb dstsp, int sp, int sr, int maxIter, float eps, hipStream_t stream)
        {
            dim3 grid(1, 1, 1);
            dim3 threads(32, 8, 1);
            grid.x = divUp(src.cols, threads.x);
            grid.y = divUp(src.rows, threads.y);

            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
            cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

            meanshiftproc_kernel<<< grid, threads, 0, stream >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            //cudaSafeCall( hipUnbindTexture( tex_meanshift ) );
        }

        /////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

        template <typename T>
        __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
        {
            unsigned int H = ((ndisp-d) * 240)/ndisp;

            unsigned int hi = (H/60) % 6;
            float f = H/60.f - H/60;
            float p = V * (1 - S);
            float q = V * (1 - f * S);
            float t = V * (1 - (1 - f) * S);

            float3 res;

            if (hi == 0) //R = V,	G = t,	B = p
            {
                res.x = p;
                res.y = t;
                res.z = V;
            }

            if (hi == 1) // R = q,	G = V,	B = p
            {
                res.x = p;
                res.y = V;
                res.z = q;
            }

            if (hi == 2) // R = p,	G = V,	B = t
            {
                res.x = t;
                res.y = V;
                res.z = p;
            }

            if (hi == 3) // R = p,	G = q,	B = V
            {
                res.x = V;
                res.y = q;
                res.z = p;
            }

            if (hi == 4) // R = t,	G = p,	B = V
            {
                res.x = V;
                res.y = p;
                res.z = t;
            }

            if (hi == 5) // R = V,	G = p,	B = q
            {
                res.x = q;
                res.y = p;
                res.z = V;
            }
            const unsigned int b = (unsigned int)(::max(0.f, ::min(res.x, 1.f)) * 255.f);
            const unsigned int g = (unsigned int)(::max(0.f, ::min(res.y, 1.f)) * 255.f);
            const unsigned int r = (unsigned int)(::max(0.f, ::min(res.z, 1.f)) * 255.f);
            const unsigned int a = 255U;

            return (a << 24) + (r << 16) + (g << 8) + b;
        }

        __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if(x < width && y < height)
            {
                uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

                uint4 res;
                res.x = cvtPixel(d4.x, ndisp);
                res.y = cvtPixel(d4.y, ndisp);
                res.z = cvtPixel(d4.z, ndisp);
                res.w = cvtPixel(d4.w, ndisp);

                uint4* line = (uint4*)(out_image + y * out_step);
                line[x >> 2] = res;
            }
        }

        __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if(x < width && y < height)
            {
                short2 d2 = *(short2*)(disp + y * disp_step + x);

                uint2 res;
                res.x = cvtPixel(d2.x, ndisp);
                res.y = cvtPixel(d2.y, ndisp);

                uint2* line = (uint2*)(out_image + y * out_step);
                line[x >> 1] = res;
            }
        }


        void drawColorDisp_gpu(const PtrStepSzb& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
        {
            dim3 threads(16, 16, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(src.cols, threads.x << 2);
            grid.y = divUp(src.rows, threads.y);

            drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void drawColorDisp_gpu(const PtrStepSz<short>& src, const PtrStepSzb& dst, int ndisp, const hipStream_t& stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(src.cols, threads.x << 1);
            grid.y = divUp(src.rows, threads.y);

            drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        /////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

        __constant__ float cq[16];

        template <typename T, typename D>
        __global__ void reprojectImageTo3D(const PtrStepSz<T> disp, PtrStep<D> xyz)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= disp.rows || x >= disp.cols)
                return;

            const float qx = x * cq[ 0] + y * cq[ 1] + cq[ 3];
            const float qy = x * cq[ 4] + y * cq[ 5] + cq[ 7];
            const float qz = x * cq[ 8] + y * cq[ 9] + cq[11];
            const float qw = x * cq[12] + y * cq[13] + cq[15];

            const T d = disp(y, x);

            const float iW = 1.f / (qw + cq[14] * d);

            D v = VecTraits<D>::all(1.0f);
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;

            xyz(y, x) = v;
        }

        template <typename T, typename D>
        void reprojectImageTo3D_gpu(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(disp.cols, block.x), divUp(disp.rows, block.y));

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

            reprojectImageTo3D<T, D><<<grid, block, 0, stream>>>((PtrStepSz<T>)disp, (PtrStepSz<D>)xyz);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void reprojectImageTo3D_gpu<uchar, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<uchar, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<short, float3>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);
        template void reprojectImageTo3D_gpu<short, float4>(const PtrStepSzb disp, PtrStepSzb xyz, const float* q, hipStream_t stream);

        /////////////////////////////////////////// Corner Harris /////////////////////////////////////////////////

        texture<float, hipTextureType2D, hipReadModeElementType> harrisDxTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<float, hipTextureType2D, hipReadModeElementType> harrisDyTex(0, hipFilterModePoint, hipAddressModeClamp);

        __global__ void cornerHarris_kernel(const int block_size, const float k, PtrStepSzf dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    for (int j = jbegin; j < jend; ++j)
                    {
                        float dx = tex2D(harrisDxTex, j, i);
                        float dy = tex2D(harrisDyTex, j, i);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                dst(y, x) = a * c - b * b - k * (a + c) * (a + c);
            }
        }

        template <typename BR, typename BC>
        __global__ void cornerHarris_kernel(const int block_size, const float k, PtrStepSzf dst, const BR border_row, const BC border_col)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    const int y = border_col.idx_row(i);

                    for (int j = jbegin; j < jend; ++j)
                    {
                        const int x = border_row.idx_col(j);

                        float dx = tex2D(harrisDxTex, x, y);
                        float dy = tex2D(harrisDyTex, x, y);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                dst(y, x) = a * c - b * b - k * (a + c) * (a + c);
            }
        }

        void cornerHarris_gpu(int block_size, float k, PtrStepSzf Dx, PtrStepSzf Dy, PtrStepSzf dst, int border_type, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(Dx.cols, block.x), divUp(Dx.rows, block.y));

            bindTexture(&harrisDxTex, Dx);
            bindTexture(&harrisDyTex, Dy);

            switch (border_type)
            {
            case BORDER_REFLECT101_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst, BrdRowReflect101<void>(Dx.cols), BrdColReflect101<void>(Dx.rows));
                break;

            case BORDER_REFLECT_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst, BrdRowReflect<void>(Dx.cols), BrdColReflect<void>(Dx.rows));
                break;

            case BORDER_REPLICATE_GPU:
                cornerHarris_kernel<<<grid, block, 0, stream>>>(block_size, k, dst);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        /////////////////////////////////////////// Corner Min Eigen Val /////////////////////////////////////////////////

        texture<float, hipTextureType2D, hipReadModeElementType> minEigenValDxTex(0, hipFilterModePoint, hipAddressModeClamp);
        texture<float, hipTextureType2D, hipReadModeElementType> minEigenValDyTex(0, hipFilterModePoint, hipAddressModeClamp);

        __global__ void cornerMinEigenVal_kernel(const int block_size, PtrStepSzf dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    for (int j = jbegin; j < jend; ++j)
                    {
                        float dx = tex2D(minEigenValDxTex, j, i);
                        float dy = tex2D(minEigenValDyTex, j, i);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                a *= 0.5f;
                c *= 0.5f;

                dst(y, x) = (a + c) - sqrtf((a - c) * (a - c) + b * b);
            }
        }


        template <typename BR, typename BC>
        __global__ void cornerMinEigenVal_kernel(const int block_size, PtrStepSzf dst, const BR border_row, const BC border_col)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                float a = 0.f;
                float b = 0.f;
                float c = 0.f;

                const int ibegin = y - (block_size / 2);
                const int jbegin = x - (block_size / 2);
                const int iend = ibegin + block_size;
                const int jend = jbegin + block_size;

                for (int i = ibegin; i < iend; ++i)
                {
                    int y = border_col.idx_row(i);

                    for (int j = jbegin; j < jend; ++j)
                    {
                        int x = border_row.idx_col(j);

                        float dx = tex2D(minEigenValDxTex, x, y);
                        float dy = tex2D(minEigenValDyTex, x, y);

                        a += dx * dx;
                        b += dx * dy;
                        c += dy * dy;
                    }
                }

                a *= 0.5f;
                c *= 0.5f;

                dst(y, x) = (a + c) - sqrtf((a - c) * (a - c) + b * b);
            }
        }

        void cornerMinEigenVal_gpu(int block_size, PtrStepSzf Dx, PtrStepSzf Dy, PtrStepSzf dst, int border_type, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(Dx.cols, block.x), divUp(Dx.rows, block.y));

            bindTexture(&minEigenValDxTex, Dx);
            bindTexture(&minEigenValDyTex, Dy);

            switch (border_type)
            {
            case BORDER_REFLECT101_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst, BrdRowReflect101<void>(Dx.cols), BrdColReflect101<void>(Dx.rows));
                break;

            case BORDER_REFLECT_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst, BrdRowReflect<void>(Dx.cols), BrdColReflect<void>(Dx.rows));
                break;

            case BORDER_REPLICATE_GPU:
                cornerMinEigenVal_kernel<<<grid, block, 0, stream>>>(block_size, dst);
                break;
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }

        ////////////////////////////// Column Sum //////////////////////////////////////

        __global__ void column_sumKernel_32F(int cols, int rows, const PtrStepb src, const PtrStepb dst)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;

            if (x < cols)
            {
                const unsigned char* src_data = src.data + x * sizeof(float);
                unsigned char* dst_data = dst.data + x * sizeof(float);

                float sum = 0.f;
                for (int y = 0; y < rows; ++y)
                {
                    sum += *(const float*)src_data;
                    *(float*)dst_data = sum;
                    src_data += src.step;
                    dst_data += dst.step;
                }
            }
        }


        void columnSum_32F(const PtrStepSzb src, const PtrStepSzb dst)
        {
            dim3 threads(256);
            dim3 grid(divUp(src.cols, threads.x));

            column_sumKernel_32F<<<grid, threads>>>(src.cols, src.rows, src, dst);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }


        //////////////////////////////////////////////////////////////////////////
        // mulSpectrums

#ifdef HAVE_CUFFT
        __global__ void mulSpectrumsKernel(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < c.cols && y < c.rows)
            {
                c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
            }
        }


        void mulSpectrums(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c, hipStream_t stream)
        {
            dim3 threads(256);
            dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

            mulSpectrumsKernel<<<grid, threads, 0, stream>>>(a, b, c);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
#endif


        //////////////////////////////////////////////////////////////////////////
        // mulSpectrums_CONJ

#ifdef HAVE_CUFFT
        __global__ void mulSpectrumsKernel_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < c.cols && y < c.rows)
            {
                c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
            }
        }


        void mulSpectrums_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c, hipStream_t stream)
        {
            dim3 threads(256);
            dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

            mulSpectrumsKernel_CONJ<<<grid, threads, 0, stream>>>(a, b, c);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
#endif


        //////////////////////////////////////////////////////////////////////////
        // mulAndScaleSpectrums

#ifdef HAVE_CUFFT
        __global__ void mulAndScaleSpectrumsKernel(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < c.cols && y < c.rows)
            {
                hipfftComplex v = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
                c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
            }
        }


        void mulAndScaleSpectrums(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c, hipStream_t stream)
        {
            dim3 threads(256);
            dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

            mulAndScaleSpectrumsKernel<<<grid, threads, 0, stream>>>(a, b, scale, c);
            cudaSafeCall( hipGetLastError() );

            if (stream)
                cudaSafeCall( hipDeviceSynchronize() );
        }
#endif


        //////////////////////////////////////////////////////////////////////////
        // mulAndScaleSpectrums_CONJ

#ifdef HAVE_CUFFT
        __global__ void mulAndScaleSpectrumsKernel_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < c.cols && y < c.rows)
            {
                hipfftComplex v = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
                c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
            }
        }


        void mulAndScaleSpectrums_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c, hipStream_t stream)
        {
            dim3 threads(256);
            dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

            mulAndScaleSpectrumsKernel_CONJ<<<grid, threads, 0, stream>>>(a, b, scale, c);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
#endif

        //////////////////////////////////////////////////////////////////////////
        // buildWarpMaps

        // TODO use intrinsics like __sinf and so on

        namespace build_warp_maps
        {

            __constant__ float ck_rinv[9];
            __constant__ float cr_kinv[9];
            __constant__ float ct[3];
            __constant__ float cscale;
        }


        class PlaneMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                float x_ = u / cscale - ct[0];
                float y_ = v / cscale - ct[1];

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * (1 - ct[2]);
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * (1 - ct[2]);
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * (1 - ct[2]);

                x /= z;
                y /= z;
            }
        };


        class CylindricalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                u /= cscale;
                float x_ = ::sinf(u);
                float y_ = v / cscale;
                float z_ = ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        class SphericalMapper
        {
        public:
            static __device__ __forceinline__ void mapBackward(float u, float v, float &x, float &y)
            {
                using namespace build_warp_maps;

                v /= cscale;
                u /= cscale;

                float sinv = ::sinf(v);
                float x_ = sinv * ::sinf(u);
                float y_ = -::cosf(v);
                float z_ = sinv * ::cosf(u);

                float z;
                x = ck_rinv[0] * x_ + ck_rinv[1] * y_ + ck_rinv[2] * z_;
                y = ck_rinv[3] * x_ + ck_rinv[4] * y_ + ck_rinv[5] * z_;
                z = ck_rinv[6] * x_ + ck_rinv[7] * y_ + ck_rinv[8] * z_;

                if (z > 0) { x /= z; y /= z; }
                else x = y = -1;
            }
        };


        template <typename Mapper>
        __global__ void buildWarpMapsKernel(int tl_u, int tl_v, int cols, int rows,
                                            PtrStepf map_x, PtrStepf map_y)
        {
            int du = blockIdx.x * blockDim.x + threadIdx.x;
            int dv = blockIdx.y * blockDim.y + threadIdx.y;
            if (du < cols && dv < rows)
            {
                float u = tl_u + du;
                float v = tl_v + dv;
                float x, y;
                Mapper::mapBackward(u, v, x, y);
                map_x.ptr(dv)[du] = x;
                map_y.ptr(dv)[du] = y;
            }
        }


        void buildWarpPlaneMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                const float k_rinv[9], const float r_kinv[9], const float t[3],
                                float scale, hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ct), t, 3*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<PlaneMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpCylindricalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                      const float k_rinv[9], const float r_kinv[9], float scale,
                                      hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<CylindricalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void buildWarpSphericalMaps(int tl_u, int tl_v, PtrStepSzf map_x, PtrStepSzf map_y,
                                    const float k_rinv[9], const float r_kinv[9], float scale,
                                    hipStream_t stream)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::ck_rinv), k_rinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cr_kinv), r_kinv, 9*sizeof(float)));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(build_warp_maps::cscale), &scale, sizeof(float)));

            int cols = map_x.cols;
            int rows = map_x.rows;

            dim3 threads(32, 8);
            dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

            buildWarpMapsKernel<SphericalMapper><<<grid,threads>>>(tl_u, tl_v, cols, rows, map_x, map_y);
            cudaSafeCall(hipGetLastError());
            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }

        //////////////////////////////////////////////////////////////////////////
        // filter2D

        #define FILTER2D_MAX_KERNEL_SIZE 16

        __constant__ float c_filter2DKernel[FILTER2D_MAX_KERNEL_SIZE * FILTER2D_MAX_KERNEL_SIZE];

        template <class SrcT, typename D>
        __global__ void filter2D(const SrcT src, PtrStepSz<D> dst, const int kWidth, const int kHeight, const int anchorX, const int anchorY)
        {
            typedef typename TypeVec<float, VecTraits<D>::cn>::vec_type sum_t;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= dst.cols || y >= dst.rows)
                return;

            sum_t res = VecTraits<sum_t>::all(0);
            int kInd = 0;

            for (int i = 0; i < kHeight; ++i)
            {
                for (int j = 0; j < kWidth; ++j)
                    res = res + src(y - anchorY + i, x - anchorX + j) * c_filter2DKernel[kInd++];
            }

            dst(y, x) = saturate_cast<D>(res);
        }

        template <typename T, typename D, template <typename> class Brd> struct Filter2DCaller;

        #define IMPLEMENT_FILTER2D_TEX_READER(type) \
            texture< type , hipTextureType2D, hipReadModeElementType> tex_filter2D_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_filter2D_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                const int xoff; \
                const int yoff; \
                tex_filter2D_ ## type ## _reader (int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_filter2D_ ## type , x + xoff, y + yoff); \
                } \
            }; \
            template <typename D, template <typename> class Brd> struct Filter2DCaller< type , D, Brd> \
            { \
                static void call(const PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSz<D> dst, \
                    int kWidth, int kHeight, int anchorX, int anchorY, const float* borderValue, hipStream_t stream) \
                { \
                    typedef typename TypeVec<float, VecTraits< type >::cn>::vec_type work_type; \
                    dim3 block(16, 16); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_filter2D_ ## type , srcWhole); \
                    tex_filter2D_ ## type ##_reader texSrc(xoff, yoff); \
                    Brd<work_type> brd(dst.rows, dst.cols, VecTraits<work_type>::make(borderValue)); \
                    BorderReader< tex_filter2D_ ## type ##_reader, Brd<work_type> > brdSrc(texSrc, brd); \
                    filter2D<<<grid, block, 0, stream>>>(brdSrc, dst, kWidth, kHeight, anchorX, anchorY); \
                    cudaSafeCall( hipGetLastError() ); \
                    if (stream == 0) \
                        cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        IMPLEMENT_FILTER2D_TEX_READER(uchar);
        IMPLEMENT_FILTER2D_TEX_READER(uchar4);

        IMPLEMENT_FILTER2D_TEX_READER(ushort);
        IMPLEMENT_FILTER2D_TEX_READER(ushort4);

        IMPLEMENT_FILTER2D_TEX_READER(float);
        IMPLEMENT_FILTER2D_TEX_READER(float4);

        #undef IMPLEMENT_FILTER2D_TEX_READER

        template <typename T, typename D>
        void filter2D_gpu(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst,
                          int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel,
                          int borderMode, const float* borderValue, hipStream_t stream)
        {
            typedef void (*func_t)(const PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<D> dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* borderValue, hipStream_t stream);
#ifdef OPENCV_TINY_GPU_MODULE
            static const func_t funcs[] =
            {
                Filter2DCaller<T, D, BrdReflect101>::call,
                Filter2DCaller<T, D, BrdReplicate>::call,
                Filter2DCaller<T, D, BrdConstant>::call,
                Filter2DCaller<T, D, BrdReflect>::call,
                0
            };
#else
            static const func_t funcs[] =
            {
                Filter2DCaller<T, D, BrdReflect101>::call,
                Filter2DCaller<T, D, BrdReplicate>::call,
                Filter2DCaller<T, D, BrdConstant>::call,
                Filter2DCaller<T, D, BrdReflect>::call,
                Filter2DCaller<T, D, BrdWrap>::call
            };
#endif

            const func_t func = funcs[borderMode];
            if (!func)
                cv::gpu::error("Unsupported input parameters for filter2D", __FILE__, __LINE__, "");

            if (stream == 0)
                cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_filter2DKernel), kernel, kWidth * kHeight * sizeof(float), 0, hipMemcpyDeviceToDevice) );
            else
                cudaSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(c_filter2DKernel), kernel, kWidth * kHeight * sizeof(float), 0, hipMemcpyDeviceToDevice, stream) );

            func(static_cast< PtrStepSz<T> >(srcWhole), ofsX, ofsY, static_cast< PtrStepSz<D> >(dst), kWidth, kHeight, anchorX, anchorY, borderValue, stream);
        }

        template void filter2D_gpu<uchar, uchar>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
        template void filter2D_gpu<uchar4, uchar4>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
        template void filter2D_gpu<ushort, ushort>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
        template void filter2D_gpu<ushort4, ushort4>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
#endif
        template void filter2D_gpu<float, float>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
        template void filter2D_gpu<float4, float4>(PtrStepSzb srcWhole, int ofsX, int ofsY, PtrStepSzb dst, int kWidth, int kHeight, int anchorX, int anchorY, const float* kernel, int borderMode, const float* borderValue, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device {


#endif /* CUDA_DISABLER */
