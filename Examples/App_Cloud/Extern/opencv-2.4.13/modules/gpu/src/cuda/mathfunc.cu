#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace mathfunc
    {
        //////////////////////////////////////////////////////////////////////////////////////
        // Cart <-> Polar

        struct Nothing
        {
            static __device__ __forceinline__ void calc(int, int, float, float, float*, size_t, float)
            {
            }
        };
        struct Magnitude
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
            {
                dst[y * dst_step + x] = ::sqrtf(x_data * x_data + y_data * y_data);
            }
        };
        struct MagnitudeSqr
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
            {
                dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
            }
        };
        struct Atan2
        {
            static __device__ __forceinline__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
            {
                float angle = ::atan2f(y_data, x_data);
                angle += (angle < 0) * 2.0f * CV_PI_F;
                dst[y * dst_step + x] = scale * angle;
            }
        };
        template <typename Mag, typename Angle>
        __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step,
                                    float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < width && y < height)
            {
                float x_data = xptr[y * x_step + x];
                float y_data = yptr[y * y_step + x];

                Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
                Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
            }
        }

        struct NonEmptyMag
        {
            static __device__ __forceinline__ float get(const float* mag, size_t mag_step, int x, int y)
            {
                return mag[y * mag_step + x];
            }
        };
        struct EmptyMag
        {
            static __device__ __forceinline__ float get(const float*, size_t, int, int)
            {
                return 1.0f;
            }
        };
        template <typename Mag>
        __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
            float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < width && y < height)
            {
                float mag_data = Mag::get(mag, mag_step, x, y);
                float angle_data = angle[y * angle_step + x];
                float sin_a, cos_a;

                ::sincosf(scale * angle_data, &sin_a, &cos_a);

                xptr[y * x_step + x] = mag_data * cos_a;
                yptr[y * y_step + x] = mag_data * sin_a;
            }
        }

        template <typename Mag, typename Angle>
        void cartToPolar_caller(PtrStepSzf x, PtrStepSzf y, PtrStepSzf mag, PtrStepSzf angle, bool angleInDegrees, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(x.cols, threads.x);
            grid.y = divUp(x.rows, threads.y);

            const float scale = angleInDegrees ? (180.0f / CV_PI_F) : 1.f;

            cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
                x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(),
                mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void cartToPolar_gpu(PtrStepSzf x, PtrStepSzf y, PtrStepSzf mag, bool magSqr, PtrStepSzf angle, bool angleInDegrees, hipStream_t stream)
        {
            typedef void (*caller_t)(PtrStepSzf x, PtrStepSzf y, PtrStepSzf mag, PtrStepSzf angle, bool angleInDegrees, hipStream_t stream);
            static const caller_t callers[2][2][2] =
            {
                {
                    {
                        cartToPolar_caller<Magnitude, Atan2>,
                        cartToPolar_caller<Magnitude, Nothing>
                    },
                    {
                        cartToPolar_caller<MagnitudeSqr, Atan2>,
                        cartToPolar_caller<MagnitudeSqr, Nothing>,
                    }
                },
                {
                    {
                        cartToPolar_caller<Nothing, Atan2>,
                        cartToPolar_caller<Nothing, Nothing>
                    },
                    {
                        cartToPolar_caller<Nothing, Atan2>,
                        cartToPolar_caller<Nothing, Nothing>,
                    }
                }
            };

            callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
        }

        template <typename Mag>
        void polarToCart_caller(PtrStepSzf mag, PtrStepSzf angle, PtrStepSzf x, PtrStepSzf y, bool angleInDegrees, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(mag.cols, threads.x);
            grid.y = divUp(mag.rows, threads.y);

            const float scale = angleInDegrees ? (CV_PI_F / 180.0f) : 1.0f;

            polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(),
                angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void polarToCart_gpu(PtrStepSzf mag, PtrStepSzf angle, PtrStepSzf x, PtrStepSzf y, bool angleInDegrees, hipStream_t stream)
        {
            typedef void (*caller_t)(PtrStepSzf mag, PtrStepSzf angle, PtrStepSzf x, PtrStepSzf y, bool angleInDegrees, hipStream_t stream);
            static const caller_t callers[2] =
            {
                polarToCart_caller<NonEmptyMag>,
                polarToCart_caller<EmptyMag>
            };

            callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
        }
    } // namespace mathfunc
}}} // namespace cv { namespace gpu { namespace device

#endif /* CUDA_DISABLER */
