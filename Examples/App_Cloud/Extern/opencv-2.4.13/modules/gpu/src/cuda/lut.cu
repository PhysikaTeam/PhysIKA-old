#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <cstring>
#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/functional.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace
{
    texture<uchar, hipTextureType1D, hipReadModeElementType> texLutTable;

    struct LutC1 : public unary_function<uchar, uchar>
    {
        typedef uchar value_type;
        typedef uchar index_type;

        hipTextureObject_t texLutTableObj;

        __device__ __forceinline__ uchar operator ()(uchar x) const
        {
        #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 300)
            // Use the texture reference
            return tex1Dfetch(texLutTable, x);
        #else
            // Use the texture object
            return tex1Dfetch<uchar>(texLutTableObj, x);
        #endif
        }
    };
    struct LutC3 : public unary_function<uchar3, uchar3>
    {
        typedef uchar3 value_type;
        typedef uchar3 index_type;

        hipTextureObject_t texLutTableObj;

        __device__ __forceinline__ uchar3 operator ()(const uchar3& x) const
        {
        #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 300)
            // Use the texture reference
            return make_uchar3(tex1Dfetch(texLutTable, x.x * 3), tex1Dfetch(texLutTable, x.y * 3 + 1), tex1Dfetch(texLutTable, x.z * 3 + 2));
        #else
            // Use the texture object
            return make_uchar3(tex1Dfetch<uchar>(texLutTableObj, x.x * 3), tex1Dfetch<uchar>(texLutTableObj, x.y * 3 + 1), tex1Dfetch<uchar>(texLutTableObj, x.z * 3 + 2));
        #endif
        }
    };
}

namespace arithm
{
    void lut(PtrStepSzb src, uchar* lut, int lut_cn, PtrStepSzb dst, bool cc30, hipStream_t stream)
    {
        hipTextureObject_t texLutTableObj;

        if (cc30)
        {
            // Use the texture object
            hipResourceDesc texRes;
            std::memset(&texRes, 0, sizeof(texRes));
            texRes.resType = hipResourceTypeLinear;
            texRes.res.linear.devPtr = lut;
            texRes.res.linear.desc = hipCreateChannelDesc<uchar>();
            texRes.res.linear.sizeInBytes = 256 * lut_cn * sizeof(uchar);

            hipTextureDesc texDescr;
            std::memset(&texDescr, 0, sizeof(texDescr));

            cudaSafeCall( hipCreateTextureObject(&texLutTableObj, &texRes, &texDescr, 0) );
        }
        else
        {
            // Use the texture reference
            hipChannelFormatDesc desc = hipCreateChannelDesc<uchar>();
            cudaSafeCall( hipBindTexture(0, &texLutTable, lut, &desc) );
        }

        if (lut_cn == 1)
        {
            LutC1 op;
            op.texLutTableObj = texLutTableObj;

            transform((PtrStepSz<uchar>) src, (PtrStepSz<uchar>) dst, op, WithOutMask(), stream);
        }
        else if (lut_cn == 3)
        {
            LutC3 op;
            op.texLutTableObj = texLutTableObj;

            transform((PtrStepSz<uchar3>) src, (PtrStepSz<uchar3>) dst, op, WithOutMask(), stream);
        }

        if (cc30)
        {
            // Use the texture object
            cudaSafeCall( hipDestroyTextureObject(texLutTableObj) );
        }
        else
        {
            // Use the texture reference
            cudaSafeCall( hipUnbindTexture(texLutTable) );
        }
    }
}

#endif
