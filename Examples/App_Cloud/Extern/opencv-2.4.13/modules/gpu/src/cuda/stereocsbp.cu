#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/reduce.hpp"
#include "opencv2/gpu/device/functional.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace stereocsbp
    {
        ///////////////////////////////////////////////////////////////
        /////////////////////// load constants ////////////////////////
        ///////////////////////////////////////////////////////////////

        __constant__ int cndisp;

        __constant__ float cmax_data_term;
        __constant__ float cdata_weight;
        __constant__ float cmax_disc_term;
        __constant__ float cdisc_single_jump;

        __constant__ int cth;

        __constant__ size_t cimg_step;
        __constant__ size_t cmsg_step;
        __constant__ size_t cdisp_step1;
        __constant__ size_t cdisp_step2;

        __constant__ uchar* cleft;
        __constant__ uchar* cright;
        __constant__ uchar* ctemp;


        void load_constants(int ndisp, float max_data_term, float data_weight, float max_disc_term, float disc_single_jump, int min_disp_th,
                            const PtrStepSzb& left, const PtrStepSzb& right, const PtrStepSzb& temp)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cndisp), &ndisp, sizeof(int)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_data_term),    &max_data_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdata_weight),      &data_weight,      sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_disc_term),    &max_disc_term,    sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisc_single_jump), &disc_single_jump, sizeof(float)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cth), &min_disp_th, sizeof(int)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cimg_step), &left.step, sizeof(size_t)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cleft),  &left.data,  sizeof(left.data)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cright), &right.data, sizeof(right.data)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(ctemp), &temp.data, sizeof(temp.data)) );
        }

        ///////////////////////////////////////////////////////////////
        /////////////////////// init data cost ////////////////////////
        ///////////////////////////////////////////////////////////////

        template <int channels> struct DataCostPerPixel;
        template <> struct DataCostPerPixel<1>
        {
            static __device__ __forceinline__ float compute(const uchar* left, const uchar* right)
            {
                int l = *(left);
                int r = *(right);

                return fmin(cdata_weight * ::abs(l - r), cdata_weight * cmax_data_term);
            }
        };
        template <> struct DataCostPerPixel<3>
        {
            static __device__ __forceinline__ float compute(const uchar* left, const uchar* right)
            {
                uchar3 l = *((const uchar3*)left);
                uchar3 r = *((const uchar3*)right);

                float tb = 0.114f * ::abs((int)l.x - r.x);
                float tg = 0.587f * ::abs((int)l.y - r.y);
                float tr = 0.299f * ::abs((int)l.z - r.z);

                return fmin(cdata_weight * (tr + tg + tb), cdata_weight * cmax_data_term);
            }
        };
        template <> struct DataCostPerPixel<4>
        {
            static __device__ __forceinline__ float compute(const uchar* left, const uchar* right)
            {
                uchar4 l = *((const uchar4*)left);
                uchar4 r = *((const uchar4*)right);

                float tb = 0.114f * ::abs((int)l.x - r.x);
                float tg = 0.587f * ::abs((int)l.y - r.y);
                float tr = 0.299f * ::abs((int)l.z - r.z);

                return fmin(cdata_weight * (tr + tg + tb), cdata_weight * cmax_data_term);
            }
        };

        template <typename T>
        __global__ void get_first_k_initial_global(T* data_cost_selected_, T *selected_disp_pyr, int h, int w, int nr_plane)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                T* selected_disparity = selected_disp_pyr + y * cmsg_step + x;
                T* data_cost_selected = data_cost_selected_ + y * cmsg_step + x;
                T* data_cost = (T*)ctemp + y * cmsg_step + x;

                for(int i = 0; i < nr_plane; i++)
                {
                    T minimum = device::numeric_limits<T>::max();
                    int id = 0;
                    for(int d = 0; d < cndisp; d++)
                    {
                        T cur = data_cost[d * cdisp_step1];
                        if(cur < minimum)
                        {
                            minimum = cur;
                            id = d;
                        }
                    }

                    data_cost_selected[i  * cdisp_step1] = minimum;
                    selected_disparity[i  * cdisp_step1] = id;
                    data_cost         [id * cdisp_step1] = numeric_limits<T>::max();
                }
            }
        }


        template <typename T>
        __global__ void get_first_k_initial_local(T* data_cost_selected_, T* selected_disp_pyr, int h, int w, int nr_plane)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                T* selected_disparity = selected_disp_pyr + y * cmsg_step + x;
                T* data_cost_selected = data_cost_selected_ + y * cmsg_step + x;
                T* data_cost = (T*)ctemp + y * cmsg_step + x;

                int nr_local_minimum = 0;

                T prev = data_cost[0 * cdisp_step1];
                T cur  = data_cost[1 * cdisp_step1];
                T next = data_cost[2 * cdisp_step1];

                for (int d = 1; d < cndisp - 1 && nr_local_minimum < nr_plane; d++)
                {
                    if (cur < prev && cur < next)
                    {
                        data_cost_selected[nr_local_minimum * cdisp_step1] = cur;
                        selected_disparity[nr_local_minimum * cdisp_step1] = d;

                        data_cost[d * cdisp_step1] = numeric_limits<T>::max();

                        nr_local_minimum++;
                    }
                    prev = cur;
                    cur = next;
                    next = data_cost[(d + 1) * cdisp_step1];
                }

                for (int i = nr_local_minimum; i < nr_plane; i++)
                {
                    T minimum = numeric_limits<T>::max();
                    int id = 0;

                    for (int d = 0; d < cndisp; d++)
                    {
                        cur = data_cost[d * cdisp_step1];
                        if (cur < minimum)
                        {
                            minimum = cur;
                            id = d;
                        }
                    }
                    data_cost_selected[i * cdisp_step1] = minimum;
                    selected_disparity[i * cdisp_step1] = id;

                    data_cost[id * cdisp_step1] = numeric_limits<T>::max();
                }
            }
        }

        template <typename T, int channels>
        __global__ void init_data_cost(int h, int w, int level)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                int y0 = y << level;
                int yt = (y + 1) << level;

                int x0 = x << level;
                int xt = (x + 1) << level;

                T* data_cost = (T*)ctemp + y * cmsg_step + x;

                for(int d = 0; d < cndisp; ++d)
                {
                    float val = 0.0f;
                    for(int yi = y0; yi < yt; yi++)
                    {
                        for(int xi = x0; xi < xt; xi++)
                        {
                            int xr = xi - d;
                            if(d < cth || xr < 0)
                                val += cdata_weight * cmax_data_term;
                            else
                            {
                                const uchar* lle = cleft + yi * cimg_step + xi * channels;
                                const uchar* lri = cright + yi * cimg_step + xr * channels;

                                val += DataCostPerPixel<channels>::compute(lle, lri);
                            }
                        }
                    }
                    data_cost[cdisp_step1 * d] = saturate_cast<T>(val);
                }
            }
        }

        template <typename T, int winsz, int channels>
        __global__ void init_data_cost_reduce(int level, int rows, int cols, int h)
        {
            int x_out = blockIdx.x;
            int y_out = blockIdx.y % h;
            int d = (blockIdx.y / h) * blockDim.z + threadIdx.z;

            int tid = threadIdx.x;

            if (d < cndisp)
            {
                int x0 = x_out << level;
                int y0 = y_out << level;

                int len = ::min(y0 + winsz, rows) - y0;

                float val = 0.0f;
                if (x0 + tid < cols)
                {
                    if (x0 + tid - d < 0 || d < cth)
                        val = cdata_weight * cmax_data_term * len;
                    else
                    {
                        const uchar* lle =  cleft + y0 * cimg_step + channels * (x0 + tid    );
                        const uchar* lri = cright + y0 * cimg_step + channels * (x0 + tid - d);

                        for(int y = 0; y < len; ++y)
                        {
                            val += DataCostPerPixel<channels>::compute(lle, lri);

                            lle += cimg_step;
                            lri += cimg_step;
                        }
                    }
                }

                extern __shared__ float smem[];

                reduce<winsz>(smem + winsz * threadIdx.z, val, tid, plus<float>());

                T* data_cost = (T*)ctemp + y_out * cmsg_step + x_out;

                if (tid == 0)
                    data_cost[cdisp_step1 * d] = saturate_cast<T>(val);
            }
        }


        template <typename T>
        void init_data_cost_caller_(int /*rows*/, int /*cols*/, int h, int w, int level, int /*ndisp*/, int channels, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            switch (channels)
            {
            case 1: init_data_cost<T, 1><<<grid, threads, 0, stream>>>(h, w, level); break;
            case 3: init_data_cost<T, 3><<<grid, threads, 0, stream>>>(h, w, level); break;
            case 4: init_data_cost<T, 4><<<grid, threads, 0, stream>>>(h, w, level); break;
            default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__, "init_data_cost_caller_");
            }
        }

        template <typename T, int winsz>
        void init_data_cost_reduce_caller_(int rows, int cols, int h, int w, int level, int ndisp, int channels, hipStream_t stream)
        {
            const int threadsNum = 256;
            const size_t smem_size = threadsNum * sizeof(float);

            dim3 threads(winsz, 1, threadsNum / winsz);
            dim3 grid(w, h, 1);
            grid.y *= divUp(ndisp, threads.z);

            switch (channels)
            {
            case 1: init_data_cost_reduce<T, winsz, 1><<<grid, threads, smem_size, stream>>>(level, rows, cols, h); break;
            case 3: init_data_cost_reduce<T, winsz, 3><<<grid, threads, smem_size, stream>>>(level, rows, cols, h); break;
            case 4: init_data_cost_reduce<T, winsz, 4><<<grid, threads, smem_size, stream>>>(level, rows, cols, h); break;
            default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__, "init_data_cost_reduce_caller_");
            }
        }

        template<class T>
        void init_data_cost(int rows, int cols, T* disp_selected_pyr, T* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, bool use_local_init_data_cost, hipStream_t stream)
        {

            typedef void (*InitDataCostCaller)(int cols, int rows, int w, int h, int level, int ndisp, int channels, hipStream_t stream);

            static const InitDataCostCaller init_data_cost_callers[] =
            {
                init_data_cost_caller_<T>, init_data_cost_caller_<T>, init_data_cost_reduce_caller_<T, 4>,
                init_data_cost_reduce_caller_<T, 8>, init_data_cost_reduce_caller_<T, 16>, init_data_cost_reduce_caller_<T, 32>,
                init_data_cost_reduce_caller_<T, 64>, init_data_cost_reduce_caller_<T, 128>, init_data_cost_reduce_caller_<T, 256>
            };

            size_t disp_step = msg_step * h;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step1), &disp_step, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmsg_step),  &msg_step,  sizeof(size_t)) );

            init_data_cost_callers[level](rows, cols, h, w, level, ndisp, channels, stream);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            if (use_local_init_data_cost == true)
                get_first_k_initial_local<<<grid, threads, 0, stream>>> (data_cost_selected, disp_selected_pyr, h, w, nr_plane);
            else
                get_first_k_initial_global<<<grid, threads, 0, stream>>>(data_cost_selected, disp_selected_pyr, h, w, nr_plane);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void init_data_cost(int rows, int cols, short* disp_selected_pyr, short* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, bool use_local_init_data_cost, hipStream_t stream);

        template void init_data_cost(int rows, int cols, float* disp_selected_pyr, float* data_cost_selected, size_t msg_step,
                    int h, int w, int level, int nr_plane, int ndisp, int channels, bool use_local_init_data_cost, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////// compute data cost //////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T, int channels>
        __global__ void compute_data_cost(const T* selected_disp_pyr, T* data_cost_, int h, int w, int level, int nr_plane)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                int y0 = y << level;
                int yt = (y + 1) << level;

                int x0 = x << level;
                int xt = (x + 1) << level;

                const T* selected_disparity = selected_disp_pyr + y/2 * cmsg_step + x/2;
                T* data_cost = data_cost_ + y * cmsg_step + x;

                for(int d = 0; d < nr_plane; d++)
                {
                    float val = 0.0f;
                    for(int yi = y0; yi < yt; yi++)
                    {
                        for(int xi = x0; xi < xt; xi++)
                        {
                            int sel_disp = selected_disparity[d * cdisp_step2];
                            int xr = xi - sel_disp;

                            if (xr < 0 || sel_disp < cth)
                                val += cdata_weight * cmax_data_term;
                            else
                            {
                                const uchar* left_x = cleft + yi * cimg_step + xi * channels;
                                const uchar* right_x = cright + yi * cimg_step + xr * channels;

                                val += DataCostPerPixel<channels>::compute(left_x, right_x);
                            }
                        }
                    }
                    data_cost[cdisp_step1 * d] = saturate_cast<T>(val);
                }
            }
        }

        template <typename T, int winsz, int channels>
        __global__ void compute_data_cost_reduce(const T* selected_disp_pyr, T* data_cost_, int level, int rows, int cols, int h, int nr_plane)
        {
            int x_out = blockIdx.x;
            int y_out = blockIdx.y % h;
            int d = (blockIdx.y / h) * blockDim.z + threadIdx.z;

            int tid = threadIdx.x;

            const T* selected_disparity = selected_disp_pyr + y_out/2 * cmsg_step + x_out/2;
            T* data_cost = data_cost_ + y_out * cmsg_step + x_out;

            if (d < nr_plane)
            {
                int sel_disp = selected_disparity[d * cdisp_step2];

                int x0 = x_out << level;
                int y0 = y_out << level;

                int len = ::min(y0 + winsz, rows) - y0;

                float val = 0.0f;
                if (x0 + tid < cols)
                {
                    if (x0 + tid - sel_disp < 0 || sel_disp < cth)
                        val = cdata_weight * cmax_data_term * len;
                    else
                    {
                        const uchar* lle =  cleft + y0 * cimg_step + channels * (x0 + tid    );
                        const uchar* lri = cright + y0 * cimg_step + channels * (x0 + tid - sel_disp);

                        for(int y = 0; y < len; ++y)
                        {
                            val += DataCostPerPixel<channels>::compute(lle, lri);

                            lle += cimg_step;
                            lri += cimg_step;
                        }
                    }
                }

                extern __shared__ float smem[];

                reduce<winsz>(smem + winsz * threadIdx.z, val, tid, plus<float>());

                if (tid == 0)
                    data_cost[cdisp_step1 * d] = saturate_cast<T>(val);
            }
        }

        template <typename T>
        void compute_data_cost_caller_(const T* disp_selected_pyr, T* data_cost, int /*rows*/, int /*cols*/,
                                      int h, int w, int level, int nr_plane, int channels, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            switch(channels)
            {
            case 1: compute_data_cost<T, 1><<<grid, threads, 0, stream>>>(disp_selected_pyr, data_cost, h, w, level, nr_plane); break;
            case 3: compute_data_cost<T, 3><<<grid, threads, 0, stream>>>(disp_selected_pyr, data_cost, h, w, level, nr_plane); break;
            case 4: compute_data_cost<T, 4><<<grid, threads, 0, stream>>>(disp_selected_pyr, data_cost, h, w, level, nr_plane); break;
            default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__, "compute_data_cost_caller_");
            }
        }

        template <typename T, int winsz>
        void compute_data_cost_reduce_caller_(const T* disp_selected_pyr, T* data_cost, int rows, int cols,
                                      int h, int w, int level, int nr_plane, int channels, hipStream_t stream)
        {
            const int threadsNum = 256;
            const size_t smem_size = threadsNum * sizeof(float);

            dim3 threads(winsz, 1, threadsNum / winsz);
            dim3 grid(w, h, 1);
            grid.y *= divUp(nr_plane, threads.z);

            switch (channels)
            {
            case 1: compute_data_cost_reduce<T, winsz, 1><<<grid, threads, smem_size, stream>>>(disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane); break;
            case 3: compute_data_cost_reduce<T, winsz, 3><<<grid, threads, smem_size, stream>>>(disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane); break;
            case 4: compute_data_cost_reduce<T, winsz, 4><<<grid, threads, smem_size, stream>>>(disp_selected_pyr, data_cost, level, rows, cols, h, nr_plane); break;
            default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__, "compute_data_cost_reduce_caller_");
            }
        }

        template<class T>
        void compute_data_cost(const T* disp_selected_pyr, T* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, hipStream_t stream)
        {
            typedef void (*ComputeDataCostCaller)(const T* disp_selected_pyr, T* data_cost, int rows, int cols,
                int h, int w, int level, int nr_plane, int channels, hipStream_t stream);

            static const ComputeDataCostCaller callers[] =
            {
                compute_data_cost_caller_<T>, compute_data_cost_caller_<T>, compute_data_cost_reduce_caller_<T, 4>,
                compute_data_cost_reduce_caller_<T, 8>, compute_data_cost_reduce_caller_<T, 16>, compute_data_cost_reduce_caller_<T, 32>,
                compute_data_cost_reduce_caller_<T, 64>, compute_data_cost_reduce_caller_<T, 128>, compute_data_cost_reduce_caller_<T, 256>
            };

            size_t disp_step1 = msg_step * h;
            size_t disp_step2 = msg_step * h2;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step1), &disp_step1, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step2), &disp_step2, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmsg_step),  &msg_step,  sizeof(size_t)) );

            callers[level](disp_selected_pyr, data_cost, rows, cols, h, w, level, nr_plane, channels, stream);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void compute_data_cost(const short* disp_selected_pyr, short* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, hipStream_t stream);

        template void compute_data_cost(const float* disp_selected_pyr, float* data_cost, size_t msg_step,
                               int rows, int cols, int h, int w, int h2, int level, int nr_plane, int channels, hipStream_t stream);


        ///////////////////////////////////////////////////////////////
        //////////////////////// init message /////////////////////////
        ///////////////////////////////////////////////////////////////


         template <typename T>
        __device__ void get_first_k_element_increase(T* u_new, T* d_new, T* l_new, T* r_new,
                                                     const T* u_cur, const T* d_cur, const T* l_cur, const T* r_cur,
                                                     T* data_cost_selected, T* disparity_selected_new, T* data_cost_new,
                                                     const T* data_cost_cur, const T* disparity_selected_cur,
                                                     int nr_plane, int nr_plane2)
        {
            for(int i = 0; i < nr_plane; i++)
            {
                T minimum = numeric_limits<T>::max();
                int id = 0;
                for(int j = 0; j < nr_plane2; j++)
                {
                    T cur = data_cost_new[j * cdisp_step1];
                    if(cur < minimum)
                    {
                        minimum = cur;
                        id = j;
                    }
                }

                data_cost_selected[i * cdisp_step1] = data_cost_cur[id * cdisp_step1];
                disparity_selected_new[i * cdisp_step1] = disparity_selected_cur[id * cdisp_step2];

                u_new[i * cdisp_step1] = u_cur[id * cdisp_step2];
                d_new[i * cdisp_step1] = d_cur[id * cdisp_step2];
                l_new[i * cdisp_step1] = l_cur[id * cdisp_step2];
                r_new[i * cdisp_step1] = r_cur[id * cdisp_step2];

                data_cost_new[id * cdisp_step1] = numeric_limits<T>::max();
            }
        }

        template <typename T>
        __global__ void init_message(T* u_new_, T* d_new_, T* l_new_, T* r_new_,
                                     const T* u_cur_, const T* d_cur_, const T* l_cur_, const T* r_cur_,
                                     T* selected_disp_pyr_new, const T* selected_disp_pyr_cur,
                                     T* data_cost_selected_, const T* data_cost_,
                                     int h, int w, int nr_plane, int h2, int w2, int nr_plane2)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y < h && x < w)
            {
                const T* u_cur = u_cur_ + ::min(h2-1, y/2 + 1) * cmsg_step + x/2;
                const T* d_cur = d_cur_ + ::max(0, y/2 - 1)    * cmsg_step + x/2;
                const T* l_cur = l_cur_ + (y/2)                * cmsg_step + ::min(w2-1, x/2 + 1);
                const T* r_cur = r_cur_ + (y/2)                * cmsg_step + ::max(0, x/2 - 1);

                T* data_cost_new = (T*)ctemp + y * cmsg_step + x;

                const T* disparity_selected_cur = selected_disp_pyr_cur + y/2 * cmsg_step + x/2;
                const T* data_cost = data_cost_ + y * cmsg_step + x;

                for(int d = 0; d < nr_plane2; d++)
                {
                    int idx2 = d * cdisp_step2;

                    T val  = data_cost[d * cdisp_step1] + u_cur[idx2] + d_cur[idx2] + l_cur[idx2] + r_cur[idx2];
                    data_cost_new[d * cdisp_step1] = val;
                }

                T* data_cost_selected = data_cost_selected_ + y * cmsg_step + x;
                T* disparity_selected_new = selected_disp_pyr_new + y * cmsg_step + x;

                T* u_new = u_new_ + y * cmsg_step + x;
                T* d_new = d_new_ + y * cmsg_step + x;
                T* l_new = l_new_ + y * cmsg_step + x;
                T* r_new = r_new_ + y * cmsg_step + x;

                u_cur = u_cur_ + y/2 * cmsg_step + x/2;
                d_cur = d_cur_ + y/2 * cmsg_step + x/2;
                l_cur = l_cur_ + y/2 * cmsg_step + x/2;
                r_cur = r_cur_ + y/2 * cmsg_step + x/2;

                get_first_k_element_increase(u_new, d_new, l_new, r_new, u_cur, d_cur, l_cur, r_cur,
                                             data_cost_selected, disparity_selected_new, data_cost_new,
                                             data_cost, disparity_selected_cur, nr_plane, nr_plane2);
            }
        }


        template<class T>
        void init_message(T* u_new, T* d_new, T* l_new, T* r_new,
                          const T* u_cur, const T* d_cur, const T* l_cur, const T* r_cur,
                          T* selected_disp_pyr_new, const T* selected_disp_pyr_cur,
                          T* data_cost_selected, const T* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream)
        {

            size_t disp_step1 = msg_step * h;
            size_t disp_step2 = msg_step * h2;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step1), &disp_step1, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step2), &disp_step2, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmsg_step),   &msg_step, sizeof(size_t)) );

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x);
            grid.y = divUp(h, threads.y);

            init_message<<<grid, threads, 0, stream>>>(u_new, d_new, l_new, r_new,
                                                       u_cur, d_cur, l_cur, r_cur,
                                                       selected_disp_pyr_new, selected_disp_pyr_cur,
                                                       data_cost_selected, data_cost,
                                                       h, w, nr_plane, h2, w2, nr_plane2);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }


        template void init_message(short* u_new, short* d_new, short* l_new, short* r_new,
                          const short* u_cur, const short* d_cur, const short* l_cur, const short* r_cur,
                          short* selected_disp_pyr_new, const short* selected_disp_pyr_cur,
                          short* data_cost_selected, const short* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream);

        template void init_message(float* u_new, float* d_new, float* l_new, float* r_new,
                          const float* u_cur, const float* d_cur, const float* l_cur, const float* r_cur,
                          float* selected_disp_pyr_new, const float* selected_disp_pyr_cur,
                          float* data_cost_selected, const float* data_cost, size_t msg_step,
                          int h, int w, int nr_plane, int h2, int w2, int nr_plane2, hipStream_t stream);

        ///////////////////////////////////////////////////////////////
        ////////////////////  calc all iterations /////////////////////
        ///////////////////////////////////////////////////////////////

        template <typename T>
        __device__ void message_per_pixel(const T* data, T* msg_dst, const T* msg1, const T* msg2, const T* msg3,
                                          const T* dst_disp, const T* src_disp, int nr_plane, volatile T* temp)
        {
            T minimum = numeric_limits<T>::max();

            for(int d = 0; d < nr_plane; d++)
            {
                int idx = d * cdisp_step1;
                T val  = data[idx] + msg1[idx] + msg2[idx] + msg3[idx];

                if(val < minimum)
                    minimum = val;

                msg_dst[idx] = val;
            }

            float sum = 0;
            for(int d = 0; d < nr_plane; d++)
            {
                float cost_min = minimum + cmax_disc_term;
                T src_disp_reg = src_disp[d * cdisp_step1];

                for(int d2 = 0; d2 < nr_plane; d2++)
                    cost_min = fmin(cost_min, msg_dst[d2 * cdisp_step1] + cdisc_single_jump * ::abs(dst_disp[d2 * cdisp_step1] - src_disp_reg));

                temp[d * cdisp_step1] = saturate_cast<T>(cost_min);
                sum += cost_min;
            }
            sum /= nr_plane;

            for(int d = 0; d < nr_plane; d++)
                msg_dst[d * cdisp_step1] = saturate_cast<T>(temp[d * cdisp_step1] - sum);
        }

        template <typename T>
        __global__ void compute_message(T* u_, T* d_, T* l_, T* r_, const T* data_cost_selected, const T* selected_disp_pyr_cur, int h, int w, int nr_plane, int i)
        {
            int y = blockIdx.y * blockDim.y + threadIdx.y;
            int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + i) & 1);

            if (y > 0 && y < h - 1 && x > 0 && x < w - 1)
            {
                const T* data = data_cost_selected + y * cmsg_step + x;

                T* u = u_ + y * cmsg_step + x;
                T* d = d_ + y * cmsg_step + x;
                T* l = l_ + y * cmsg_step + x;
                T* r = r_ + y * cmsg_step + x;

                const T* disp = selected_disp_pyr_cur + y * cmsg_step + x;

                T* temp = (T*)ctemp + y * cmsg_step + x;

                message_per_pixel(data, u, r - 1, u + cmsg_step, l + 1, disp, disp - cmsg_step, nr_plane, temp);
                message_per_pixel(data, d, d - cmsg_step, r - 1, l + 1, disp, disp + cmsg_step, nr_plane, temp);
                message_per_pixel(data, l, u + cmsg_step, d - cmsg_step, l + 1, disp, disp - 1, nr_plane, temp);
                message_per_pixel(data, r, u + cmsg_step, d - cmsg_step, r - 1, disp, disp + 1, nr_plane, temp);
            }
        }


        template<class T>
        void calc_all_iterations(T* u, T* d, T* l, T* r, const T* data_cost_selected,
            const T* selected_disp_pyr_cur, size_t msg_step, int h, int w, int nr_plane, int iters, hipStream_t stream)
        {
            size_t disp_step = msg_step * h;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step1), &disp_step, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmsg_step),  &msg_step,  sizeof(size_t)) );

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(w, threads.x << 1);
            grid.y = divUp(h, threads.y);

            for(int t = 0; t < iters; ++t)
            {
                compute_message<<<grid, threads, 0, stream>>>(u, d, l, r, data_cost_selected, selected_disp_pyr_cur, h, w, nr_plane, t & 1);
                cudaSafeCall( hipGetLastError() );
            }
            if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
        };

        template void calc_all_iterations(short* u, short* d, short* l, short* r, const short* data_cost_selected, const short* selected_disp_pyr_cur, size_t msg_step,
            int h, int w, int nr_plane, int iters, hipStream_t stream);

        template void calc_all_iterations(float* u, float* d, float* l, float* r, const float* data_cost_selected, const float* selected_disp_pyr_cur, size_t msg_step,
            int h, int w, int nr_plane, int iters, hipStream_t stream);


        ///////////////////////////////////////////////////////////////
        /////////////////////////// output ////////////////////////////
        ///////////////////////////////////////////////////////////////


        template <typename T>
        __global__ void compute_disp(const T* u_, const T* d_, const T* l_, const T* r_,
                                     const T* data_cost_selected, const T* disp_selected_pyr,
                                     PtrStepSz<short> disp, int nr_plane)
        {
            int x = blockIdx.x * blockDim.x + threadIdx.x;
            int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y > 0 && y < disp.rows - 1 && x > 0 && x < disp.cols - 1)
            {
                const T* data = data_cost_selected + y * cmsg_step + x;
                const T* disp_selected = disp_selected_pyr + y * cmsg_step + x;

                const T* u = u_ + (y+1) * cmsg_step + (x+0);
                const T* d = d_ + (y-1) * cmsg_step + (x+0);
                const T* l = l_ + (y+0) * cmsg_step + (x+1);
                const T* r = r_ + (y+0) * cmsg_step + (x-1);

                int best = 0;
                T best_val = numeric_limits<T>::max();
                for (int i = 0; i < nr_plane; ++i)
                {
                    int idx = i * cdisp_step1;
                    T val = data[idx]+ u[idx] + d[idx] + l[idx] + r[idx];

                    if (val < best_val)
                    {
                        best_val = val;
                        best = saturate_cast<short>(disp_selected[idx]);
                    }
                }
                disp(y, x) = best;
            }
        }

        template<class T>
        void compute_disp(const T* u, const T* d, const T* l, const T* r, const T* data_cost_selected, const T* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream)
        {
            size_t disp_step = disp.rows * msg_step;
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisp_step1), &disp_step, sizeof(size_t)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmsg_step),  &msg_step,  sizeof(size_t)) );

            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);

            grid.x = divUp(disp.cols, threads.x);
            grid.y = divUp(disp.rows, threads.y);

            compute_disp<<<grid, threads, 0, stream>>>(u, d, l, r, data_cost_selected, disp_selected, disp, nr_plane);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void compute_disp(const short* u, const short* d, const short* l, const short* r, const short* data_cost_selected, const short* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream);

        template void compute_disp(const float* u, const float* d, const float* l, const float* r, const float* data_cost_selected, const float* disp_selected, size_t msg_step,
            const PtrStepSz<short>& disp, int nr_plane, hipStream_t stream);
    } // namespace stereocsbp
}}} // namespace cv { namespace gpu { namespace device {

#endif /* CUDA_DISABLER */
