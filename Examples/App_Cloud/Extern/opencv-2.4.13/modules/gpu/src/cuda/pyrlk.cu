#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/utility.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/reduce.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace pyrlk
{
    __constant__ int c_winSize_x;
    __constant__ int c_winSize_y;
    __constant__ int c_halfWin_x;
    __constant__ int c_halfWin_y;
    __constant__ int c_iters;

    texture<float, hipTextureType2D, hipReadModeElementType> tex_If(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<float4, hipTextureType2D, hipReadModeElementType> tex_If4(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<uchar, hipTextureType2D, hipReadModeElementType> tex_Ib(false, hipFilterModePoint, hipAddressModeClamp);

    texture<float, hipTextureType2D, hipReadModeElementType> tex_Jf(false, hipFilterModeLinear, hipAddressModeClamp);
    texture<float4, hipTextureType2D, hipReadModeElementType> tex_Jf4(false, hipFilterModeLinear, hipAddressModeClamp);

    template <int cn> struct Tex_I;
    template <> struct Tex_I<1>
    {
        static __device__ __forceinline__ float read(float x, float y)
        {
            return tex2D(tex_If, x, y);
        }
    };
    template <> struct Tex_I<4>
    {
        static __device__ __forceinline__ float4 read(float x, float y)
        {
            return tex2D(tex_If4, x, y);
        }
    };

    template <int cn> struct Tex_J;
    template <> struct Tex_J<1>
    {
        static __device__ __forceinline__ float read(float x, float y)
        {
            return tex2D(tex_Jf, x, y);
        }
    };
    template <> struct Tex_J<4>
    {
        static __device__ __forceinline__ float4 read(float x, float y)
        {
            return tex2D(tex_Jf4, x, y);
        }
    };

    __device__ __forceinline__ void accum(float& dst, float val)
    {
        dst += val;
    }
    __device__ __forceinline__ void accum(float& dst, const float4& val)
    {
        dst += val.x + val.y + val.z;
    }

    __device__ __forceinline__ float abs_(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ float4 abs_(const float4& a)
    {
        return abs(a);
    }

    template <int cn, int PATCH_X, int PATCH_Y, bool calcErr>
    __global__ void sparseKernel(const float2* prevPts, float2* nextPts, uchar* status, float* err, const int level, const int rows, const int cols)
    {
    #if __CUDA_ARCH__ <= 110
        const int BLOCK_SIZE = 128;
    #else
        const int BLOCK_SIZE = 256;
    #endif

        __shared__ float smem1[BLOCK_SIZE];
        __shared__ float smem2[BLOCK_SIZE];
        __shared__ float smem3[BLOCK_SIZE];

        const unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        float2 prevPt = prevPts[blockIdx.x];
        prevPt.x *= (1.0f / (1 << level));
        prevPt.y *= (1.0f / (1 << level));

        if (prevPt.x < 0 || prevPt.x >= cols || prevPt.y < 0 || prevPt.y >= rows)
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        prevPt.x -= c_halfWin_x;
        prevPt.y -= c_halfWin_y;

        // extract the patch from the first image, compute covariation matrix of derivatives

        float A11 = 0;
        float A12 = 0;
        float A22 = 0;

        typedef typename TypeVec<float, cn>::vec_type work_type;

        work_type I_patch   [PATCH_Y][PATCH_X];
        work_type dIdx_patch[PATCH_Y][PATCH_X];
        work_type dIdy_patch[PATCH_Y][PATCH_X];

        for (int yBase = threadIdx.y, i = 0; yBase < c_winSize_y; yBase += blockDim.y, ++i)
        {
            for (int xBase = threadIdx.x, j = 0; xBase < c_winSize_x; xBase += blockDim.x, ++j)
            {
                float x = prevPt.x + xBase + 0.5f;
                float y = prevPt.y + yBase + 0.5f;

                I_patch[i][j] = Tex_I<cn>::read(x, y);

                // Sharr Deriv

                work_type dIdx = 3.0f * Tex_I<cn>::read(x+1, y-1) + 10.0f * Tex_I<cn>::read(x+1, y) + 3.0f * Tex_I<cn>::read(x+1, y+1) -
                                 (3.0f * Tex_I<cn>::read(x-1, y-1) + 10.0f * Tex_I<cn>::read(x-1, y) + 3.0f * Tex_I<cn>::read(x-1, y+1));

                work_type dIdy = 3.0f * Tex_I<cn>::read(x-1, y+1) + 10.0f * Tex_I<cn>::read(x, y+1) + 3.0f * Tex_I<cn>::read(x+1, y+1) -
                                (3.0f * Tex_I<cn>::read(x-1, y-1) + 10.0f * Tex_I<cn>::read(x, y-1) + 3.0f * Tex_I<cn>::read(x+1, y-1));

                dIdx_patch[i][j] = dIdx;
                dIdy_patch[i][j] = dIdy;

                accum(A11, dIdx * dIdx);
                accum(A12, dIdx * dIdy);
                accum(A22, dIdy * dIdy);
            }
        }

        reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2, smem3), thrust::tie(A11, A12, A22), tid, thrust::make_tuple(plus<float>(), plus<float>(), plus<float>()));

    #if __CUDA_ARCH__ >= 300
        if (tid == 0)
        {
            smem1[0] = A11;
            smem2[0] = A12;
            smem3[0] = A22;
        }
    #endif

        __syncthreads();

        A11 = smem1[0];
        A12 = smem2[0];
        A22 = smem3[0];

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (tid == 0 && level == 0)
                status[blockIdx.x] = 0;

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt = nextPts[blockIdx.x];
        nextPt.x *= 2.f;
        nextPt.y *= 2.f;

        nextPt.x -= c_halfWin_x;
        nextPt.y -= c_halfWin_y;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < -c_halfWin_x || nextPt.x >= cols || nextPt.y < -c_halfWin_y || nextPt.y >= rows)
            {
                if (tid == 0 && level == 0)
                    status[blockIdx.x] = 0;

                return;
            }

            float b1 = 0;
            float b2 = 0;

            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J<cn>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = (J_val - I_val) * 32.0f;

                    accum(b1, diff * dIdx_patch[i][j]);
                    accum(b2, diff * dIdy_patch[i][j]);
                }
            }

            reduce<BLOCK_SIZE>(smem_tuple(smem1, smem2), thrust::tie(b1, b2), tid, thrust::make_tuple(plus<float>(), plus<float>()));

        #if __CUDA_ARCH__ >= 300
            if (tid == 0)
            {
                smem1[0] = b1;
                smem2[0] = b2;
            }
        #endif

            __syncthreads();

            b1 = smem1[0];
            b2 = smem2[0];

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        float errval = 0;
        if (calcErr)
        {
            for (int y = threadIdx.y, i = 0; y < c_winSize_y; y += blockDim.y, ++i)
            {
                for (int x = threadIdx.x, j = 0; x < c_winSize_x; x += blockDim.x, ++j)
                {
                    work_type I_val = I_patch[i][j];
                    work_type J_val = Tex_J<cn>::read(nextPt.x + x + 0.5f, nextPt.y + y + 0.5f);

                    work_type diff = J_val - I_val;

                    accum(errval, abs_(diff));
                }
            }

            reduce<BLOCK_SIZE>(smem1, errval, tid, plus<float>());
        }

        if (tid == 0)
        {
            nextPt.x += c_halfWin_x;
            nextPt.y += c_halfWin_y;

            nextPts[blockIdx.x] = nextPt;

            if (calcErr)
                err[blockIdx.x] = static_cast<float>(errval) / (cn * c_winSize_x * c_winSize_y);
        }
    }

    template <int cn, int PATCH_X, int PATCH_Y>
    void sparse_caller(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                       int level, dim3 block, hipStream_t stream)
    {
        dim3 grid(ptcount);

        if (level == 0 && err)
            sparseKernel<cn, PATCH_X, PATCH_Y, true><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);
        else
            sparseKernel<cn, PATCH_X, PATCH_Y, false><<<grid, block>>>(prevPts, nextPts, status, err, level, rows, cols);

        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <bool calcErr>
    __global__ void denseKernel(PtrStepf u, PtrStepf v, const PtrStepf prevU, const PtrStepf prevV, PtrStepf err, const int rows, const int cols)
    {
        extern __shared__ int smem[];

        const int patchWidth  = blockDim.x + 2 * c_halfWin_x;
        const int patchHeight = blockDim.y + 2 * c_halfWin_y;

        int* I_patch = smem;
        int* dIdx_patch = I_patch + patchWidth * patchHeight;
        int* dIdy_patch = dIdx_patch + patchWidth * patchHeight;

        const int xBase = blockIdx.x * blockDim.x;
        const int yBase = blockIdx.y * blockDim.y;

        for (int i = threadIdx.y; i < patchHeight; i += blockDim.y)
        {
            for (int j = threadIdx.x; j < patchWidth; j += blockDim.x)
            {
                float x = xBase - c_halfWin_x + j + 0.5f;
                float y = yBase - c_halfWin_y + i + 0.5f;

                I_patch[i * patchWidth + j] = tex2D(tex_Ib, x, y);

                // Sharr Deriv

                dIdx_patch[i * patchWidth + j] = 3 * tex2D(tex_Ib, x+1, y-1) + 10 * tex2D(tex_Ib, x+1, y) + 3 * tex2D(tex_Ib, x+1, y+1) -
                                                (3 * tex2D(tex_Ib, x-1, y-1) + 10 * tex2D(tex_Ib, x-1, y) + 3 * tex2D(tex_Ib, x-1, y+1));

                dIdy_patch[i * patchWidth + j] = 3 * tex2D(tex_Ib, x-1, y+1) + 10 * tex2D(tex_Ib, x, y+1) + 3 * tex2D(tex_Ib, x+1, y+1) -
                                                (3 * tex2D(tex_Ib, x-1, y-1) + 10 * tex2D(tex_Ib, x, y-1) + 3 * tex2D(tex_Ib, x+1, y-1));
            }
        }

        __syncthreads();

        const int x = xBase + threadIdx.x;
        const int y = yBase + threadIdx.y;

        if (x >= cols || y >= rows)
            return;

        int A11i = 0;
        int A12i = 0;
        int A22i = 0;

        for (int i = 0; i < c_winSize_y; ++i)
        {
            for (int j = 0; j < c_winSize_x; ++j)
            {
                int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                A11i += dIdx * dIdx;
                A12i += dIdx * dIdy;
                A22i += dIdy * dIdy;
            }
        }

        float A11 = A11i;
        float A12 = A12i;
        float A22 = A22i;

        float D = A11 * A22 - A12 * A12;

        if (D < numeric_limits<float>::epsilon())
        {
            if (calcErr)
                err(y, x) = numeric_limits<float>::max();

            return;
        }

        D = 1.f / D;

        A11 *= D;
        A12 *= D;
        A22 *= D;

        float2 nextPt;
        nextPt.x = x + prevU(y/2, x/2) * 2.0f;
        nextPt.y = y + prevV(y/2, x/2) * 2.0f;

        for (int k = 0; k < c_iters; ++k)
        {
            if (nextPt.x < 0 || nextPt.x >= cols || nextPt.y < 0 || nextPt.y >= rows)
            {
                if (calcErr)
                    err(y, x) = numeric_limits<float>::max();

                return;
            }

            int b1 = 0;
            int b2 = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);

                    int diff = (J - I) * 32;

                    int dIdx = dIdx_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];
                    int dIdy = dIdy_patch[(threadIdx.y + i) * patchWidth + (threadIdx.x + j)];

                    b1 += diff * dIdx;
                    b2 += diff * dIdy;
                }
            }

            float2 delta;
            delta.x = A12 * b2 - A22 * b1;
            delta.y = A12 * b1 - A11 * b2;

            nextPt.x += delta.x;
            nextPt.y += delta.y;

            if (::fabs(delta.x) < 0.01f && ::fabs(delta.y) < 0.01f)
                break;
        }

        u(y, x) = nextPt.x - x;
        v(y, x) = nextPt.y - y;

        if (calcErr)
        {
            int errval = 0;

            for (int i = 0; i < c_winSize_y; ++i)
            {
                for (int j = 0; j < c_winSize_x; ++j)
                {
                    int I = I_patch[(threadIdx.y + i) * patchWidth + threadIdx.x + j];
                    int J = tex2D(tex_Jf, nextPt.x - c_halfWin_x + j + 0.5f, nextPt.y - c_halfWin_y + i + 0.5f);

                    errval += ::abs(J - I);
                }
            }

            err(y, x) = static_cast<float>(errval) / (c_winSize_x * c_winSize_y);
        }
    }

    void loadConstants(int2 winSize, int iters)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_winSize_x), &winSize.x, sizeof(int)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_winSize_y), &winSize.y, sizeof(int)) );

        int2 halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_halfWin_x), &halfWin.x, sizeof(int)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_halfWin_y), &halfWin.y, sizeof(int)) );

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_iters), &iters, sizeof(int)) );
    }

    void sparse1(PtrStepSzf I, PtrStepSzf J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5] =
        {
            {sparse_caller<1, 1, 1>, sparse_caller<1, 2, 1>, sparse_caller<1, 3, 1>, sparse_caller<1, 4, 1>, sparse_caller<1, 5, 1>},
            {sparse_caller<1, 1, 2>, sparse_caller<1, 2, 2>, sparse_caller<1, 3, 2>, sparse_caller<1, 4, 2>, sparse_caller<1, 5, 2>},
            {sparse_caller<1, 1, 3>, sparse_caller<1, 2, 3>, sparse_caller<1, 3, 3>, sparse_caller<1, 4, 3>, sparse_caller<1, 5, 3>},
            {sparse_caller<1, 1, 4>, sparse_caller<1, 2, 4>, sparse_caller<1, 3, 4>, sparse_caller<1, 4, 4>, sparse_caller<1, 5, 4>},
            {sparse_caller<1, 1, 5>, sparse_caller<1, 2, 5>, sparse_caller<1, 3, 5>, sparse_caller<1, 4, 5>, sparse_caller<1, 5, 5>}
        };

        bindTexture(&tex_If, I);
        bindTexture(&tex_Jf, J);

        funcs[patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

    void sparse4(PtrStepSz<float4> I, PtrStepSz<float4> J, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                 int level, dim3 block, dim3 patch, hipStream_t stream)
    {
        typedef void (*func_t)(int rows, int cols, const float2* prevPts, float2* nextPts, uchar* status, float* err, int ptcount,
                               int level, dim3 block, hipStream_t stream);

        static const func_t funcs[5][5] =
        {
            {sparse_caller<4, 1, 1>, sparse_caller<4, 2, 1>, sparse_caller<4, 3, 1>, sparse_caller<4, 4, 1>, sparse_caller<4, 5, 1>},
            {sparse_caller<4, 1, 2>, sparse_caller<4, 2, 2>, sparse_caller<4, 3, 2>, sparse_caller<4, 4, 2>, sparse_caller<4, 5, 2>},
            {sparse_caller<4, 1, 3>, sparse_caller<4, 2, 3>, sparse_caller<4, 3, 3>, sparse_caller<4, 4, 3>, sparse_caller<4, 5, 3>},
            {sparse_caller<4, 1, 4>, sparse_caller<4, 2, 4>, sparse_caller<4, 3, 4>, sparse_caller<4, 4, 4>, sparse_caller<4, 5, 4>},
            {sparse_caller<4, 1, 5>, sparse_caller<4, 2, 5>, sparse_caller<4, 3, 5>, sparse_caller<4, 4, 5>, sparse_caller<4, 5, 5>}
        };

        bindTexture(&tex_If4, I);
        bindTexture(&tex_Jf4, J);

        funcs[patch.y - 1][patch.x - 1](I.rows, I.cols, prevPts, nextPts, status, err, ptcount,
            level, block, stream);
    }

    void dense(PtrStepSzb I, PtrStepSzf J, PtrStepSzf u, PtrStepSzf v, PtrStepSzf prevU, PtrStepSzf prevV, PtrStepSzf err, int2 winSize, hipStream_t stream)
    {
        dim3 block(16, 16);
        dim3 grid(divUp(I.cols, block.x), divUp(I.rows, block.y));

        bindTexture(&tex_Ib, I);
        bindTexture(&tex_Jf, J);

        int2 halfWin = make_int2((winSize.x - 1) / 2, (winSize.y - 1) / 2);
        const int patchWidth  = block.x + 2 * halfWin.x;
        const int patchHeight = block.y + 2 * halfWin.y;
        size_t smem_size = 3 * patchWidth * patchHeight * sizeof(int);

        if (err.data)
        {
            denseKernel<true><<<grid, block, smem_size, stream>>>(u, v, prevU, prevV, err, I.rows, I.cols);
            cudaSafeCall( hipGetLastError() );
        }
        else
        {
            denseKernel<false><<<grid, block, smem_size, stream>>>(u, v, prevU, prevV, PtrStepf(), I.rows, I.cols);
            cudaSafeCall( hipGetLastError() );
        }

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

#endif /* CUDA_DISABLER */
