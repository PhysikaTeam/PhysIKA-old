#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/utility.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace gfft
    {
        texture<float, hipTextureType2D, hipReadModeElementType> eigTex(0, hipFilterModePoint, hipAddressModeClamp);

        __device__ uint g_counter = 0;

        template <class Mask> __global__ void findCorners(float threshold, const Mask mask, float2* corners, uint max_count, int rows, int cols)
        {
            #if __CUDA_ARCH__ >= 110

            const int j = blockIdx.x * blockDim.x + threadIdx.x;
            const int i = blockIdx.y * blockDim.y + threadIdx.y;

            if (i > 0 && i < rows - 1 && j > 0 && j < cols - 1 && mask(i, j))
            {
                float val = tex2D(eigTex, j, i);

                if (val > threshold)
                {
                    float maxVal = val;

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i - 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i - 1), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i), maxVal);

                    maxVal = ::fmax(tex2D(eigTex, j - 1, i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j    , i + 1), maxVal);
                    maxVal = ::fmax(tex2D(eigTex, j + 1, i + 1), maxVal);

                    if (val == maxVal)
                    {
                        const uint ind = atomicInc(&g_counter, (uint)(-1));

                        if (ind < max_count)
                            corners[ind] = make_float2(j, i);
                    }
                }
            }

            #endif // __CUDA_ARCH__ >= 110
        }

        int findCorners_gpu(PtrStepSzf eig, float threshold, PtrStepSzb mask, float2* corners, int max_count)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(uint)) );

            bindTexture(&eigTex, eig);

            dim3 block(16, 16);
            dim3 grid(divUp(eig.cols, block.x), divUp(eig.rows, block.y));

            if (mask.data)
                findCorners<<<grid, block>>>(threshold, SingleMask(mask), corners, max_count, eig.rows, eig.cols);
            else
                findCorners<<<grid, block>>>(threshold, WithOutMask(), corners, max_count, eig.rows, eig.cols);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            uint count;
            cudaSafeCall( hipMemcpy(&count, counter_ptr, sizeof(uint), hipMemcpyDeviceToHost) );

            return min(count, max_count);
        }

        class EigGreater
        {
        public:
            __device__ __forceinline__ bool operator()(float2 a, float2 b) const
            {
                return tex2D(eigTex, a.x, a.y) > tex2D(eigTex, b.x, b.y);
            }
        };


        void sortCorners_gpu(PtrStepSzf eig, float2* corners, int count)
        {
            bindTexture(&eigTex, eig);

            thrust::device_ptr<float2> ptr(corners);

            thrust::sort(ptr, ptr + count, EigGreater());
        }
    } // namespace optical_flow
}}}


#endif /* CUDA_DISABLER */
