/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/color.hpp"
#include "cvt_color_internal.h"

namespace cv { namespace gpu { namespace device
{
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_x = 8 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_bgr555_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_bgr555_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_bgr565_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_bgr565_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgr555_to_bgra_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgr555_to_rgba_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgr565_to_bgra_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgr565_to_rgba_traits::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(gray_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(gray_to_bgr555_traits::functor_type)
    {
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(gray_to_bgr565_traits::functor_type)
    {
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_yuv4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_yuv4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(yuv4_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(yuv4_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_YCrCb4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_YCrCb4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(YCrCb4_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(YCrCb4_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_xyz4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_xyz4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(xyz4_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(xyz4_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_hsv4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_hsv4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(hsv4_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(hsv4_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(bgra_to_hls4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(rgba_to_hls4_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(hls4_to_bgra_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    OPENCV_GPU_TRANSFORM_FUNCTOR_TRAITS(hls4_to_rgba_traits<uchar>::functor_type)
    {
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

#define OPENCV_GPU_IMPLEMENT_CVTCOLOR(name, traits) \
    void name(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream) \
    { \
        traits::functor_type functor = traits::create_functor(); \
        typedef typename traits::functor_type::argument_type src_t; \
        typedef typename traits::functor_type::result_type   dst_t; \
        cv::gpu::device::transform((PtrStepSz<src_t>)src, (PtrStepSz<dst_t>)dst, functor, WithOutMask(), stream); \
    }

#define OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(name) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name, name ## _traits)

#ifdef OPENCV_TINY_GPU_MODULE
    #define OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(name) \
        OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _8u, name ## _traits<uchar>) \
        OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _32f, name ## _traits<float>)
#else
    #define OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(name) \
        OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _8u, name ## _traits<uchar>) \
        OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _16u, name ## _traits<ushort>) \
        OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _32f, name ## _traits<float>)
#endif

#define OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(name) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _8u, name ## _traits<uchar>) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _32f, name ## _traits<float>)

#define OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(name) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _8u, name ## _traits<uchar>) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _32f, name ## _traits<float>) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _full_8u, name ## _full_traits<uchar>) \
    OPENCV_GPU_IMPLEMENT_CVTCOLOR(name ## _full_32f, name ## _full_traits<float>)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_rgba)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr_to_bgr555)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr_to_bgr565)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(rgb_to_bgr555)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(rgb_to_bgr565)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgra_to_bgr555)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgra_to_bgr565)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(rgba_to_bgr555)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(rgba_to_bgr565)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr555_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr565_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr555_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr565_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr555_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr565_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr555_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr565_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(gray_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(gray_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(gray_to_bgr555)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(gray_to_bgr565)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr555_to_gray)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE(bgr565_to_gray)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_gray)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_gray)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_gray)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_gray)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_yuv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_yuv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_yuv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_yuv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_yuv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_yuv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_yuv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_yuv4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(yuv4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_YCrCb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_YCrCb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_YCrCb4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_YCrCb4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_YCrCb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_YCrCb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_YCrCb4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_YCrCb4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(YCrCb4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_xyz)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_xyz)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgb_to_xyz4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(rgba_to_xyz4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_xyz)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_xyz)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgr_to_xyz4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(bgra_to_xyz4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL(xyz4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgb_to_hsv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgba_to_hsv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgb_to_hsv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgba_to_hsv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgr_to_hsv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgra_to_hsv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgr_to_hsv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgra_to_hsv4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hsv4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgb_to_hls)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgba_to_hls)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgb_to_hls4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(rgba_to_hls4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgr_to_hls)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgra_to_hls)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgr_to_hls4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(bgra_to_hls4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL(hls4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgb_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgba_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgb_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgba_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgr_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgra_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgr_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgra_to_lab4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgb_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgba_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgb_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgba_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgr_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgra_to_lab)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgr_to_lab4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgra_to_lab4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_lrgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_lrgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_lrgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_lrgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_lbgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_lbgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab_to_lbgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lab4_to_lbgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgb_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgba_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgb_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(rgba_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgr_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgra_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgr_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(bgra_to_luv4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgb_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgba_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgb_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lrgba_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgr_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgra_to_luv)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgr_to_luv4)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(lbgra_to_luv4)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_rgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_rgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_bgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_bgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_bgra)

    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_lrgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_lrgb)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_lrgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_lrgba)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_lbgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_lbgr)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv_to_lbgra)
    OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F(luv4_to_lbgra)

    #undef OPENCV_GPU_IMPLEMENT_CVTCOLOR
    #undef OPENCV_GPU_IMPLEMENT_CVTCOLOR_ONE
    #undef OPENCV_GPU_IMPLEMENT_CVTCOLOR_ALL
    #undef OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F
    #undef OPENCV_GPU_IMPLEMENT_CVTCOLOR_8U32F_FULL
}}} // namespace cv { namespace gpu { namespace device

#endif /* CUDA_DISABLER */
