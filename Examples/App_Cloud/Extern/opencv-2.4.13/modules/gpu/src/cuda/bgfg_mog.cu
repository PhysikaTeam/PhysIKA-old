#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/limits.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace mog
    {
        ///////////////////////////////////////////////////////////////
        // Utility

        __device__ __forceinline__ float cvt(uchar val)
        {
            return val;
        }
        __device__ __forceinline__ float3 cvt(const uchar3& val)
        {
            return make_float3(val.x, val.y, val.z);
        }
        __device__ __forceinline__ float4 cvt(const uchar4& val)
        {
            return make_float4(val.x, val.y, val.z, val.w);
        }

        __device__ __forceinline__ float sqr(float val)
        {
            return val * val;
        }
        __device__ __forceinline__ float sqr(const float3& val)
        {
            return val.x * val.x + val.y * val.y + val.z * val.z;
        }
        __device__ __forceinline__ float sqr(const float4& val)
        {
            return val.x * val.x + val.y * val.y + val.z * val.z;
        }

        __device__ __forceinline__ float sum(float val)
        {
            return val;
        }
        __device__ __forceinline__ float sum(const float3& val)
        {
            return val.x + val.y + val.z;
        }
        __device__ __forceinline__ float sum(const float4& val)
        {
            return val.x + val.y + val.z;
        }

        __device__ __forceinline__ float clamp(float var, float learningRate, float diff, float minVar)
        {
             return ::fmaxf(var + learningRate * (diff * diff - var), minVar);
        }
        __device__ __forceinline__ float3 clamp(const float3& var, float learningRate, const float3& diff, float minVar)
        {
             return make_float3(::fmaxf(var.x + learningRate * (diff.x * diff.x - var.x), minVar),
                                ::fmaxf(var.y + learningRate * (diff.y * diff.y - var.y), minVar),
                                ::fmaxf(var.z + learningRate * (diff.z * diff.z - var.z), minVar));
        }
        __device__ __forceinline__ float4 clamp(const float4& var, float learningRate, const float4& diff, float minVar)
        {
             return make_float4(::fmaxf(var.x + learningRate * (diff.x * diff.x - var.x), minVar),
                                ::fmaxf(var.y + learningRate * (diff.y * diff.y - var.y), minVar),
                                ::fmaxf(var.z + learningRate * (diff.z * diff.z - var.z), minVar),
                                0.0f);
        }

        template <class Ptr2D>
        __device__ __forceinline__ void swap(Ptr2D& ptr, int x, int y, int k, int rows)
        {
            typename Ptr2D::elem_type val = ptr(k * rows + y, x);
            ptr(k * rows + y, x) = ptr((k + 1) * rows + y, x);
            ptr((k + 1) * rows + y, x) = val;
        }

        ///////////////////////////////////////////////////////////////
        // MOG without learning

        template <typename SrcT, typename WorkT>
        __global__ void mog_withoutLearning(const PtrStepSz<SrcT> frame, PtrStepb fgmask,
                                            const PtrStepf gmm_weight, const PtrStep<WorkT> gmm_mean, const PtrStep<WorkT> gmm_var,
                                            const int nmixtures, const float varThreshold, const float backgroundRatio)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            WorkT pix = cvt(frame(y, x));

            int kHit = -1;
            int kForeground = -1;

            for (int k = 0; k < nmixtures; ++k)
            {
                if (gmm_weight(k * frame.rows + y, x) < numeric_limits<float>::epsilon())
                    break;

                WorkT mu = gmm_mean(k * frame.rows + y, x);
                WorkT var = gmm_var(k * frame.rows + y, x);

                WorkT diff = pix - mu;

                if (sqr(diff) < varThreshold * sum(var))
                {
                    kHit = k;
                    break;
                }
            }

            if (kHit >= 0)
            {
                float wsum = 0.0f;
                for (int k = 0; k < nmixtures; ++k)
                {
                    wsum += gmm_weight(k * frame.rows + y, x);

                    if (wsum > backgroundRatio)
                    {
                        kForeground = k + 1;
                        break;
                    }
                }
            }

            fgmask(y, x) = (uchar) (-(kHit < 0 || kHit >= kForeground));
        }

        template <typename SrcT, typename WorkT>
        void mog_withoutLearning_caller(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb var,
                                        int nmixtures, float varThreshold, float backgroundRatio, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog_withoutLearning<SrcT), WorkT>, hipFuncCachePreferL1) );

            mog_withoutLearning<SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask,
                                                                         weight, (PtrStepSz<WorkT>) mean, (PtrStepSz<WorkT>) var,
                                                                         nmixtures, varThreshold, backgroundRatio);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////
        // MOG with learning

        template <typename SrcT, typename WorkT>
        __global__ void mog_withLearning(const PtrStepSz<SrcT> frame, PtrStepb fgmask,
                                         PtrStepf gmm_weight, PtrStepf gmm_sortKey, PtrStep<WorkT> gmm_mean, PtrStep<WorkT> gmm_var,
                                         const int nmixtures, const float varThreshold, const float backgroundRatio, const float learningRate, const float minVar)
        {
            const float w0 = 0.05f;
            const float sk0 = w0 / (30.0f * 0.5f * 2.0f);
            const float var0 = 30.0f * 0.5f * 30.0f * 0.5f * 4.0f;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            WorkT pix = cvt(frame(y, x));

            float wsum = 0.0f;
            int kHit = -1;
            int kForeground = -1;

            int k = 0;
            for (; k < nmixtures; ++k)
            {
                float w = gmm_weight(k * frame.rows + y, x);
                wsum += w;

                if (w < numeric_limits<float>::epsilon())
                    break;

                WorkT mu = gmm_mean(k * frame.rows + y, x);
                WorkT var = gmm_var(k * frame.rows + y, x);

                WorkT diff = pix - mu;

                if (sqr(diff) < varThreshold * sum(var))
                {
                    wsum -= w;
                    float dw = learningRate * (1.0f - w);

                    var = clamp(var, learningRate, diff, minVar);

                    float sortKey_prev = w / ::sqrtf(sum(var));
                    gmm_sortKey(k * frame.rows + y, x) = sortKey_prev;

                    float weight_prev = w + dw;
                    gmm_weight(k * frame.rows + y, x) = weight_prev;

                    WorkT mean_prev = mu + learningRate * diff;
                    gmm_mean(k * frame.rows + y, x) = mean_prev;

                    WorkT var_prev = var;
                    gmm_var(k * frame.rows + y, x) = var_prev;

                    int k1 = k - 1;

                    if (k1 >= 0)
                    {
                        float sortKey_next = gmm_sortKey(k1 * frame.rows + y, x);
                        float weight_next = gmm_weight(k1 * frame.rows + y, x);
                        WorkT mean_next = gmm_mean(k1 * frame.rows + y, x);
                        WorkT var_next = gmm_var(k1 * frame.rows + y, x);

                        for (; sortKey_next < sortKey_prev && k1 >= 0; --k1)
                        {
                            gmm_sortKey(k1 * frame.rows + y, x) = sortKey_prev;
                            gmm_sortKey((k1 + 1) * frame.rows + y, x) = sortKey_next;

                            gmm_weight(k1 * frame.rows + y, x) = weight_prev;
                            gmm_weight((k1 + 1) * frame.rows + y, x) = weight_next;

                            gmm_mean(k1 * frame.rows + y, x) = mean_prev;
                            gmm_mean((k1 + 1) * frame.rows + y, x) = mean_next;

                            gmm_var(k1 * frame.rows + y, x) = var_prev;
                            gmm_var((k1 + 1) * frame.rows + y, x) = var_next;

                            sortKey_prev = sortKey_next;
                            sortKey_next = k1 > 0 ? gmm_sortKey((k1 - 1) * frame.rows + y, x) : 0.0f;

                            weight_prev = weight_next;
                            weight_next = k1 > 0 ? gmm_weight((k1 - 1) * frame.rows + y, x) : 0.0f;

                            mean_prev = mean_next;
                            mean_next = k1 > 0 ? gmm_mean((k1 - 1) * frame.rows + y, x) : VecTraits<WorkT>::all(0.0f);

                            var_prev = var_next;
                            var_next = k1 > 0 ? gmm_var((k1 - 1) * frame.rows + y, x) : VecTraits<WorkT>::all(0.0f);
                        }
                    }

                    kHit = k1 + 1;
                    break;
                }
            }

            if (kHit < 0)
            {
                // no appropriate gaussian mixture found at all, remove the weakest mixture and create a new one
                kHit = k = ::min(k, nmixtures - 1);
                wsum += w0 - gmm_weight(k * frame.rows + y, x);

                gmm_weight(k * frame.rows + y, x) = w0;
                gmm_mean(k * frame.rows + y, x) = pix;
                gmm_var(k * frame.rows + y, x) = VecTraits<WorkT>::all(var0);
                gmm_sortKey(k * frame.rows + y, x) = sk0;
            }
            else
            {
                for( ; k < nmixtures; k++)
                    wsum += gmm_weight(k * frame.rows + y, x);
            }

            float wscale = 1.0f / wsum;
            wsum = 0;
            for (k = 0; k < nmixtures; ++k)
            {
                float w = gmm_weight(k * frame.rows + y, x);
                wsum += w *= wscale;

                gmm_weight(k * frame.rows + y, x) = w;
                gmm_sortKey(k * frame.rows + y, x) *= wscale;

                if (wsum > backgroundRatio && kForeground < 0)
                    kForeground = k + 1;
            }

            fgmask(y, x) = (uchar)(-(kHit >= kForeground));
        }

        template <typename SrcT, typename WorkT>
        void mog_withLearning_caller(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzf weight, PtrStepSzf sortKey, PtrStepSzb mean, PtrStepSzb var,
                                     int nmixtures, float varThreshold, float backgroundRatio, float learningRate, float minVar,
                                     hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog_withLearning<SrcT), WorkT>, hipFuncCachePreferL1) );

            mog_withLearning<SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask,
                                                                      weight, sortKey, (PtrStepSz<WorkT>) mean, (PtrStepSz<WorkT>) var,
                                                                      nmixtures, varThreshold, backgroundRatio, learningRate, minVar);

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        ///////////////////////////////////////////////////////////////
        // MOG

        void mog_gpu(PtrStepSzb frame, int cn, PtrStepSzb fgmask, PtrStepSzf weight, PtrStepSzf sortKey, PtrStepSzb mean, PtrStepSzb var, int nmixtures, float varThreshold, float learningRate, float backgroundRatio, float noiseSigma, hipStream_t stream)
        {
            typedef void (*withoutLearning_t)(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb var, int nmixtures, float varThreshold, float backgroundRatio, hipStream_t stream);
            typedef void (*withLearning_t)(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzf weight, PtrStepSzf sortKey, PtrStepSzb mean, PtrStepSzb var, int nmixtures, float varThreshold, float backgroundRatio, float learningRate, float minVar, hipStream_t stream);

            static const withoutLearning_t withoutLearning[] =
            {
                0, mog_withoutLearning_caller<uchar, float>, 0, mog_withoutLearning_caller<uchar3, float3>, mog_withoutLearning_caller<uchar4, float4>
            };
            static const withLearning_t withLearning[] =
            {
                0, mog_withLearning_caller<uchar, float>, 0, mog_withLearning_caller<uchar3, float3>, mog_withLearning_caller<uchar4, float4>
            };

            const float minVar = noiseSigma * noiseSigma;

            if (learningRate > 0.0f)
                withLearning[cn](frame, fgmask, weight, sortKey, mean, var, nmixtures, varThreshold, backgroundRatio, learningRate, minVar, stream);
            else
                withoutLearning[cn](frame, fgmask, weight, mean, var, nmixtures, varThreshold, backgroundRatio, stream);
        }

        template <typename WorkT, typename OutT>
        __global__ void getBackgroundImage(const PtrStepf gmm_weight, const PtrStep<WorkT> gmm_mean, PtrStepSz<OutT> dst, const int nmixtures, const float backgroundRatio)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= dst.cols || y >= dst.rows)
                return;

            WorkT meanVal = VecTraits<WorkT>::all(0.0f);
            float totalWeight = 0.0f;

            for (int mode = 0; mode < nmixtures; ++mode)
            {
                float weight = gmm_weight(mode * dst.rows + y, x);

                WorkT mean = gmm_mean(mode * dst.rows + y, x);
                meanVal = meanVal + weight * mean;

                totalWeight += weight;

                if(totalWeight > backgroundRatio)
                    break;
            }

            meanVal = meanVal * (1.f / totalWeight);

            dst(y, x) = saturate_cast<OutT>(meanVal);
        }

        template <typename WorkT, typename OutT>
        void getBackgroundImage_caller(PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, int nmixtures, float backgroundRatio, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(getBackgroundImage<WorkT), OutT>, hipFuncCachePreferL1) );

            getBackgroundImage<WorkT, OutT><<<grid, block, 0, stream>>>(weight, (PtrStepSz<WorkT>) mean, (PtrStepSz<OutT>) dst, nmixtures, backgroundRatio);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void getBackgroundImage_gpu(int cn, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, int nmixtures, float backgroundRatio, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, int nmixtures, float backgroundRatio, hipStream_t stream);

            static const func_t funcs[] =
            {
                0, getBackgroundImage_caller<float, uchar>, 0, getBackgroundImage_caller<float3, uchar3>, getBackgroundImage_caller<float4, uchar4>
            };

            funcs[cn](weight, mean, dst, nmixtures, backgroundRatio, stream);
        }

        ///////////////////////////////////////////////////////////////
        // MOG2

        __constant__ int           c_nmixtures;
        __constant__ float         c_Tb;
        __constant__ float         c_TB;
        __constant__ float         c_Tg;
        __constant__ float         c_varInit;
        __constant__ float         c_varMin;
        __constant__ float         c_varMax;
        __constant__ float         c_tau;
        __constant__ unsigned char c_shadowVal;

        void loadConstants(int nmixtures, float Tb, float TB, float Tg, float varInit, float varMin, float varMax, float tau, unsigned char shadowVal)
        {
            varMin = ::fminf(varMin, varMax);
            varMax = ::fmaxf(varMin, varMax);

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_nmixtures), &nmixtures, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Tb), &Tb, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_TB), &TB, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_Tg), &Tg, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varInit), &varInit, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varMin), &varMin, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_varMax), &varMax, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &tau, sizeof(float)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_shadowVal), &shadowVal, sizeof(unsigned char)) );
        }

        template <bool detectShadows, typename SrcT, typename WorkT>
        __global__ void mog2(const PtrStepSz<SrcT> frame, PtrStepb fgmask, PtrStepb modesUsed,
                             PtrStepf gmm_weight, PtrStepf gmm_variance, PtrStep<WorkT> gmm_mean,
                             const float alphaT, const float alpha1, const float prune)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= frame.cols || y >= frame.rows)
                return;

            WorkT pix = cvt(frame(y, x));

            //calculate distances to the modes (+ sort)
            //here we need to go in descending order!!!

            bool background = false; // true - the pixel classified as background

            //internal:

            bool fitsPDF = false; //if it remains zero a new GMM mode will be added

            int nmodes = modesUsed(y, x);
            int nNewModes = nmodes; //current number of modes in GMM

            float totalWeight = 0.0f;

            //go through all modes

            for (int mode = 0; mode < nmodes; ++mode)
            {
                //need only weight if fit is found
                float weight = alpha1 * gmm_weight(mode * frame.rows + y, x) + prune;
                int swap_count = 0;
                //fit not found yet
                if (!fitsPDF)
                {
                    //check if it belongs to some of the remaining modes
                    float var = gmm_variance(mode * frame.rows + y, x);

                    WorkT mean = gmm_mean(mode * frame.rows + y, x);

                    //calculate difference and distance
                    WorkT diff = mean - pix;
                    float dist2 = sqr(diff);

                    //background? - Tb - usually larger than Tg
                    if (totalWeight < c_TB && dist2 < c_Tb * var)
                        background = true;

                    //check fit
                    if (dist2 < c_Tg * var)
                    {
                        //belongs to the mode
                        fitsPDF = true;

                        //update distribution

                        //update weight
                        weight += alphaT;
                        float k = alphaT / weight;

                        //update mean
                        gmm_mean(mode * frame.rows + y, x) = mean - k * diff;

                        //update variance
                        float varnew = var + k * (dist2 - var);

                        //limit the variance
                        varnew = ::fmaxf(varnew, c_varMin);
                        varnew = ::fminf(varnew, c_varMax);

                        gmm_variance(mode * frame.rows + y, x) = varnew;

                        //sort
                        //all other weights are at the same place and
                        //only the matched (iModes) is higher -> just find the new place for it

                        for (int i = mode; i > 0; --i)
                        {
                            //check one up
                            if (weight < gmm_weight((i - 1) * frame.rows + y, x))
                                break;

                            swap_count++;
                            //swap one up
                            swap(gmm_weight, x, y, i - 1, frame.rows);
                            swap(gmm_variance, x, y, i - 1, frame.rows);
                            swap(gmm_mean, x, y, i - 1, frame.rows);
                        }

                        //belongs to the mode - bFitsPDF becomes 1
                    }
                } // !fitsPDF

                //check prune
                if (weight < -prune)
                {
                    weight = 0.0;
                    nmodes--;
                }

                gmm_weight((mode - swap_count) * frame.rows + y, x) = weight; //update weight by the calculated value
                totalWeight += weight;
            }

            //renormalize weights

            totalWeight = 1.f / totalWeight;
            for (int mode = 0; mode < nmodes; ++mode)
                gmm_weight(mode * frame.rows + y, x) *= totalWeight;

            nmodes = nNewModes;

            //make new mode if needed and exit

            if (!fitsPDF)
            {
                // replace the weakest or add a new one
                int mode = nmodes == c_nmixtures ? c_nmixtures - 1 : nmodes++;

                if (nmodes == 1)
                    gmm_weight(mode * frame.rows + y, x) = 1.f;
                else
                {
                    gmm_weight(mode * frame.rows + y, x) = alphaT;

                    // renormalize all other weights

                    for (int i = 0; i < nmodes - 1; ++i)
                        gmm_weight(i * frame.rows + y, x) *= alpha1;
                }

                // init

                gmm_mean(mode * frame.rows + y, x) = pix;
                gmm_variance(mode * frame.rows + y, x) = c_varInit;

                //sort
                //find the new place for it

                for (int i = nmodes - 1; i > 0; --i)
                {
                    // check one up
                    if (alphaT < gmm_weight((i - 1) * frame.rows + y, x))
                        break;

                    //swap one up
                    swap(gmm_weight, x, y, i - 1, frame.rows);
                    swap(gmm_variance, x, y, i - 1, frame.rows);
                    swap(gmm_mean, x, y, i - 1, frame.rows);
                }
            }

            //set the number of modes
            modesUsed(y, x) = nmodes;

            bool isShadow = false;
            if (detectShadows && !background)
            {
                float tWeight = 0.0f;

                // check all the components  marked as background:
                for (int mode = 0; mode < nmodes; ++mode)
                {
                    WorkT mean = gmm_mean(mode * frame.rows + y, x);

                    WorkT pix_mean = pix * mean;

                    float numerator = sum(pix_mean);
                    float denominator = sqr(mean);

                    // no division by zero allowed
                    if (denominator == 0)
                        break;

                    // if tau < a < 1 then also check the color distortion
                    if (numerator <= denominator && numerator >= c_tau * denominator)
                    {
                        float a = numerator / denominator;

                        WorkT dD = a * mean - pix;

                        if (sqr(dD) < c_Tb * gmm_variance(mode * frame.rows + y, x) * a * a)
                        {
                            isShadow = true;
                            break;
                        }
                    };

                    tWeight += gmm_weight(mode * frame.rows + y, x);
                    if (tWeight > c_TB)
                        break;
                }
            }

            fgmask(y, x) = background ? 0 : isShadow ? c_shadowVal : 255;
        }

        template <typename SrcT, typename WorkT>
        void mog2_caller(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean,
                         float alphaT, float prune, bool detectShadows, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));

            const float alpha1 = 1.0f - alphaT;

            if (detectShadows)
            {
                cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog2<true), SrcT, WorkT>, hipFuncCachePreferL1) );

                mog2<true, SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, modesUsed,
                                                                    weight, variance, (PtrStepSz<WorkT>) mean,
                                                                    alphaT, alpha1, prune);
            }
            else
            {
                cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(mog2<false), SrcT, WorkT>, hipFuncCachePreferL1) );

                mog2<false, SrcT, WorkT><<<grid, block, 0, stream>>>((PtrStepSz<SrcT>) frame, fgmask, modesUsed,
                                                                    weight, variance, (PtrStepSz<WorkT>) mean,
                                                                    alphaT, alpha1, prune);
            }

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void mog2_gpu(PtrStepSzb frame, int cn, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean,
                      float alphaT, float prune, bool detectShadows, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb frame, PtrStepSzb fgmask, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzf variance, PtrStepSzb mean, float alphaT, float prune, bool detectShadows, hipStream_t stream);

            static const func_t funcs[] =
            {
                0, mog2_caller<uchar, float>, 0, mog2_caller<uchar3, float3>, mog2_caller<uchar4, float4>
            };

            funcs[cn](frame, fgmask, modesUsed, weight, variance, mean, alphaT, prune, detectShadows, stream);
        }

        template <typename WorkT, typename OutT>
        __global__ void getBackgroundImage2(const PtrStepSzb modesUsed, const PtrStepf gmm_weight, const PtrStep<WorkT> gmm_mean, PtrStep<OutT> dst)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= modesUsed.cols || y >= modesUsed.rows)
                return;

            int nmodes = modesUsed(y, x);

            WorkT meanVal = VecTraits<WorkT>::all(0.0f);
            float totalWeight = 0.0f;

            for (int mode = 0; mode < nmodes; ++mode)
            {
                float weight = gmm_weight(mode * modesUsed.rows + y, x);

                WorkT mean = gmm_mean(mode * modesUsed.rows + y, x);
                meanVal = meanVal + weight * mean;

                totalWeight += weight;

                if(totalWeight > c_TB)
                    break;
            }

            meanVal = meanVal * (1.f / totalWeight);

            dst(y, x) = saturate_cast<OutT>(meanVal);
        }

        template <typename WorkT, typename OutT>
        void getBackgroundImage2_caller(PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(modesUsed.cols, block.x), divUp(modesUsed.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(getBackgroundImage2<WorkT), OutT>, hipFuncCachePreferL1) );

            getBackgroundImage2<WorkT, OutT><<<grid, block, 0, stream>>>(modesUsed, weight, (PtrStepSz<WorkT>) mean, (PtrStepSz<OutT>) dst);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void getBackgroundImage2_gpu(int cn, PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream)
        {
            typedef void (*func_t)(PtrStepSzb modesUsed, PtrStepSzf weight, PtrStepSzb mean, PtrStepSzb dst, hipStream_t stream);

            static const func_t funcs[] =
            {
                0, getBackgroundImage2_caller<float, uchar>, 0, getBackgroundImage2_caller<float3, uchar3>, getBackgroundImage2_caller<float4, uchar4>
            };

            funcs[cn](modesUsed, weight, mean, dst, stream);
        }
    }
}}}


#endif /* CUDA_DISABLER */
