#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"
#include "opencv2/gpu/device/border_interpolate.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        template <typename Ptr2D, typename T> __global__ void copyMakeBorder(const Ptr2D src, PtrStepSz<T> dst, int top, int left)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
                dst.ptr(y)[x] = src(y - top, x - left);
        }

        template <template <typename> class B, typename T> struct CopyMakeBorderDispatcher
        {
            static void call(const PtrStepSz<T>& src, const PtrStepSz<T>& dst, int top, int left,
                const typename VecTraits<T>::elem_type* borderValue, hipStream_t stream)
            {
                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<T> brd(src.rows, src.cols, VecTraits<T>::make(borderValue));
                BorderReader< PtrStep<T>, B<T> > brdSrc(src, brd);

                copyMakeBorder<<<grid, block, 0, stream>>>(brdSrc, dst, top, left);
                cudaSafeCall( hipGetLastError() );

                if (stream == 0)
                    cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        template <typename T, int cn> void copyMakeBorder_gpu(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode,
            const T* borderValue, hipStream_t stream)
        {
            typedef typename TypeVec<T, cn>::vec_type vec_type;

            typedef void (*caller_t)(const PtrStepSz<vec_type>& src, const PtrStepSz<vec_type>& dst, int top, int left, const T* borderValue, hipStream_t stream);

            static const caller_t callers[5] =
            {
                CopyMakeBorderDispatcher<BrdReflect101, vec_type>::call,
                CopyMakeBorderDispatcher<BrdReplicate, vec_type>::call,
                CopyMakeBorderDispatcher<BrdConstant, vec_type>::call,
                CopyMakeBorderDispatcher<BrdReflect, vec_type>::call,
    #ifdef OPENCV_TINY_GPU_MODULE
                0,
    #else
                CopyMakeBorderDispatcher<BrdWrap, vec_type>::call
    #endif
            };

            const caller_t caller = callers[borderMode];
            if (!caller)
                cv::gpu::error("Unsupported input parameters for copyMakeBorder", __FILE__, __LINE__, "");

            caller(PtrStepSz<vec_type>(src), PtrStepSz<vec_type>(dst), top, left, borderValue, stream);
        }

        template void copyMakeBorder_gpu<uchar, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const uchar* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<uchar, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const uchar* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<uchar, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const uchar* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<uchar, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const uchar* borderValue, hipStream_t stream);

#ifndef OPENCV_TINY_GPU_MODULE
        //template void copyMakeBorder_gpu<schar, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const schar* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<schar, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const schar* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<schar, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const schar* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<schar, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const schar* borderValue, hipStream_t stream);

        template void copyMakeBorder_gpu<ushort, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const ushort* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<ushort, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const ushort* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<ushort, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const ushort* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<ushort, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const ushort* borderValue, hipStream_t stream);

        template void copyMakeBorder_gpu<short, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const short* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<short, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const short* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<short, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const short* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<short, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const short* borderValue, hipStream_t stream);

        //template void copyMakeBorder_gpu<int, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const int* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<int, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const int* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<int, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const int* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<int, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const int* borderValue, hipStream_t stream);
#endif

        template void copyMakeBorder_gpu<float, 1>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const float* borderValue, hipStream_t stream);
        //template void copyMakeBorder_gpu<float, 2>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const float* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<float, 3>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const float* borderValue, hipStream_t stream);
        template void copyMakeBorder_gpu<float, 4>(const PtrStepSzb& src, const PtrStepSzb& dst, int top, int left, int borderMode, const float* borderValue, hipStream_t stream);
    } // namespace imgproc
}}} // namespace cv { namespace gpu { namespace device

#endif /* CUDA_DISABLER */
