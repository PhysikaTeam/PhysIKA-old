#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/vec_traits.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace video_encoding
    {
        __device__ __forceinline__ void rgbtoy(const uchar b, const uchar g, const uchar r, uchar& y)
        {
            y = static_cast<uchar>(((int)(30 * r) + (int)(59 * g) + (int)(11 * b)) / 100);
        }

        __device__ __forceinline__ void rgbtoyuv(const uchar b, const uchar g, const uchar r, uchar& y, uchar& u, uchar& v)
        {
            rgbtoy(b, g, r, y);
            u = static_cast<uchar>(((int)(-17 * r) - (int)(33 * g) + (int)(50 * b) + 12800) / 100);
            v = static_cast<uchar>(((int)(50 * r) - (int)(42 * g) - (int)(8 * b) + 12800) / 100);
        }

        __global__ void Gray_to_YV12(const PtrStepSzb src, PtrStepb dst)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
            const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

            if (x + 1 >= src.cols || y + 1 >= src.rows)
                return;

            // get pointers to the data
            const size_t planeSize = src.rows * dst.step;
            PtrStepb y_plane(dst.data, dst.step);
            PtrStepb u_plane(y_plane.data + planeSize, dst.step / 2);
            PtrStepb v_plane(u_plane.data + (planeSize / 4), dst.step / 2);

            uchar pix;
            uchar y_val, u_val, v_val;

            pix = src(y, x);
            rgbtoy(pix, pix, pix, y_val);
            y_plane(y, x) = y_val;

            pix = src(y, x + 1);
            rgbtoy(pix, pix, pix, y_val);
            y_plane(y, x + 1) = y_val;

            pix = src(y + 1, x);
            rgbtoy(pix, pix, pix, y_val);
            y_plane(y + 1, x) = y_val;

            pix = src(y + 1, x + 1);
            rgbtoyuv(pix, pix, pix, y_val, u_val, v_val);
            y_plane(y + 1, x + 1) = y_val;
            u_plane(y / 2, x / 2) = u_val;
            v_plane(y / 2, x / 2) = v_val;
        }

        template <typename T>
        __global__ void BGR_to_YV12(const PtrStepSz<T> src, PtrStepb dst)
        {
            const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
            const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

            if (x + 1 >= src.cols || y + 1 >= src.rows)
                return;

            // get pointers to the data
            const size_t planeSize = src.rows * dst.step;
            PtrStepb y_plane(dst.data, dst.step);
            PtrStepb u_plane(y_plane.data + planeSize, dst.step / 2);
            PtrStepb v_plane(u_plane.data + (planeSize / 4), dst.step / 2);

            T pix;
            uchar y_val, u_val, v_val;

            pix = src(y, x);
            rgbtoy(pix.z, pix.y, pix.x, y_val);
            y_plane(y, x) = y_val;

            pix = src(y, x + 1);
            rgbtoy(pix.z, pix.y, pix.x, y_val);
            y_plane(y, x + 1) = y_val;

            pix = src(y + 1, x);
            rgbtoy(pix.z, pix.y, pix.x, y_val);
            y_plane(y + 1, x) = y_val;

            pix = src(y + 1, x + 1);
            rgbtoyuv(pix.z, pix.y, pix.x, y_val, u_val, v_val);
            y_plane(y + 1, x + 1) = y_val;
            u_plane(y / 2, x / 2) = u_val;
            v_plane(y / 2, x / 2) = v_val;
        }

        void Gray_to_YV12_caller(const PtrStepSzb src, PtrStepb dst)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(src.cols, block.x * 2), divUp(src.rows, block.y * 2));

            Gray_to_YV12<<<grid, block>>>(src, dst);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }
        template <int cn>
        void BGR_to_YV12_caller(const PtrStepSzb src, PtrStepb dst)
        {
            typedef typename TypeVec<uchar, cn>::vec_type src_t;

            dim3 block(32, 8);
            dim3 grid(divUp(src.cols, block.x * 2), divUp(src.rows, block.y * 2));

            BGR_to_YV12<<<grid, block>>>(static_cast< PtrStepSz<src_t> >(src), dst);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        void YV12_gpu(const PtrStepSzb src, int cn, PtrStepSzb dst)
        {
            typedef void (*func_t)(const PtrStepSzb src, PtrStepb dst);

            static const func_t funcs[] =
            {
                0, Gray_to_YV12_caller, 0, BGR_to_YV12_caller<3>, BGR_to_YV12_caller<4>
            };

            funcs[cn](src, dst);
        }
    }
}}}

#endif /* CUDA_DISABLER */
