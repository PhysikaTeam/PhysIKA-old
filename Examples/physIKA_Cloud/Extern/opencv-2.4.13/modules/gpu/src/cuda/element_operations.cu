#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/functional.hpp"
#include "opencv2/gpu/device/vec_math.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/limits.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/simd_functions.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace arithm
{
    template <size_t src_size, size_t dst_size> struct ArithmFuncTraits
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 1 };
    };

    template <> struct ArithmFuncTraits<1, 1>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<1, 2>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<1, 4>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <> struct ArithmFuncTraits<2, 1>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<2, 2>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<2, 4>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };

    template <> struct ArithmFuncTraits<4, 1>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<4, 2>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
    template <> struct ArithmFuncTraits<4, 4>
    {
        enum { simple_block_dim_x = 32 };
        enum { simple_block_dim_y = 8 };

        enum { smart_block_dim_x = 32 };
        enum { smart_block_dim_y = 8 };
        enum { smart_shift = 4 };
    };
}

//////////////////////////////////////////////////////////////////////////
// addMat

namespace arithm
{
    struct VAdd4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vadd4(a, b);
        }

        __host__ __device__ __forceinline__ VAdd4() {}
        __host__ __device__ __forceinline__ VAdd4(const VAdd4&) {}
    };

    ////////////////////////////////////

    struct VAdd2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vadd2(a, b);
        }

        __host__ __device__ __forceinline__ VAdd2() {}
        __host__ __device__ __forceinline__ VAdd2(const VAdd2&) {}
    };

    ////////////////////////////////////

    template <typename T, typename D> struct AddMat : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a + b);
        }

        __host__ __device__ __forceinline__ AddMat() {}
        __host__ __device__ __forceinline__ AddMat(const AddMat&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VAdd4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <> struct TransformFunctorTraits< arithm::VAdd2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <typename T, typename D> struct TransformFunctorTraits< arithm::AddMat<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void addMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VAdd4(), WithOutMask(), stream);
    }

    void addMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VAdd2(), WithOutMask(), stream);
    }

    template <typename T, typename D>
    void addMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, AddMat<T, D>(), mask, stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, AddMat<T, D>(), WithOutMask(), stream);
    }

    template void addMat<uchar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addMat<uchar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void addMat<schar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<ushort, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<ushort, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<short, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<short, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<int, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
    template void addMat<float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addMat<float, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// addScalar

namespace arithm
{
    template <typename T, typename S, typename D> struct AddScalar : unary_function<T, D>
    {
        S val;

        explicit AddScalar(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a + val);
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::AddScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void addScalar(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        AddScalar<T, S, D> op(static_cast<S>(val));

        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, mask, stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void addScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void addScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
    template void addScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// subMat

namespace arithm
{
    struct VSub4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub4(a, b);
        }

        __host__ __device__ __forceinline__ VSub4() {}
        __host__ __device__ __forceinline__ VSub4(const VSub4&) {}
    };

    ////////////////////////////////////

    struct VSub2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub2(a, b);
        }

        __host__ __device__ __forceinline__ VSub2() {}
        __host__ __device__ __forceinline__ VSub2(const VSub2&) {}
    };

    ////////////////////////////////////

    template <typename T, typename D> struct SubMat : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a - b);
        }

        __host__ __device__ __forceinline__ SubMat() {}
        __host__ __device__ __forceinline__ SubMat(const SubMat&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VSub4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <> struct TransformFunctorTraits< arithm::VSub2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <typename T, typename D> struct TransformFunctorTraits< arithm::SubMat<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void subMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VSub4(), WithOutMask(), stream);
    }

    void subMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VSub2(), WithOutMask(), stream);
    }

    template <typename T, typename D>
    void subMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, SubMat<T, D>(), mask, stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, SubMat<T, D>(), WithOutMask(), stream);
    }

    template void subMat<uchar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void subMat<uchar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void subMat<schar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<ushort, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<ushort, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<short, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<short, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<int, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
    template void subMat<float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void subMat<float, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// subScalar

namespace arithm
{
    template <typename T, typename S, typename D>
    void subScalar(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        AddScalar<T, S, D> op(-static_cast<S>(val));

        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, mask, stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void subScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void subScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void subScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
    template void subScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void subScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// mulMat

namespace arithm
{
    struct Mul_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
            res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
            res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
            res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);

            return res;
        }

        __host__ __device__ __forceinline__ Mul_8uc4_32f() {}
        __host__ __device__ __forceinline__ Mul_8uc4_32f(const Mul_8uc4_32f&) {}
    };

    struct Mul_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x * b), saturate_cast<short>(a.y * b),
                               saturate_cast<short>(a.z * b), saturate_cast<short>(a.w * b));
        }

        __host__ __device__ __forceinline__ Mul_16sc4_32f() {}
        __host__ __device__ __forceinline__ Mul_16sc4_32f(const Mul_16sc4_32f&) {}
    };

    template <typename T, typename D> struct Mul : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a * b);
        }

        __host__ __device__ __forceinline__ Mul() {}
        __host__ __device__ __forceinline__ Mul(const Mul&) {}
    };

    template <typename T, typename S, typename D> struct MulScale : binary_function<T, T, D>
    {
        S scale;

        explicit MulScale(S scale_) : scale(scale_) {}

        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(scale * a * b);
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits<arithm::Mul_8uc4_32f> : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::Mul<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };

    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::MulScale<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void mulMat_8uc4_32f(PtrStepSz<uint> src1, PtrStepSzf src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, Mul_8uc4_32f(), WithOutMask(), stream);
    }

    void mulMat_16sc4_32f(PtrStepSz<short4> src1, PtrStepSzf src2, PtrStepSz<short4> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, Mul_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename S, typename D>
    void mulMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream)
    {
        if (scale == 1)
        {
            Mul<T, D> op;
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
        else
        {
            MulScale<T, S, D> op(static_cast<S>(scale));
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
    }

    template void mulMat<uchar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void mulMat<uchar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    template void mulMat<schar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<ushort, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<ushort, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<short, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<short, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<int, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<float, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#endif
    template void mulMat<float, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void mulMat<float, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<double, double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<double, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// mulScalar

namespace arithm
{
    template <typename T, typename S, typename D> struct MulScalar : unary_function<T, D>
    {
        S val;

        explicit MulScalar(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a * val);
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::MulScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void mulScalar(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        MulScalar<T, S, D> op(static_cast<S>(val));
        transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void mulScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void mulScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    template void mulScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
    template void mulScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void mulScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// divMat

namespace arithm
{
    struct Div_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            if (b != 0)
            {
                b = 1.0f / b;
                res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
                res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
                res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
                res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);
            }

            return res;
        }
    };

    struct Div_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return b != 0 ? make_short4(saturate_cast<short>(a.x / b), saturate_cast<short>(a.y / b),
                                        saturate_cast<short>(a.z / b), saturate_cast<short>(a.w / b))
                          : make_short4(0,0,0,0);
        }
    };

    template <typename T, typename D> struct Div : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(a / b) : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };
    template <typename T> struct Div<T, float> : binary_function<T, T, float>
    {
        __device__ __forceinline__ float operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<float>(a) / b : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };
    template <typename T> struct Div<T, double> : binary_function<T, T, double>
    {
        __device__ __forceinline__ double operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<double>(a) / b : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };

    template <typename T, typename S, typename D> struct DivScale : binary_function<T, T, D>
    {
        S scale;

        explicit DivScale(S scale_) : scale(scale_) {}

        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(scale * a / b) : 0;
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits<arithm::Div_8uc4_32f> : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::Div<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };

    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::DivScale<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void divMat_8uc4_32f(PtrStepSz<uint> src1, PtrStepSzf src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, Div_8uc4_32f(), WithOutMask(), stream);
    }

    void divMat_16sc4_32f(PtrStepSz<short4> src1, PtrStepSzf src2, PtrStepSz<short4> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, Div_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename S, typename D>
    void divMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream)
    {
        if (scale == 1)
        {
            Div<T, D> op;
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
        else
        {
            DivScale<T, S, D> op(static_cast<S>(scale));
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
    }

    template void divMat<uchar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divMat<uchar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    template void divMat<schar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<ushort, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<ushort, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<short, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<short, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<int, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<float, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#endif
    template void divMat<float, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divMat<float, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<double, double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<double, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// divScalar

namespace arithm
{
    template <typename T, typename S, typename D>
    void divScalar(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        MulScalar<T, S, D> op(static_cast<S>(1.0 / val));
        transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void divScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    template void divScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
    template void divScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// divInv

namespace arithm
{
    template <typename T, typename S, typename D> struct DivInv : unary_function<T, D>
    {
        S val;

        explicit DivInv(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return a != 0 ? saturate_cast<D>(val / a) : 0;
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::DivInv<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void divInv(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        DivInv<T, S, D> op(static_cast<S>(val));
        transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void divInv<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divInv<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    template void divInv<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divInv<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divInv<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divInv<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divInv<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
    template void divInv<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void divInv<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void divInv<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void divInv<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void divInv<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// absDiffMat

namespace arithm
{
    struct VAbsDiff4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff4(a, b);
        }

        __host__ __device__ __forceinline__ VAbsDiff4() {}
        __host__ __device__ __forceinline__ VAbsDiff4(const VAbsDiff4&) {}
    };

    ////////////////////////////////////

    struct VAbsDiff2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff2(a, b);
        }

        __host__ __device__ __forceinline__ VAbsDiff2() {}
        __host__ __device__ __forceinline__ VAbsDiff2(const VAbsDiff2&) {}
    };

    ////////////////////////////////////

    __device__ __forceinline__ int _abs(int a)
    {
        return ::abs(a);
    }
    __device__ __forceinline__ float _abs(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ double _abs(double a)
    {
        return ::fabs(a);
    }

    template <typename T> struct AbsDiffMat : binary_function<T, T, T>
    {
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return saturate_cast<T>(_abs(a - b));
        }

        __host__ __device__ __forceinline__ AbsDiffMat() {}
        __host__ __device__ __forceinline__ AbsDiffMat(const AbsDiffMat&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VAbsDiff4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <> struct TransformFunctorTraits< arithm::VAbsDiff2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <typename T> struct TransformFunctorTraits< arithm::AbsDiffMat<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void absDiffMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VAbsDiff4(), WithOutMask(), stream);
    }

    void absDiffMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VAbsDiff2(), WithOutMask(), stream);
    }

    template <typename T>
    void absDiffMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, AbsDiffMat<T>(), WithOutMask(), stream);
    }

    template void absDiffMat<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void absDiffMat<schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void absDiffMat<float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void absDiffMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// absDiffScalar

namespace arithm
{
    template <typename T, typename S> struct AbsDiffScalar : unary_function<T, T>
    {
        S val;

        explicit AbsDiffScalar(S val_) : val(val_) {}

        __device__ __forceinline__ T operator ()(T a) const
        {
            abs_func<S> f;
            return saturate_cast<T>(f(a - val));
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T, typename S> struct TransformFunctorTraits< arithm::AbsDiffScalar<T, S> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S>
    void absDiffScalar(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        AbsDiffScalar<T, S> op(static_cast<S>(val));

        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, op, WithOutMask(), stream);
    }

    template void absDiffScalar<uchar, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void absDiffScalar<schar, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffScalar<ushort, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffScalar<short, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffScalar<int, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void absDiffScalar<float, float>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void absDiffScalar<double, double>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// absMat

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< abs_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T>
    void absMat(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, abs_func<T>(), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void absMat<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void absMat<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void absMat<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void absMat<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void absMat<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
    template void absMat<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void absMat<double>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// sqrMat

namespace arithm
{
    template <typename T> struct Sqr : unary_function<T, T>
    {
        __device__ __forceinline__ T operator ()(T x) const
        {
            return saturate_cast<T>(x * x);
        }

        __host__ __device__ __forceinline__ Sqr() {}
        __host__ __device__ __forceinline__ Sqr(const Sqr&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< arithm::Sqr<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T>
    void sqrMat(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, Sqr<T>(), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void sqrMat<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrMat<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrMat<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrMat<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrMat<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
    template void sqrMat<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void sqrMat<double>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// sqrtMat

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< sqrt_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T>
    void sqrtMat(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, sqrt_func<T>(), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void sqrtMat<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrtMat<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrtMat<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrtMat<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void sqrtMat<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
    template void sqrtMat<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void sqrtMat<double>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// logMat

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< log_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T>
    void logMat(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, log_func<T>(), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void logMat<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void logMat<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void logMat<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void logMat<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void logMat<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
    template void logMat<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void logMat<double>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// expMat

namespace arithm
{
    template <typename T> struct Exp : unary_function<T, T>
    {
        __device__ __forceinline__ T operator ()(T x) const
        {
            exp_func<T> f;
            return saturate_cast<T>(f(x));
        }

        __host__ __device__ __forceinline__ Exp() {}
        __host__ __device__ __forceinline__ Exp(const Exp&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< arithm::Exp<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T>
    void expMat(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, Exp<T>(), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void expMat<uchar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void expMat<schar>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void expMat<ushort>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void expMat<short>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
    template void expMat<int>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
    template void expMat<float>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void expMat<double>(PtrStepSzb src, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////////////////
// cmpMat

namespace arithm
{
    struct VCmpEq4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmpeq4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpEq4() {}
        __host__ __device__ __forceinline__ VCmpEq4(const VCmpEq4&) {}
    };
    struct VCmpNe4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmpne4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpNe4() {}
        __host__ __device__ __forceinline__ VCmpNe4(const VCmpNe4&) {}
    };
    struct VCmpLt4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmplt4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpLt4() {}
        __host__ __device__ __forceinline__ VCmpLt4(const VCmpLt4&) {}
    };
    struct VCmpLe4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmple4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpLe4() {}
        __host__ __device__ __forceinline__ VCmpLe4(const VCmpLe4&) {}
    };

    ////////////////////////////////////

    template <class Op, typename T>
    struct Cmp : binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T a, T b) const
        {
            Op op;
            return -op(a, b);
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VCmpEq4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpNe4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpLt4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpLe4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <class Op, typename T> struct TransformFunctorTraits< arithm::Cmp<Op, T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(uchar)>
    {
    };
}}}

namespace arithm
{
    void cmpMatEq_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VCmpEq4(), WithOutMask(), stream);
    }
    void cmpMatNe_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VCmpNe4(), WithOutMask(), stream);
    }
    void cmpMatLt_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VCmpLt4(), WithOutMask(), stream);
    }
    void cmpMatLe_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VCmpLe4(), WithOutMask(), stream);
    }

    template <template <typename> class Op, typename T>
    void cmpMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        Cmp<Op<T>, T> op;
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, dst, op, WithOutMask(), stream);
    }

    template <typename T> void cmpMatEq(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatNe(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<not_equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatLt(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<less, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatLe(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<less_equal, T>(src1, src2, dst, stream);
    }

    template void cmpMatEq<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatEq<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpMatEq<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatEq<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpMatNe<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatNe<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpMatNe<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatNe<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpMatLt<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatLt<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpMatLt<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatLt<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpMatLe<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatLe<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpMatLe<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpMatLe<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////////////////
// cmpScalar

namespace arithm
{
#define TYPE_VEC(type, cn) typename TypeVec<type, cn>::vec_type

    template <class Op, typename T, int cn> struct CmpScalar;
    template <class Op, typename T>
    struct CmpScalar<Op, T, 1> : unary_function<T, uchar>
    {
        const T val;

        __host__ explicit CmpScalar(T val_) : val(val_) {}

        __device__ __forceinline__ uchar operator()(T src) const
        {
            Cmp<Op, T> op;
            return op(src, val);
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 2> : unary_function<TYPE_VEC(T, 2), TYPE_VEC(uchar, 2)>
    {
        const TYPE_VEC(T, 2) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 2) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 2) operator()(const TYPE_VEC(T, 2) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 2)>::make(op(src.x, val.x), op(src.y, val.y));
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 3> : unary_function<TYPE_VEC(T, 3), TYPE_VEC(uchar, 3)>
    {
        const TYPE_VEC(T, 3) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 3) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 3) operator()(const TYPE_VEC(T, 3) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 3)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z));
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 4> : unary_function<TYPE_VEC(T, 4), TYPE_VEC(uchar, 4)>
    {
        const TYPE_VEC(T, 4) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 4) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 4) operator()(const TYPE_VEC(T, 4) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 4)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z), op(src.w, val.w));
        }
    };

#undef TYPE_VEC
}

namespace cv { namespace gpu { namespace device
{
    template <class Op, typename T> struct TransformFunctorTraits< arithm::CmpScalar<Op, T, 1> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(uchar)>
    {
    };
}}}

namespace arithm
{
    template <template <typename> class Op, typename T, int cn>
    void cmpScalar(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type src_t;
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        T sval[] = {static_cast<T>(val[0]), static_cast<T>(val[1]), static_cast<T>(val[2]), static_cast<T>(val[3])};
        src_t val1 = VecTraits<src_t>::make(sval);

        CmpScalar<Op<T>, T, cn> op(val1);
        transform((PtrStepSz<src_t>) src, (PtrStepSz<dst_t>) dst, op, WithOutMask(), stream);
    }

    template <typename T> void cmpScalarEq(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<equal_to, T, 1>,
            cmpScalar<equal_to, T, 2>,
            cmpScalar<equal_to, T, 3>,
            cmpScalar<equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarNe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<not_equal_to, T, 1>,
            cmpScalar<not_equal_to, T, 2>,
            cmpScalar<not_equal_to, T, 3>,
            cmpScalar<not_equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarLt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<less, T, 1>,
            cmpScalar<less, T, 2>,
            cmpScalar<less, T, 3>,
            cmpScalar<less, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarLe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<less_equal, T, 1>,
            cmpScalar<less_equal, T, 2>,
            cmpScalar<less_equal, T, 3>,
            cmpScalar<less_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarGt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<greater, T, 1>,
            cmpScalar<greater, T, 2>,
            cmpScalar<greater, T, 3>,
            cmpScalar<greater, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarGe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<greater_equal, T, 1>,
            cmpScalar<greater_equal, T, 2>,
            cmpScalar<greater_equal, T, 3>,
            cmpScalar<greater_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }

    template void cmpScalarEq<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarEq<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarEq<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarEq<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpScalarNe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarNe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarNe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarNe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpScalarLt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarLt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarLt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarLt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpScalarLe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarLe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarLe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarLe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpScalarGt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarGt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarGt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarGt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif

    template void cmpScalarGe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarGe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
    template void cmpScalarGe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void cmpScalarGe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////////////////
// bitMat

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< bit_not<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_and<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_or<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_xor<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T> void bitMatNot(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, bit_not<T>(), SingleMaskChannels(mask, num_channels), stream);
        else
            transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, bit_not<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatAnd(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_and<T>(), SingleMaskChannels(mask, num_channels), stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_and<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatOr(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_or<T>(), SingleMaskChannels(mask, num_channels), stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_or<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatXor(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream)
    {
        if (mask.data)
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_xor<T>(), SingleMaskChannels(mask, num_channels), stream);
        else
            transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_xor<T>(), WithOutMask(), stream);
    }

    template void bitMatNot<uchar>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatNot<ushort>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatNot<uint>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);

    template void bitMatAnd<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatAnd<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatAnd<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);

    template void bitMatOr<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatOr<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatOr<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);

    template void bitMatXor<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatXor<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
    template void bitMatXor<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, int num_channels, hipStream_t stream);
}

//////////////////////////////////////////////////////////////////////////////////////
// bitScalar

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< binder2nd< bit_and<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< bit_or<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< bit_xor<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T> void bitScalarAnd(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::gpu::device::bind2nd(bit_and<T>(), src2), WithOutMask(), stream);
    }

    template <typename T> void bitScalarOr(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::gpu::device::bind2nd(bit_or<T>(), src2), WithOutMask(), stream);
    }

    template <typename T> void bitScalarXor(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::gpu::device::bind2nd(bit_xor<T>(), src2), WithOutMask(), stream);
    }

    template void bitScalarAnd<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void bitScalarAnd<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarAnd<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarAnd<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template void bitScalarOr<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void bitScalarOr<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarOr<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarOr<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template void bitScalarXor<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void bitScalarXor<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarXor<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarXor<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// min

namespace arithm
{
    struct VMin4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin4(a, b);
        }

        __host__ __device__ __forceinline__ VMin4() {}
        __host__ __device__ __forceinline__ VMin4(const VMin4&) {}
    };

    ////////////////////////////////////

    struct VMin2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin2(a, b);
        }

        __host__ __device__ __forceinline__ VMin2() {}
        __host__ __device__ __forceinline__ VMin2(const VMin2&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VMin4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <> struct TransformFunctorTraits< arithm::VMin2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <typename T> struct TransformFunctorTraits< minimum<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< minimum<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void minMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VMin4(), WithOutMask(), stream);
    }

    void minMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VMin2(), WithOutMask(), stream);
    }

    template <typename T> void minMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, minimum<T>(), WithOutMask(), stream);
    }

    template void minMat<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void minMat<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void minMat<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void minMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template <typename T> void minScalar(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::gpu::device::bind2nd(minimum<T>(), src2), WithOutMask(), stream);
    }

#ifdef OPENCV_TINY_GPU_MODULE
    template void minScalar<uchar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<int   >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<float >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#else
    template void minScalar<uchar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<schar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<ushort>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<short >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<int   >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<float >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<double>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// max

namespace arithm
{
    struct VMax4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax4(a, b);
        }

        __host__ __device__ __forceinline__ VMax4() {}
        __host__ __device__ __forceinline__ VMax4(const VMax4&) {}
    };

    ////////////////////////////////////

    struct VMax2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax2(a, b);
        }

        __host__ __device__ __forceinline__ VMax2() {}
        __host__ __device__ __forceinline__ VMax2(const VMax2&) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VMax4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <> struct TransformFunctorTraits< arithm::VMax2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    ////////////////////////////////////

    template <typename T> struct TransformFunctorTraits< maximum<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< maximum<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void maxMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VMax4(), WithOutMask(), stream);
    }

    void maxMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        transform(src1, src2, dst, VMax2(), WithOutMask(), stream);
    }

    template <typename T> void maxMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, maximum<T>(), WithOutMask(), stream);
    }

    template void maxMat<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void maxMat<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void maxMat<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void maxMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
#endif

    template <typename T> void maxScalar(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::gpu::device::bind2nd(maximum<T>(), src2), WithOutMask(), stream);
    }

    template void maxScalar<uchar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void maxScalar<schar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<ushort>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<short >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<int   >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#endif
    template void maxScalar<float >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void maxScalar<double>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// threshold

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< thresh_binary_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_binary_inv_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_trunc_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_to_zero_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_to_zero_inv_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <template <typename> class Op, typename T>
    void threshold_caller(PtrStepSz<T> src, PtrStepSz<T> dst, T thresh, T maxVal, hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        transform(src, dst, op, WithOutMask(), stream);
    }

    template <typename T>
    void threshold(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream)
    {
        typedef void (*caller_t)(PtrStepSz<T> src, PtrStepSz<T> dst, T thresh, T maxVal, hipStream_t stream);

        static const caller_t callers[] =
        {
            threshold_caller<thresh_binary_func, T>,
            threshold_caller<thresh_binary_inv_func, T>,
            threshold_caller<thresh_trunc_func, T>,
            threshold_caller<thresh_to_zero_func, T>,
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((PtrStepSz<T>) src, (PtrStepSz<T>) dst, static_cast<T>(thresh), static_cast<T>(maxVal), stream);
    }

    template void threshold<uchar>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void threshold<schar>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<ushort>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<short>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<int>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
#endif
    template void threshold<float>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void threshold<double>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// pow

namespace arithm
{
    template<typename T, bool Signed = numeric_limits<T>::is_signed> struct PowOp : unary_function<T, T>
    {
        float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            return saturate_cast<T>(__powf((float)e, power));
        }
    };
    template<typename T> struct PowOp<T, true> : unary_function<T, T>
    {
        float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ T operator()(T e) const
        {
            T res = saturate_cast<T>(__powf((float)e, power));

            if ((e < 0) && (1 & static_cast<int>(power)))
                res *= -1;

            return res;
        }
    };
    template<> struct PowOp<float> : unary_function<float, float>
    {
        const float power;

        PowOp(double power_) : power(static_cast<float>(power_)) {}

        __device__ __forceinline__ float operator()(float e) const
        {
            return __powf(::fabs(e), power);
        }
    };
    template<> struct PowOp<double> : unary_function<double, double>
    {
        double power;

        PowOp(double power_) : power(power_) {}

        __device__ __forceinline__ double operator()(double e) const
        {
            return ::pow(::fabs(e), power);
        }
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T> struct TransformFunctorTraits< arithm::PowOp<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template<typename T>
    void pow(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream)
    {
        transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, PowOp<T>(power), WithOutMask(), stream);
    }

#ifndef OPENCV_TINY_GPU_MODULE
    template void pow<uchar>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow<schar>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow<short>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow<ushort>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
    template void pow<int>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
#endif
    template void pow<float>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void pow<double>(PtrStepSzb src, double power, PtrStepSzb dst, hipStream_t stream);
#endif
}

//////////////////////////////////////////////////////////////////////////
// addWeighted

namespace arithm
{
    template <typename T> struct UseDouble_
    {
        enum {value = 0};
    };
    template <> struct UseDouble_<double>
    {
        enum {value = 1};
    };
    template <typename T1, typename T2, typename D> struct UseDouble
    {
        enum {value = (UseDouble_<T1>::value || UseDouble_<T2>::value || UseDouble_<D>::value)};
    };

    template <typename T1, typename T2, typename D, bool useDouble> struct AddWeighted_;
    template <typename T1, typename T2, typename D> struct AddWeighted_<T1, T2, D, false> : binary_function<T1, T2, D>
    {
        float alpha;
        float beta;
        float gamma;

        AddWeighted_(double alpha_, double beta_, double gamma_) : alpha(static_cast<float>(alpha_)), beta(static_cast<float>(beta_)), gamma(static_cast<float>(gamma_)) {}

        __device__ __forceinline__ D operator ()(T1 a, T2 b) const
        {
            return saturate_cast<D>(a * alpha + b * beta + gamma);
        }
    };
    template <typename T1, typename T2, typename D> struct AddWeighted_<T1, T2, D, true> : binary_function<T1, T2, D>
    {
        double alpha;
        double beta;
        double gamma;

        AddWeighted_(double alpha_, double beta_, double gamma_) : alpha(alpha_), beta(beta_), gamma(gamma_) {}

        __device__ __forceinline__ D operator ()(T1 a, T2 b) const
        {
            return saturate_cast<D>(a * alpha + b * beta + gamma);
        }
    };
    template <typename T1, typename T2, typename D> struct AddWeighted : AddWeighted_<T1, T2, D, UseDouble<T1, T2, D>::value>
    {
        AddWeighted(double alpha_, double beta_, double gamma_) : AddWeighted_<T1, T2, D, UseDouble<T1, T2, D>::value>(alpha_, beta_, gamma_) {}
    };
}

namespace cv { namespace gpu { namespace device
{
    template <typename T1, typename T2, typename D, size_t src1_size, size_t src2_size, size_t dst_size> struct AddWeightedTraits : DefaultTransformFunctorTraits< arithm::AddWeighted<T1, T2, D> >
    {
    };
    template <typename T1, typename T2, typename D, size_t src_size, size_t dst_size> struct AddWeightedTraits<T1, T2, D, src_size, src_size, dst_size> : arithm::ArithmFuncTraits<src_size, dst_size>
    {
    };

    template <typename T1, typename T2, typename D> struct TransformFunctorTraits< arithm::AddWeighted<T1, T2, D> > : AddWeightedTraits<T1, T2, D, sizeof(T1), sizeof(T2), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T1, typename T2, typename D>
    void addWeighted(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream)
    {
        AddWeighted<T1, T2, D> op(alpha, beta, gamma);

        transform((PtrStepSz<T1>) src1, (PtrStepSz<T2>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void addWeighted<uchar, uchar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<uchar, uchar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, schar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif


#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<schar, schar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif


#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<ushort, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif


#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<short, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif


#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<int, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<int, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<int, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif


#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<float, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif
    template void addWeighted<float, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<float, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<float, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif



#ifndef OPENCV_TINY_GPU_MODULE
    template void addWeighted<double, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
#endif
}

#endif /* CUDA_DISABLER */
