#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/gpu/device/common.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace imgproc
    {
        // Utility function to extract unsigned chars from an unsigned integer
        __device__ uchar4 int_to_uchar4(unsigned int in)
        {
            uchar4 bytes;
            bytes.x = (in & 0x000000ff) >>  0;
            bytes.y = (in & 0x0000ff00) >>  8;
            bytes.z = (in & 0x00ff0000) >> 16;
            bytes.w = (in & 0xff000000) >> 24;
            return bytes;
        }

        __global__ void shfl_integral_horizontal(const PtrStep<uint4> img, PtrStep<uint4> integral)
        {
        #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
            __shared__ int sums[128];

            const int id = threadIdx.x;
            const int lane_id = id % warpSize;
            const int warp_id = id / warpSize;

            const uint4 data = img(blockIdx.x, id);

            const uchar4 a = int_to_uchar4(data.x);
            const uchar4 b = int_to_uchar4(data.y);
            const uchar4 c = int_to_uchar4(data.z);
            const uchar4 d = int_to_uchar4(data.w);

            int result[16];

            result[0]  =              a.x;
            result[1]  = result[0]  + a.y;
            result[2]  = result[1]  + a.z;
            result[3]  = result[2]  + a.w;

            result[4]  = result[3]  + b.x;
            result[5]  = result[4]  + b.y;
            result[6]  = result[5]  + b.z;
            result[7]  = result[6]  + b.w;

            result[8]  = result[7]  + c.x;
            result[9]  = result[8]  + c.y;
            result[10] = result[9]  + c.z;
            result[11] = result[10] + c.w;

            result[12] = result[11] + d.x;
            result[13] = result[12] + d.y;
            result[14] = result[13] + d.z;
            result[15] = result[14] + d.w;

            int sum = result[15];

            // the prefix sum for each thread's 16 value is computed,
            // now the final sums (result[15]) need to be shared
            // with the other threads and add.  To do this,
            // the __shfl_up() instruction is used and a shuffle scan
            // operation is performed to distribute the sums to the correct
            // threads
            #pragma unroll
            for (int i = 1; i < 32; i *= 2)
            {
                const int n = __shfl_up(sum, i, 32);

                if (lane_id >= i)
                {
                    #pragma unroll
                    for (int i = 0; i < 16; ++i)
                        result[i] += n;

                    sum += n;
                }
            }

            // Now the final sum for the warp must be shared
            // between warps.  This is done by each warp
            // having a thread store to shared memory, then
            // having some other warp load the values and
            // compute a prefix sum, again by using __shfl_up.
            // The results are uniformly added back to the warps.
            // last thread in the warp holding sum of the warp
            // places that in shared
            if (threadIdx.x % warpSize == warpSize - 1)
                sums[warp_id] = result[15];

            __syncthreads();

            if (warp_id == 0)
            {
                int warp_sum = sums[lane_id];

                #pragma unroll
                for (int i = 1; i <= 32; i *= 2)
                {
                    const int n = __shfl_up(warp_sum, i, 32);

                    if (lane_id >= i)
                        warp_sum += n;
                }

                sums[lane_id] = warp_sum;
            }

            __syncthreads();

            int blockSum = 0;

            // fold in unused warp
            if (warp_id > 0)
            {
                blockSum = sums[warp_id - 1];

                #pragma unroll
                for (int i = 0; i < 16; ++i)
                    result[i] += blockSum;
            }

            // assemble result
            // Each thread has 16 values to write, which are
            // now integer data (to avoid overflow).  Instead of
            // each thread writing consecutive uint4s, the
            // approach shown here experiments using
            // the shuffle command to reformat the data
            // inside the registers so that each thread holds
            // consecutive data to be written so larger contiguous
            // segments can be assembled for writing.

            /*
                For example data that needs to be written as

                GMEM[16] <- x0 x1 x2 x3 y0 y1 y2 y3 z0 z1 z2 z3 w0 w1 w2 w3
                but is stored in registers (r0..r3), in four threads (0..3) as:

                threadId   0  1  2  3
                  r0      x0 y0 z0 w0
                  r1      x1 y1 z1 w1
                  r2      x2 y2 z2 w2
                  r3      x3 y3 z3 w3

                  after apply __shfl_xor operations to move data between registers r1..r3:

                threadId  00 01 10 11
                          x0 y0 z0 w0
                 xor(01)->y1 x1 w1 z1
                 xor(10)->z2 w2 x2 y2
                 xor(11)->w3 z3 y3 x3

                 and now x0..x3, and z0..z3 can be written out in order by all threads.

                 In the current code, each register above is actually representing
                 four integers to be written as uint4's to GMEM.
            */

            result[4]  = __shfl_xor(result[4] , 1, 32);
            result[5]  = __shfl_xor(result[5] , 1, 32);
            result[6]  = __shfl_xor(result[6] , 1, 32);
            result[7]  = __shfl_xor(result[7] , 1, 32);

            result[8]  = __shfl_xor(result[8] , 2, 32);
            result[9]  = __shfl_xor(result[9] , 2, 32);
            result[10] = __shfl_xor(result[10], 2, 32);
            result[11] = __shfl_xor(result[11], 2, 32);

            result[12] = __shfl_xor(result[12], 3, 32);
            result[13] = __shfl_xor(result[13], 3, 32);
            result[14] = __shfl_xor(result[14], 3, 32);
            result[15] = __shfl_xor(result[15], 3, 32);

            uint4* integral_row = integral.ptr(blockIdx.x);
            uint4 output;

            ///////

            if (threadIdx.x % 4 == 0)
                output = make_uint4(result[0], result[1], result[2], result[3]);

            if (threadIdx.x % 4 == 1)
                output = make_uint4(result[4], result[5], result[6], result[7]);

            if (threadIdx.x % 4 == 2)
                output = make_uint4(result[8], result[9], result[10], result[11]);

            if (threadIdx.x % 4 == 3)
                output = make_uint4(result[12], result[13], result[14], result[15]);

            integral_row[threadIdx.x % 4 + (threadIdx.x / 4) * 16] = output;

            ///////

            if (threadIdx.x % 4 == 2)
                output = make_uint4(result[0], result[1], result[2], result[3]);

            if (threadIdx.x % 4 == 3)
                output = make_uint4(result[4], result[5], result[6], result[7]);

            if (threadIdx.x % 4 == 0)
                output = make_uint4(result[8], result[9], result[10], result[11]);

            if (threadIdx.x % 4 == 1)
                output = make_uint4(result[12], result[13], result[14], result[15]);

            integral_row[(threadIdx.x + 2) % 4 + (threadIdx.x / 4) * 16 + 8] = output;

            // continuning from the above example,
            // this use of __shfl_xor() places the y0..y3 and w0..w3 data
            // in order.

            #pragma unroll
            for (int i = 0; i < 16; ++i)
                result[i] = __shfl_xor(result[i], 1, 32);

            if (threadIdx.x % 4 == 0)
                output = make_uint4(result[0], result[1], result[2], result[3]);

            if (threadIdx.x % 4 == 1)
                output = make_uint4(result[4], result[5], result[6], result[7]);

            if (threadIdx.x % 4 == 2)
                output = make_uint4(result[8], result[9], result[10], result[11]);

            if (threadIdx.x % 4 == 3)
                output = make_uint4(result[12], result[13], result[14], result[15]);

            integral_row[threadIdx.x % 4 + (threadIdx.x / 4) * 16 + 4] = output;

            ///////

            if (threadIdx.x % 4 == 2)
                output = make_uint4(result[0], result[1], result[2], result[3]);

            if (threadIdx.x % 4 == 3)
                output = make_uint4(result[4], result[5], result[6], result[7]);

            if (threadIdx.x % 4 == 0)
                output = make_uint4(result[8], result[9], result[10], result[11]);

            if (threadIdx.x % 4 == 1)
                output = make_uint4(result[12], result[13], result[14], result[15]);

            integral_row[(threadIdx.x + 2) % 4 + (threadIdx.x / 4) * 16 + 12] = output;
        #endif
        }

        // This kernel computes columnwise prefix sums.  When the data input is
        // the row sums from above, this completes the integral image.
        // The approach here is to have each block compute a local set of sums.
        // First , the data covered by the block is loaded into shared memory,
        // then instead of performing a sum in shared memory using __syncthreads
        // between stages, the data is reformatted so that the necessary sums
        // occur inside warps and the shuffle scan operation is used.
        // The final set of sums from the block is then propgated, with the block
        // computing "down" the image and adding the running sum to the local
        // block sums.
        __global__ void shfl_integral_vertical(PtrStepSz<unsigned int> integral)
        {
        #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 300)
            __shared__ unsigned int sums[32][9];

            const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
            const int lane_id = tidx % 8;

            if (tidx >= integral.cols)
                return;

            sums[threadIdx.x][threadIdx.y] = 0;
            __syncthreads();

            unsigned int stepSum = 0;

            for (int y = threadIdx.y; y < integral.rows; y += blockDim.y)
            {
                unsigned int* p = integral.ptr(y) + tidx;

                unsigned int sum = *p;

                sums[threadIdx.x][threadIdx.y] = sum;
                __syncthreads();

                // place into SMEM
                // shfl scan reduce the SMEM, reformating so the column
                // sums are computed in a warp
                // then read out properly
                const int j = threadIdx.x % 8;
                const int k = threadIdx.x / 8 + threadIdx.y * 4;

                int partial_sum = sums[k][j];

                for (int i = 1; i <= 8; i *= 2)
                {
                    int n = __shfl_up(partial_sum, i, 32);

                    if (lane_id >= i)
                        partial_sum += n;
                }

                sums[k][j] = partial_sum;
                __syncthreads();

                if (threadIdx.y > 0)
                    sum += sums[threadIdx.x][threadIdx.y - 1];

                sum += stepSum;
                stepSum += sums[threadIdx.x][blockDim.y - 1];

                __syncthreads();

                *p = sum;
            }
        #endif
        }

        void shfl_integral_gpu(const PtrStepSzb& img, PtrStepSz<unsigned int> integral, hipStream_t stream)
        {
            {
                // each thread handles 16 values, use 1 block/row
                // save, becouse step is actually can't be less 512 bytes
                int block = integral.cols / 16;

                // launch 1 block / row
                const int grid = img.rows;

                cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(shfl_integral_horizontal), hipFuncCachePreferL1) );

                shfl_integral_horizontal<<<grid, block, 0, stream>>>((const PtrStepSz<uint4>) img, (PtrStepSz<uint4>) integral);
                cudaSafeCall( hipGetLastError() );
            }

            {
                const dim3 block(32, 8);
                const dim3 grid(divUp(integral.cols, block.x), 1);

                shfl_integral_vertical<<<grid, block, 0, stream>>>(integral);
                cudaSafeCall( hipGetLastError() );
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}

#endif /* CUDA_DISABLER */
