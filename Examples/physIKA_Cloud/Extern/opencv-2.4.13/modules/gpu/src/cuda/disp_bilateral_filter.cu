#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace disp_bilateral_filter
    {
        __constant__ float* ctable_color;
        __constant__ float* ctable_space;
        __constant__ size_t ctable_space_step;

        __constant__ int cndisp;
        __constant__ int cradius;

        __constant__ short cedge_disc;
        __constant__ short cmax_disc;

        void disp_load_constants(float* table_color, PtrStepSzf table_space, int ndisp, int radius, short edge_disc, short max_disc)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(ctable_color), &table_color, sizeof(table_color)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(ctable_space), &table_space.data, sizeof(table_space.data)) );
            size_t table_space_step = table_space.step / sizeof(float);
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(ctable_space_step), &table_space_step, sizeof(size_t)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cndisp), &ndisp, sizeof(int)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cradius), &radius, sizeof(int)) );

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cedge_disc), &edge_disc, sizeof(short)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_disc), &max_disc, sizeof(short)) );
        }

        template <int channels>
        struct DistRgbMax
        {
            static __device__ __forceinline__ uchar calc(const uchar* a, const uchar* b)
            {
                uchar x = ::abs(a[0] - b[0]);
                uchar y = ::abs(a[1] - b[1]);
                uchar z = ::abs(a[2] - b[2]);
                return (::max(::max(x, y), z));
            }
        };

        template <>
        struct DistRgbMax<1>
        {
            static __device__ __forceinline__ uchar calc(const uchar* a, const uchar* b)
            {
                return ::abs(a[0] - b[0]);
            }
        };

        template <int channels, typename T>
        __global__ void disp_bilateral_filter(int t, T* disp, size_t disp_step, const uchar* img, size_t img_step, int h, int w)
        {
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            const int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + t) & 1);

            T dp[5];

            if (y > 0 && y < h - 1 && x > 0 && x < w - 1)
            {
                dp[0] = *(disp + (y  ) * disp_step + x + 0);
                dp[1] = *(disp + (y-1) * disp_step + x + 0);
                dp[2] = *(disp + (y  ) * disp_step + x - 1);
                dp[3] = *(disp + (y+1) * disp_step + x + 0);
                dp[4] = *(disp + (y  ) * disp_step + x + 1);

                if(::abs(dp[1] - dp[0]) >= cedge_disc || ::abs(dp[2] - dp[0]) >= cedge_disc || ::abs(dp[3] - dp[0]) >= cedge_disc || ::abs(dp[4] - dp[0]) >= cedge_disc)
                {
                    const int ymin = ::max(0, y - cradius);
                    const int xmin = ::max(0, x - cradius);
                    const int ymax = ::min(h - 1, y + cradius);
                    const int xmax = ::min(w - 1, x + cradius);

                    float cost[] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};

                    const uchar* ic = img + y * img_step + channels * x;

                    for(int yi = ymin; yi <= ymax; yi++)
                    {
                        const T* disp_y = disp + yi * disp_step;

                        for(int xi = xmin; xi <= xmax; xi++)
                        {
                            const uchar* in = img + yi * img_step + channels * xi;

                            uchar dist_rgb = DistRgbMax<channels>::calc(in, ic);

                            const float weight = ctable_color[dist_rgb] * (ctable_space + ::abs(y-yi)* ctable_space_step)[::abs(x-xi)];

                            const T disp_reg = disp_y[xi];

                            cost[0] += ::min(cmax_disc, ::abs(disp_reg - dp[0])) * weight;
                            cost[1] += ::min(cmax_disc, ::abs(disp_reg - dp[1])) * weight;
                            cost[2] += ::min(cmax_disc, ::abs(disp_reg - dp[2])) * weight;
                            cost[3] += ::min(cmax_disc, ::abs(disp_reg - dp[3])) * weight;
                            cost[4] += ::min(cmax_disc, ::abs(disp_reg - dp[4])) * weight;
                        }
                    }

                    float minimum = numeric_limits<float>::max();
                    int id = 0;

                    if (cost[0] < minimum)
                    {
                        minimum = cost[0];
                        id = 0;
                    }
                    if (cost[1] < minimum)
                    {
                        minimum = cost[1];
                        id = 1;
                    }
                    if (cost[2] < minimum)
                    {
                        minimum = cost[2];
                        id = 2;
                    }
                    if (cost[3] < minimum)
                    {
                        minimum = cost[3];
                        id = 3;
                    }
                    if (cost[4] < minimum)
                    {
                        minimum = cost[4];
                        id = 4;
                    }

                    *(disp + y * disp_step + x) = dp[id];
                }
            }
        }

        template <typename T>
        void disp_bilateral_filter(PtrStepSz<T> disp, PtrStepSzb img, int channels, int iters, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(disp.cols, threads.x << 1);
            grid.y = divUp(disp.rows, threads.y);

            switch (channels)
            {
            case 1:
                for (int i = 0; i < iters; ++i)
                {
                    disp_bilateral_filter<1><<<grid, threads, 0, stream>>>(0, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols);
                    cudaSafeCall( hipGetLastError() );

                    disp_bilateral_filter<1><<<grid, threads, 0, stream>>>(1, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols);
                    cudaSafeCall( hipGetLastError() );
                }
                break;
            case 3:
                for (int i = 0; i < iters; ++i)
                {
                    disp_bilateral_filter<3><<<grid, threads, 0, stream>>>(0, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols);
                    cudaSafeCall( hipGetLastError() );

                    disp_bilateral_filter<3><<<grid, threads, 0, stream>>>(1, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols);
                    cudaSafeCall( hipGetLastError() );
                }
                break;
            default:
                cv::gpu::error("Unsupported channels count", __FILE__, __LINE__, "disp_bilateral_filter");
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void disp_bilateral_filter<uchar>(PtrStepSz<uchar> disp, PtrStepSzb img, int channels, int iters, hipStream_t stream);
        template void disp_bilateral_filter<short>(PtrStepSz<short> disp, PtrStepSzb img, int channels, int iters, hipStream_t stream);
    } // namespace bilateral_filter
}}} // namespace cv { namespace gpu { namespace device

#endif /* CUDA_DISABLER */
