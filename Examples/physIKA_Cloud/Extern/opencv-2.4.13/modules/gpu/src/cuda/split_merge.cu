#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "internal_shared.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace split_merge
    {
        template <typename T, size_t elem_size = sizeof(T)>
        struct TypeTraits
        {
            typedef T type;
            typedef T type2;
            typedef T type3;
            typedef T type4;
        };

        template <typename T>
        struct TypeTraits<T, 1>
        {
            typedef char type;
            typedef char2 type2;
            typedef char3 type3;
            typedef char4 type4;
        };

        template <typename T>
        struct TypeTraits<T, 2>
        {
            typedef short type;
            typedef short2 type2;
            typedef short3 type3;
            typedef short4 type4;
        };

        template <typename T>
        struct TypeTraits<T, 4>
        {
            typedef int type;
            typedef int2 type2;
            typedef int3 type3;
            typedef int4 type4;
        };

        template <typename T>
        struct TypeTraits<T, 8>
        {
            typedef double type;
            typedef double2 type2;
            //typedef double3 type3;
            //typedef double4 type3;
        };

        typedef void (*MergeFunction)(const PtrStepSzb* src, PtrStepSzb& dst, const hipStream_t& stream);
        typedef void (*SplitFunction)(const PtrStepSzb& src, PtrStepSzb* dst, const hipStream_t& stream);

        //------------------------------------------------------------
        // Merge

        template <typename T>
        __global__ void mergeC2_(const uchar* src0, size_t src0_step,
                                 const uchar* src1, size_t src1_step,
                                 int rows, int cols, uchar* dst, size_t dst_step)
        {
            typedef typename TypeTraits<T>::type2 dst_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const T* src0_y = (const T*)(src0 + y * src0_step);
            const T* src1_y = (const T*)(src1 + y * src1_step);
            dst_type* dst_y = (dst_type*)(dst + y * dst_step);

            if (x < cols && y < rows)
            {
                dst_type dst_elem;
                dst_elem.x = src0_y[x];
                dst_elem.y = src1_y[x];
                dst_y[x] = dst_elem;
            }
        }


        template <typename T>
        __global__ void mergeC3_(const uchar* src0, size_t src0_step,
                                 const uchar* src1, size_t src1_step,
                                 const uchar* src2, size_t src2_step,
                                 int rows, int cols, uchar* dst, size_t dst_step)
        {
            typedef typename TypeTraits<T>::type3 dst_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const T* src0_y = (const T*)(src0 + y * src0_step);
            const T* src1_y = (const T*)(src1 + y * src1_step);
            const T* src2_y = (const T*)(src2 + y * src2_step);
            dst_type* dst_y = (dst_type*)(dst + y * dst_step);

            if (x < cols && y < rows)
            {
                dst_type dst_elem;
                dst_elem.x = src0_y[x];
                dst_elem.y = src1_y[x];
                dst_elem.z = src2_y[x];
                dst_y[x] = dst_elem;
            }
        }


        template <>
        __global__ void mergeC3_<double>(const uchar* src0, size_t src0_step,
                                 const uchar* src1, size_t src1_step,
                                 const uchar* src2, size_t src2_step,
                                 int rows, int cols, uchar* dst, size_t dst_step)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const double* src0_y = (const double*)(src0 + y * src0_step);
            const double* src1_y = (const double*)(src1 + y * src1_step);
            const double* src2_y = (const double*)(src2 + y * src2_step);
            double* dst_y = (double*)(dst + y * dst_step);

            if (x < cols && y < rows)
            {
                dst_y[3 * x] = src0_y[x];
                dst_y[3 * x + 1] = src1_y[x];
                dst_y[3 * x + 2] = src2_y[x];
            }
        }


        template <typename T>
        __global__ void mergeC4_(const uchar* src0, size_t src0_step,
                                 const uchar* src1, size_t src1_step,
                                 const uchar* src2, size_t src2_step,
                                 const uchar* src3, size_t src3_step,
                                 int rows, int cols, uchar* dst, size_t dst_step)
        {
            typedef typename TypeTraits<T>::type4 dst_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const T* src0_y = (const T*)(src0 + y * src0_step);
            const T* src1_y = (const T*)(src1 + y * src1_step);
            const T* src2_y = (const T*)(src2 + y * src2_step);
            const T* src3_y = (const T*)(src3 + y * src3_step);
            dst_type* dst_y = (dst_type*)(dst + y * dst_step);

            if (x < cols && y < rows)
            {
                dst_type dst_elem;
                dst_elem.x = src0_y[x];
                dst_elem.y = src1_y[x];
                dst_elem.z = src2_y[x];
                dst_elem.w = src3_y[x];
                dst_y[x] = dst_elem;
            }
        }


        template <>
        __global__ void mergeC4_<double>(const uchar* src0, size_t src0_step,
                                 const uchar* src1, size_t src1_step,
                                 const uchar* src2, size_t src2_step,
                                 const uchar* src3, size_t src3_step,
                                 int rows, int cols, uchar* dst, size_t dst_step)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const double* src0_y = (const double*)(src0 + y * src0_step);
            const double* src1_y = (const double*)(src1 + y * src1_step);
            const double* src2_y = (const double*)(src2 + y * src2_step);
            const double* src3_y = (const double*)(src3 + y * src3_step);
            double2* dst_y = (double2*)(dst + y * dst_step);

            if (x < cols && y < rows)
            {
                dst_y[2 * x] = make_double2(src0_y[x], src1_y[x]);
                dst_y[2 * x + 1] = make_double2(src2_y[x], src3_y[x]);
            }
        }


        template <typename T>
        static void mergeC2_(const PtrStepSzb* src, PtrStepSzb& dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
            mergeC2_<T><<<grid, block, 0, stream>>>(
                    src[0].data, src[0].step,
                    src[1].data, src[1].step,
                    dst.rows, dst.cols, dst.data, dst.step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        template <typename T>
        static void mergeC3_(const PtrStepSzb* src, PtrStepSzb& dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
            mergeC3_<T><<<grid, block, 0, stream>>>(
                    src[0].data, src[0].step,
                    src[1].data, src[1].step,
                    src[2].data, src[2].step,
                    dst.rows, dst.cols, dst.data, dst.step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        template <typename T>
        static void mergeC4_(const PtrStepSzb* src, PtrStepSzb& dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));
            mergeC4_<T><<<grid, block, 0, stream>>>(
                    src[0].data, src[0].step,
                    src[1].data, src[1].step,
                    src[2].data, src[2].step,
                    src[3].data, src[3].step,
                    dst.rows, dst.cols, dst.data, dst.step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void merge_caller(const PtrStepSzb* src, PtrStepSzb& dst,
                                     int total_channels, size_t elem_size,
                                     const hipStream_t& stream)
        {
            static MergeFunction merge_func_tbl[] =
            {
                mergeC2_<char>, mergeC2_<short>, mergeC2_<int>, 0, mergeC2_<double>,
                mergeC3_<char>, mergeC3_<short>, mergeC3_<int>, 0, mergeC3_<double>,
                mergeC4_<char>, mergeC4_<short>, mergeC4_<int>, 0, mergeC4_<double>,
            };

            size_t merge_func_id = (total_channels - 2) * 5 + (elem_size >> 1);
            MergeFunction merge_func = merge_func_tbl[merge_func_id];

            if (merge_func == 0)
                cv::gpu::error("Unsupported channel count or data type", __FILE__, __LINE__, "merge_caller");

            merge_func(src, dst, stream);
        }



        //------------------------------------------------------------
        // Split


        template <typename T>
        __global__ void splitC2_(const uchar* src, size_t src_step,
                                int rows, int cols,
                                uchar* dst0, size_t dst0_step,
                                uchar* dst1, size_t dst1_step)
        {
            typedef typename TypeTraits<T>::type2 src_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const src_type* src_y = (const src_type*)(src + y * src_step);
            T* dst0_y = (T*)(dst0 + y * dst0_step);
            T* dst1_y = (T*)(dst1 + y * dst1_step);

            if (x < cols && y < rows)
            {
                src_type src_elem = src_y[x];
                dst0_y[x] = src_elem.x;
                dst1_y[x] = src_elem.y;
            }
        }


        template <typename T>
        __global__ void splitC3_(const uchar* src, size_t src_step,
                                int rows, int cols,
                                uchar* dst0, size_t dst0_step,
                                uchar* dst1, size_t dst1_step,
                                uchar* dst2, size_t dst2_step)
        {
            typedef typename TypeTraits<T>::type3 src_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const src_type* src_y = (const src_type*)(src + y * src_step);
            T* dst0_y = (T*)(dst0 + y * dst0_step);
            T* dst1_y = (T*)(dst1 + y * dst1_step);
            T* dst2_y = (T*)(dst2 + y * dst2_step);

            if (x < cols && y < rows)
            {
                src_type src_elem = src_y[x];
                dst0_y[x] = src_elem.x;
                dst1_y[x] = src_elem.y;
                dst2_y[x] = src_elem.z;
            }
        }


        template <>
        __global__ void splitC3_<double>(
                const uchar* src, size_t src_step, int rows, int cols,
                uchar* dst0, size_t dst0_step,
                uchar* dst1, size_t dst1_step,
                uchar* dst2, size_t dst2_step)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const double* src_y = (const double*)(src + y * src_step);
            double* dst0_y = (double*)(dst0 + y * dst0_step);
            double* dst1_y = (double*)(dst1 + y * dst1_step);
            double* dst2_y = (double*)(dst2 + y * dst2_step);

            if (x < cols && y < rows)
            {
                dst0_y[x] = src_y[3 * x];
                dst1_y[x] = src_y[3 * x + 1];
                dst2_y[x] = src_y[3 * x + 2];
            }
        }


        template <typename T>
        __global__ void splitC4_(const uchar* src, size_t src_step, int rows, int cols,
                                uchar* dst0, size_t dst0_step,
                                uchar* dst1, size_t dst1_step,
                                uchar* dst2, size_t dst2_step,
                                uchar* dst3, size_t dst3_step)
        {
            typedef typename TypeTraits<T>::type4 src_type;

            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const src_type* src_y = (const src_type*)(src + y * src_step);
            T* dst0_y = (T*)(dst0 + y * dst0_step);
            T* dst1_y = (T*)(dst1 + y * dst1_step);
            T* dst2_y = (T*)(dst2 + y * dst2_step);
            T* dst3_y = (T*)(dst3 + y * dst3_step);

            if (x < cols && y < rows)
            {
                src_type src_elem = src_y[x];
                dst0_y[x] = src_elem.x;
                dst1_y[x] = src_elem.y;
                dst2_y[x] = src_elem.z;
                dst3_y[x] = src_elem.w;
            }
        }


        template <>
        __global__ void splitC4_<double>(
                const uchar* src, size_t src_step, int rows, int cols,
                uchar* dst0, size_t dst0_step,
                uchar* dst1, size_t dst1_step,
                uchar* dst2, size_t dst2_step,
                uchar* dst3, size_t dst3_step)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const double2* src_y = (const double2*)(src + y * src_step);
            double* dst0_y = (double*)(dst0 + y * dst0_step);
            double* dst1_y = (double*)(dst1 + y * dst1_step);
            double* dst2_y = (double*)(dst2 + y * dst2_step);
            double* dst3_y = (double*)(dst3 + y * dst3_step);

            if (x < cols && y < rows)
            {
                double2 src_elem1 = src_y[2 * x];
                double2 src_elem2 = src_y[2 * x + 1];
                dst0_y[x] = src_elem1.x;
                dst1_y[x] = src_elem1.y;
                dst2_y[x] = src_elem2.x;
                dst3_y[x] = src_elem2.y;
            }
        }

        template <typename T>
        static void splitC2_(const PtrStepSzb& src, PtrStepSzb* dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));
            splitC2_<T><<<grid, block, 0, stream>>>(
                    src.data, src.step, src.rows, src.cols,
                    dst[0].data, dst[0].step,
                    dst[1].data, dst[1].step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        template <typename T>
        static void splitC3_(const PtrStepSzb& src, PtrStepSzb* dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));
            splitC3_<T><<<grid, block, 0, stream>>>(
                    src.data, src.step, src.rows, src.cols,
                    dst[0].data, dst[0].step,
                    dst[1].data, dst[1].step,
                    dst[2].data, dst[2].step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        template <typename T>
        static void splitC4_(const PtrStepSzb& src, PtrStepSzb* dst, const hipStream_t& stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));
            splitC4_<T><<<grid, block, 0, stream>>>(
                     src.data, src.step, src.rows, src.cols,
                     dst[0].data, dst[0].step,
                     dst[1].data, dst[1].step,
                     dst[2].data, dst[2].step,
                     dst[3].data, dst[3].step);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall(hipDeviceSynchronize());
        }


        void split_caller(const PtrStepSzb& src, PtrStepSzb* dst, int num_channels, size_t elem_size1, const hipStream_t& stream)
        {
            static SplitFunction split_func_tbl[] =
            {
                splitC2_<char>, splitC2_<short>, splitC2_<int>, 0, splitC2_<double>,
                splitC3_<char>, splitC3_<short>, splitC3_<int>, 0, splitC3_<double>,
                splitC4_<char>, splitC4_<short>, splitC4_<int>, 0, splitC4_<double>,
            };

            size_t split_func_id = (num_channels - 2) * 5 + (elem_size1 >> 1);
            SplitFunction split_func = split_func_tbl[split_func_id];

            if (split_func == 0)
                cv::gpu::error("Unsupported channel count or data type", __FILE__, __LINE__, "split_caller");

            split_func(src, dst, stream);
        }
    } // namespace split_merge
}}} // namespace cv { namespace gpu { namespace device


#endif /* CUDA_DISABLER */
